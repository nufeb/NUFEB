#include "hip/hip_runtime.h"
// **************************************************************************
//                                  pppm.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for PPPM acceleration
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_preprocessor.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

// Allow PPPM to compile without atomics for NVIDIA 1.0 cards, error
// generated at runtime with use of pppm/gpu
#if (__CUDA_ARCH__ < 110)
#define atomicAdd(x,y) *(x)+=0
#endif

#else

#define pos_tex x_
#define q_tex q_
#pragma OPENCL EXTENSION cl_khr_global_int32_base_atomics: enable
#if defined(cl_amd_fp64)
#pragma OPENCL EXTENSION cl_amd_fp64 : enable
#else
#pragma OPENCL EXTENSION cl_khr_fp64 : enable
#endif

#endif

// Number of threads per pencil for charge spread
#define PENCIL_SIZE MEM_THREADS
// Number of pencils per block for charge spread
#define BLOCK_PENCILS (PPPM_BLOCK_1D/PENCIL_SIZE)

__kernel void particle_map(const __global numtyp4 *restrict x_,
                           const __global numtyp *restrict q_,
                           const grdtyp delvolinv, const int nlocal,
                           __global int *restrict counts,
                           __global grdtyp4 *restrict ans,
                           const grdtyp b_lo_x, const grdtyp b_lo_y,
                           const grdtyp b_lo_z, const grdtyp delxinv,
                           const grdtyp delyinv, const grdtyp delzinv,
                           const int nlocal_x, const int nlocal_y,
                           const int nlocal_z, const int atom_stride,
                           const int max_atoms,
                           __global int *restrict error) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;

  // Resequence the atom indices to avoid collisions during atomic ops
  int nthreads=GLOBAL_SIZE_X;
  ii=fast_mul(ii,PPPM_BLOCK_1D);
  ii-=(ii/nthreads)*(nthreads-1);

  int nx,ny,nz;

  if (ii<nlocal) {
    numtyp4 p;
    fetch4(p,ii,pos_tex);
    grdtyp4 delta;
    fetch(delta.w,ii,q_tex);
    delta.w*=delvolinv;

    if (delta.w!=(grdtyp)0.0) {
      delta.x=(p.x-b_lo_x)*delxinv;
      nx=delta.x;
      delta.y=(p.y-b_lo_y)*delyinv;
      ny=delta.y;
      delta.z=(p.z-b_lo_z)*delzinv;
      nz=delta.z;

      if (delta.x<(grdtyp)0 || delta.y<(grdtyp)0 || delta.z<(grdtyp)0 ||
          nx>=nlocal_x || ny>=nlocal_y || nz>=nlocal_z)
        *error=1;
      else {
        delta.x=nx+(grdtyp)0.5-delta.x;
        delta.y=ny+(grdtyp)0.5-delta.y;
        delta.z=nz+(grdtyp)0.5-delta.z;

        int i=nz*nlocal_y*nlocal_x+ny*nlocal_x+nx;
        int old=atom_add(counts+i, 1);
        if (old>=max_atoms) {
          *error=2;
          atom_add(counts+i, -1);
        } else
          ans[atom_stride*old+i]=delta;
      }
    }
  }
}

/* --------------------------- */

__kernel void make_rho(const __global int *restrict counts,
                       const __global grdtyp4 *restrict atoms,
                       __global grdtyp *restrict brick,
                       const __global grdtyp *restrict _rho_coeff,
                       const int atom_stride, const int npts_x,
                       const int npts_y, const int npts_z, const int nlocal_x,
                       const int nlocal_y, const int nlocal_z,
                       const int order_m_1, const int order, const int order2) {
  __local grdtyp rho_coeff[PPPM_MAX_SPLINE*PPPM_MAX_SPLINE];
  __local grdtyp front[BLOCK_PENCILS][PENCIL_SIZE+PPPM_MAX_SPLINE];
  __local grdtyp ans[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];

  int tid=THREAD_ID_X;
  if (tid<order2+order)
    rho_coeff[tid]=_rho_coeff[tid];

  int pid=tid/PENCIL_SIZE;
  int fid=tid%PENCIL_SIZE;
  int fid_halo=PENCIL_SIZE+fid;
  if (fid<order)
    front[pid][fid_halo]=(grdtyp)0.0;

  __syncthreads();

  int bt=BLOCK_ID_X*BLOCK_PENCILS+pid;
  int ny=bt%npts_y;
  int nz=bt/npts_y;
  int y_start=0;
  int z_start=0;
  int y_stop=order;
  int z_stop=order;
  if (ny<order_m_1)
    y_start=order_m_1-ny;
  if (nz<order_m_1)
    z_start=order_m_1-nz;
  if (ny>=nlocal_y)
    y_stop-=ny-nlocal_y+1;
  if (nz>=nlocal_z)
    z_stop-=nz-nlocal_z+1;
  int z_stride=fast_mul(nlocal_x,nlocal_y);

  int loop_count=npts_x/PENCIL_SIZE+1;
  int nx=fid;
  int pt=fast_mul(nz,fast_mul(npts_y,npts_x))+fast_mul(ny,npts_x)+nx;
  for (int i=0 ; i<loop_count; i++) {
    for (int n=0; n<order; n++)
      ans[n][tid]=(grdtyp)0.0;
    if (nx<nlocal_x && nz<npts_z) {
      int z_pos=fast_mul(nz+z_start-order_m_1,z_stride);
      for (int m=z_start; m<z_stop; m++) {
        int y_pos=fast_mul(ny+y_start-order_m_1,nlocal_x);
        for (int l=y_start; l<y_stop; l++) {
          int pos=z_pos+y_pos+nx;
          int natoms=fast_mul(counts[pos],atom_stride);
          for (int row=pos; row<natoms; row+=atom_stride) {
            grdtyp4 delta=atoms[row];

            grdtyp rho1d_1=(grdtyp)0.0;
            grdtyp rho1d_2=(grdtyp)0.0;
            for (int k=order2+order-1; k > -1; k-=order) {
              rho1d_1=rho_coeff[k-l]+rho1d_1*delta.y;
              rho1d_2=rho_coeff[k-m]+rho1d_2*delta.z;
            }
            delta.w*=rho1d_1*rho1d_2;

            for (int n=0; n<order; n++) {
              grdtyp rho1d_0=(grdtyp)0.0;
              for (int k=order2+n; k>=n; k-=order)
                rho1d_0=rho_coeff[k]+rho1d_0*delta.x;
              ans[n][tid]+=delta.w*rho1d_0;
            }
          }
          y_pos+=nlocal_x;
        }
        z_pos+=z_stride;
      }
    }

    __syncthreads();
    if (fid<order) {
      front[pid][fid]=front[pid][fid_halo];
      front[pid][fid_halo]=(grdtyp)0.0;
    } else
      front[pid][fid]=(grdtyp)0.0;

    for (int n=0; n<order; n++) {
      front[pid][fid+n]+=ans[n][tid];
      __syncthreads();
    }

    if (nx<npts_x && nz<npts_z)
      brick[pt]=front[pid][fid];
    pt+=PENCIL_SIZE;
    nx+=PENCIL_SIZE;
  }
}

__kernel void interp(const __global numtyp4 *restrict x_,
                     const __global numtyp *restrict q_,
                     const int nlocal,
                     const __global grdtyp4 *restrict brick,
                     const __global grdtyp *restrict _rho_coeff,
                     const int npts_x, const int npts_yx, const grdtyp b_lo_x,
                     const grdtyp b_lo_y, const grdtyp b_lo_z,
                     const grdtyp delxinv,  const grdtyp delyinv,
                     const grdtyp delzinv, const int order,
                     const int order2, const grdtyp qqrd2e_scale,
                     __global acctyp4 *restrict ans) {
  __local grdtyp rho_coeff[PPPM_MAX_SPLINE*PPPM_MAX_SPLINE];
  __local grdtyp rho1d_0[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];
  __local grdtyp rho1d_1[PPPM_MAX_SPLINE][PPPM_BLOCK_1D];

  int tid=THREAD_ID_X;
  if (tid<order2+order)
    rho_coeff[tid]=_rho_coeff[tid];
  __syncthreads();

  int ii=tid+BLOCK_ID_X*BLOCK_SIZE_X;

  int nx,ny,nz;
  grdtyp tx,ty,tz;

  if (ii<nlocal) {
    numtyp4 p;
    fetch4(p,ii,pos_tex);
    grdtyp qs;
    fetch(qs,ii,q_tex);
    qs*=qqrd2e_scale;

    acctyp4 ek;
    ek.x=(acctyp)0.0;
    ek.y=(acctyp)0.0;
    ek.z=(acctyp)0.0;
    if (qs!=(grdtyp)0.0) {
      tx=(p.x-b_lo_x)*delxinv;
      nx=tx;
      ty=(p.y-b_lo_y)*delyinv;
      ny=ty;
      tz=(p.z-b_lo_z)*delzinv;
      nz=tz;

      grdtyp dx=nx+(grdtyp)0.5-tx;
      grdtyp dy=ny+(grdtyp)0.5-ty;
      grdtyp dz=nz+(grdtyp)0.5-tz;

      for (int k=0; k<order; k++) {
        rho1d_0[k][tid]=(grdtyp)0.0;
        rho1d_1[k][tid]=(grdtyp)0.0;
        for (int l=order2+k; l>=k; l-=order) {
          rho1d_0[k][tid]=rho_coeff[l]+rho1d_0[k][tid]*dx;
          rho1d_1[k][tid]=rho_coeff[l]+rho1d_1[k][tid]*dy;
        }
      }

      int mz=fast_mul(nz,npts_yx)+nx;
      for (int n=0; n<order; n++) {
        grdtyp rho1d_2=(grdtyp)0.0;
        for (int k=order2+n; k>=n; k-=order)
          rho1d_2=rho_coeff[k]+rho1d_2*dz;
        grdtyp z0=qs*rho1d_2;
        int my=mz+fast_mul(ny,npts_x);
        for (int m=0; m<order; m++) {
          grdtyp y0=z0*rho1d_1[m][tid];
                for (int l=0; l<order; l++) {
                  grdtyp x0=y0*rho1d_0[l][tid];
                  grdtyp4 el=brick[my+l];
                  ek.x-=x0*el.x;
                  ek.y-=x0*el.y;
                  ek.z-=x0*el.z;
                }
          my+=npts_x;
        }
        mz+=npts_yx;
            }
    }
    ans[ii]=ek;
        }
}

