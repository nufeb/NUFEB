#include "hip/hip_runtime.h"
// **************************************************************************
//                               neighbor_gpu.cu
//                             -------------------
//                              Peng Wang (Nvidia)
//                           W. Michael Brown (ORNL)
//
//  Device code for handling GPU generated neighbor lists
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : penwang@nvidia.com, brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_preprocessor.h"
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#include "inttypes.h"
#define tagint int64_t
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif

__kernel void calc_cell_id(const numtyp4 *restrict pos,
                           unsigned *restrict cell_id,
                           int *restrict particle_id,
                           numtyp boxlo0, numtyp boxlo1, numtyp boxlo2,
                           numtyp i_cell_size, int ncellx, int ncelly,
                           int ncellz, int inum, int nall,
                           int cells_in_cutoff) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nall) {
    numtyp4 p;
    fetch4(p,i,pos_tex); //pos[i];

    p.x -= boxlo0;
    p.y -= boxlo1;
    p.z -= boxlo2;

    int ix = int(p.x*i_cell_size+cells_in_cutoff);
    int iy = int(p.y*i_cell_size+cells_in_cutoff);
    int iz = int(p.z*i_cell_size+cells_in_cutoff);

    int offset_lo, offset_hi;
    if (i<inum) {
      offset_lo=cells_in_cutoff;
      offset_hi=cells_in_cutoff+1;
    } else {
      offset_lo=0;
      offset_hi=1;
    }

    ix = max(ix,offset_lo);
    ix = min(ix,ncellx-offset_hi);
    iy = max(iy,offset_lo);
    iy = min(iy,ncelly-offset_hi);
    iz = max(iz,offset_lo);
    iz = min(iz,ncellz-offset_hi);

    cell_id[i] = ix+iy*ncellx+iz*ncellx*ncelly;
    particle_id[i] = i;
  }
}

__kernel void kernel_calc_cell_counts(const unsigned *restrict cell_id,
                                      int *restrict cell_counts,
                                      int nall, int ncell) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nall) {
    int id = cell_id[idx];

    // handle boundary cases
    if (idx == 0) {
      for (int i = 0; i < id + 1; i++)
        cell_counts[i] = 0;
    }
    if (idx == nall - 1) {
      for (int i = id+1; i <= ncell; i++)
        cell_counts[i] = nall;
    }

    if (idx > 0 && idx < nall) {
      int id_l = cell_id[idx-1];
      if (id != id_l) {
        for (int i = id_l+1; i <= id; i++)
          cell_counts[i] = idx;
      }
    }
  }
}

#else
#define pos_tex x_
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#define tagint long long int
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#endif

__kernel void transpose(__global tagint *restrict out,
                        const __global tagint *restrict in,
                        int columns_in, int rows_in)
{
  __local tagint block[BLOCK_CELL_2D][BLOCK_CELL_2D+1];

  unsigned ti=THREAD_ID_X;
  unsigned tj=THREAD_ID_Y;
  unsigned bi=BLOCK_ID_X;
  unsigned bj=BLOCK_ID_Y;

  unsigned i=bi*BLOCK_CELL_2D+ti;
  unsigned j=bj*BLOCK_CELL_2D+tj;
  if ((i<columns_in) && (j<rows_in))
    block[tj][ti]=in[j*columns_in+i];

   __syncthreads();

  i=bj*BLOCK_CELL_2D+ti;
  j=bi*BLOCK_CELL_2D+tj;
  if ((i<rows_in) && (j<columns_in))
    out[j*rows_in+i] = block[ti][tj];
}

__kernel void calc_neigh_list_cell(const __global numtyp4 *restrict x_,
                                   const __global int *restrict cell_particle_id,
                                   const __global int *restrict cell_counts,
                                   __global int *nbor_list,
                                   __global int *host_nbor_list,
                                   __global int *host_numj,
                                   int neigh_bin_size, numtyp cell_size,
                                   int ncellx, int ncelly, int ncellz,
                                   int inum, int nt, int nall, int t_per_atom,
                                   int cells_in_cutoff)
{
  int tid = THREAD_ID_X;
  int ix = BLOCK_ID_X + cells_in_cutoff;
  int iy = BLOCK_ID_Y % (ncelly - cells_in_cutoff*2) + cells_in_cutoff;
  int iz = BLOCK_ID_Y / (ncelly - cells_in_cutoff*2) + cells_in_cutoff;
  int bsx = BLOCK_SIZE_X;

  int icell = ix + iy*ncellx + iz*ncellx*ncelly;

  __local int cell_list_sh[BLOCK_NBOR_BUILD];
  __local numtyp4 pos_sh[BLOCK_NBOR_BUILD];

  int icell_begin = cell_counts[icell];
  int icell_end = cell_counts[icell+1];

  int nborz0 = iz-cells_in_cutoff, nborz1 = iz+cells_in_cutoff,
      nbory0 = iy-cells_in_cutoff, nbory1 = iy+cells_in_cutoff,
      nborx0 = ix-cells_in_cutoff, nborx1 = ix+cells_in_cutoff;

  numtyp4 diff;
  numtyp r2;
  int cap=ucl_ceil((numtyp)(icell_end - icell_begin)/bsx);
  for (int ii = 0; ii < cap; ii++) {
    int i = icell_begin + tid + ii*bsx;
    int pid_i = nall, pid_j, stride;
    numtyp4 atom_i, atom_j;
    int cnt = 0;
    __global int *neigh_counts, *neigh_list;

    if (i < icell_end)
      pid_i = cell_particle_id[i];

    if (pid_i < nt) {
      fetch4(atom_i,pid_i,pos_tex); //pos[i];
    }
    if (pid_i < inum) {
      stride=inum;
      neigh_counts=nbor_list+stride+pid_i;
      neigh_list=neigh_counts+stride+pid_i*(t_per_atom-1);
      stride=stride*t_per_atom-t_per_atom;
      nbor_list[pid_i]=pid_i;
    } else {
      stride=0;
      neigh_counts=host_numj+pid_i-inum;
      neigh_list=host_nbor_list+(pid_i-inum)*neigh_bin_size;
    }

    // loop through neighbors

    for (int nborz = nborz0; nborz <= nborz1; nborz++) {
      for (int nbory = nbory0; nbory <= nbory1; nbory++) {
        for (int nborx = nborx0; nborx <= nborx1; nborx++) {

          int jcell = nborx + nbory*ncellx + nborz*ncellx*ncelly;

          int jcell_begin = cell_counts[jcell];
          int jcell_end = cell_counts[jcell+1];
          int num_atom_cell = jcell_end - jcell_begin;

          // load jcell to shared memory
          int num_iter = ucl_ceil((numtyp)num_atom_cell/bsx);

          for (int k = 0; k < num_iter; k++) {
            int end_idx = min(bsx, num_atom_cell-k*bsx);

            if (tid < end_idx) {
              pid_j =  cell_particle_id[tid+k*bsx+jcell_begin];
              cell_list_sh[tid] = pid_j;
              fetch4(atom_j,pid_j,pos_tex); //[pid_j];
              pos_sh[tid].x = atom_j.x;
              pos_sh[tid].y = atom_j.y;
              pos_sh[tid].z = atom_j.z;
            }
            __syncthreads();

            if (pid_i < nt) {

              for (int j = 0; j < end_idx; j++) {
                int pid_j = cell_list_sh[j]; // gather from shared memory
                diff.x = atom_i.x - pos_sh[j].x;
                diff.y = atom_i.y - pos_sh[j].y;
                diff.z = atom_i.z - pos_sh[j].z;

                r2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
                if (r2 < cell_size*cell_size && pid_j != pid_i) { //  && r2 > 1e-5
                  cnt++;
                  if (cnt <= neigh_bin_size) {
                    *neigh_list = pid_j;
                    neigh_list++;
                    if ((cnt & (t_per_atom-1))==0)
                      neigh_list=neigh_list+stride;
                  }
                }
              }
            }
            __syncthreads();
          } // for (k)
        }
      }
    }
    if (pid_i < nt)
      *neigh_counts = cnt;
  } // for (i)
}

__kernel void kernel_special(__global int *dev_nbor,
                             __global int *host_nbor_list,
                             const __global int *host_numj,
                             const __global tagint *restrict tag,
                             const __global int *restrict nspecial,
                             const __global tagint *restrict special,
                             int inum, int nt, int max_nbors, int t_per_atom) {
  int tid=THREAD_ID_X;
  int ii=fast_mul((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom);
  ii+=tid/t_per_atom;
  int offset=tid & (t_per_atom-1);

  if (ii<nt) {
    int stride;
    __global int *list, *list_end;

    int n1=nspecial[ii*3];
    int n2=nspecial[ii*3+1];
    int n3=nspecial[ii*3+2];

    int numj;
    if (ii < inum) {
      stride=inum;
      list=dev_nbor+stride+ii;
      numj=*list;
      list+=stride+fast_mul(ii,t_per_atom-1);
      stride=fast_mul(inum,t_per_atom);
      int njt=numj/t_per_atom;
      list_end=list+fast_mul(njt,stride)+(numj & (t_per_atom-1));
      list+=offset;
    } else {
      stride=1;
      list=host_nbor_list+(ii-inum)*max_nbors;
      numj=host_numj[ii-inum];
      list_end=list+fast_mul(numj,stride);
    }

    for ( ; list<list_end; list+=stride) {
      int nbor=*list;
      tagint jtag=tag[nbor];

      int offset=ii;
      for (int i=0; i<n3; i++) {
        if (special[offset]==jtag) {
          int which = 1;
          if (i>=n1)
            which++;
          if (i>=n2)
            which++;
          nbor=nbor ^ (which << SBBITS);
          *list=nbor;
        }
        offset+=nt;
      }
    }
  } // if ii
}

