#include "hip/hip_runtime.h"
// **************************************************************************
//                                   dpd.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the dpd pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Jan 15, 2014
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> vel_tex;
#else
texture<int4,1> pos_tex;
texture<int4,1> vel_tex;
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#define EPSILON (numtyp)1.0e-10

//#define _USE_UNIFORM_SARU_LCG
//#define _USE_UNIFORM_SARU_TEA8
//#define _USE_GAUSSIAN_SARU_LCG

#if !defined(_USE_UNIFORM_SARU_LCG) && !defined(_USE_UNIFORM_SARU_TEA8) && !defined(_USE_GAUSSIAN_SARU_LCG)
#define _USE_UNIFORM_SARU_LCG
#endif

// References:
// 1. Y. Afshar, F. Schmid, A. Pishevar, S. Worley, Comput. Phys. Comm. 184 (2013), 1119–1128.
// 2. C. L. Phillips, J. A. Anderson, S. C. Glotzer, Comput. Phys. Comm. 230 (2011), 7191-7201.
// PRNG period = 3666320093*2^32 ~ 2^64 ~ 10^19

#define LCGA 0x4beb5d59 /* Full period 32 bit LCG */
#define LCGC 0x2600e1f7
#define oWeylPeriod 0xda879add /* Prime period 3666320093 */
#define oWeylOffset 0x8009d14b
#define TWO_N32 0.232830643653869628906250e-9f /* 2^-32 */

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0]
// using the inherent LCG, then multiply u with sqrt(3) to "match"
// with a normal random distribution.
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
// Curly brackets to make variables local to the scope.
#ifdef _USE_UNIFORM_SARU_LCG
#define SQRT3 (numtyp)1.7320508075688772935274463
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  state = LCGA*state + LCGC;                                                  \
  wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod);   \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0] using TEA8
// then multiply u with sqrt(3) to "match" with a normal random distribution
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
#ifdef _USE_UNIFORM_SARU_TEA8
#define SQRT3 (numtyp)1.7320508075688772935274463
#define k0 0xA341316C
#define k1 0xC8013EA4
#define k2 0xAD90777D
#define k3 0x7E95761E
#define delta 0x9e3779b9
#define rounds 8
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int sum = 0;                                                       \
  for (int i=0; i < rounds; i++) {                                            \
    sum += delta;                                                             \
    state += ((wstate<<4) + k0)^(wstate + sum)^((wstate>>5) + k1);            \
    wstate += ((state<<4) + k2)^(state + sum)^((state>>5) + k3);              \
  }                                                                           \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns two uniformly distributed random numbers r1 and r2 in [-1.0;1.0],
// and uses the polar method (Marsaglia's) to transform to a normal random value
// This is used to compared with CPU DPD using RandMars::gaussian()
#ifdef _USE_GAUSSIAN_SARU_LCG
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state=0x12345678;                                              \
  unsigned int wstate=12345678;                                               \
  state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));                      \
  wstate = (state + seed2) ^ (((signed int)state)>>8);                        \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int v, s;                                                          \
  numtyp r1, r2, rsq;                                                         \
  while (1) {                                                                 \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r1 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r2 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    rsq = r1 * r1 + r2 * r2;                                                  \
    if (rsq < (numtyp)1.0) break;                                             \
  }                                                                           \
  numtyp fac = ucl_sqrt((numtyp)-2.0*log(rsq)/rsq);                           \
  randnum = r2*fac;                                                           \
}
#endif

__kernel void k_dpd(const __global numtyp4 *restrict x_,
                    const __global numtyp4 *restrict coeff,
                    const int lj_types,
                    const __global numtyp *restrict sp_lj,
                    const __global int * dev_nbor,
                    const __global int * dev_packed,
                    __global acctyp4 *restrict ans,
                    __global acctyp *restrict engv,
                    const int eflag, const int vflag, const int inum,
                    const int nbor_pitch,
                    const __global numtyp4 *restrict v_,
                    const __global numtyp *restrict cutsq,
                    const numtyp dtinvsqrt, const int seed,
                    const int timestep, const int tstat_only,
                    const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    numtyp factor_dpd;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;
        numtyp wd = (numtyp)1.0 - r/coeff[mtype].w;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        // conservative force = a0 * wd, or 0 if tstat only
        // drag force = -gamma * wd^2 * (delx dot delv) / r
        // random force = sigma * wd * rnd * dtinvsqrt;

        numtyp force = (numtyp)0.0;
        if (!tstat_only) force = coeff[mtype].x*wd;
        force -= coeff[mtype].y*wd*wd*dot*rinv;
        force += coeff[mtype].z*wd*randnum*dtinvsqrt;
        force*=factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          // unshifted eng of conservative term:
          // evdwl = -a0[itype][jtype]*r * (1.0-0.5*r/cut[itype][jtype]);
          // eng shifted to 0.0 at cutoff
          numtyp e = (numtyp)0.5*coeff[mtype].x*coeff[mtype].w * wd*wd;
          energy+=factor_dpd*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_dpd_fast(const __global numtyp4 *restrict x_,
                         const __global numtyp4 *restrict coeff_in,
                         const __global numtyp *restrict sp_lj_in,
                         const __global int * dev_nbor,
                         const __global int * dev_packed,
                         __global acctyp4 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch,
                         const __global numtyp4 *restrict v_,
                         const __global numtyp *restrict cutsq,
                         const numtyp dtinvsqrt, const int seed,
                         const int timestep, const int tstat_only,
                         const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    numtyp factor_dpd;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;
        numtyp wd = (numtyp)1.0 - r/coeff[mtype].w;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        // conservative force = a0 * wd, or 0 if tstat only
        // drag force = -gamma * wd^2 * (delx dot delv) / r
        // random force = sigma * wd * rnd * dtinvsqrt;

        numtyp force = (numtyp)0.0;
        if (!tstat_only) force = coeff[mtype].x*wd;
        force -= coeff[mtype].y*wd*wd*dot*rinv;
        force += coeff[mtype].z*wd*randnum*dtinvsqrt;
        force*=factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          // unshifted eng of conservative term:
          // evdwl = -a0[itype][jtype]*r * (1.0-0.5*r/cut[itype][jtype]);
          // eng shifted to 0.0 at cutoff
          numtyp e = (numtyp)0.5*coeff[mtype].x*coeff[mtype].w * wd*wd;
          energy+=factor_dpd*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

