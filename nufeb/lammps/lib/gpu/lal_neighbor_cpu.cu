// **************************************************************************
//                                  atom.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for handling CPU generated neighbor lists
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_preprocessor.h"
#endif

__kernel void kernel_unpack(__global int *dev_nbor,
                            const __global int *dev_ij,
                            const int inum, const int t_per_atom) {
  int tid=THREAD_ID_X;
  int offset=tid & (t_per_atom-1);
  int ii=fast_mul((int)BLOCK_ID_X,(int)(BLOCK_SIZE_X)/t_per_atom)+tid/t_per_atom;

  if (ii<inum) {
    int nbor=ii+inum;
    int numj=dev_nbor[nbor];
    nbor+=inum;
    int list=dev_nbor[nbor];
    int list_end=list+numj;
    list+=offset;
    nbor+=fast_mul(ii,t_per_atom-1)+offset;
    int stride=fast_mul(t_per_atom,inum);

    for ( ; list<list_end; list++) {
      dev_nbor[nbor]=dev_ij[list];
      nbor+=stride;
    }
  } // if ii
}

