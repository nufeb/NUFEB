#include "hip/hip_runtime.h"
// **************************************************************************
//                                dipole_lj_sf.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the dipole/sf pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
texture<float4> mu_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
texture<int4,1> mu_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#define mu_tex mu_
#endif

#if (ARCH < 300)

#define store_answers_tq(f, tor, energy, ecoul, virial, ii, inum, tid,      \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[8][BLOCK_PAIR];                                  \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=tor.x;                                                  \
    red_acc[4][tid]=tor.y;                                                  \
    red_acc[5][tid]=tor.z;                                                  \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<6; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    tor.x=red_acc[3][tid];                                                  \
    tor.y=red_acc[4][tid];                                                  \
    tor.z=red_acc[5][tid];                                                  \
    if (eflag>0 || vflag>0) {                                               \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      red_acc[6][tid]=energy;                                               \
      red_acc[7][tid]=ecoul;                                                \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<8; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
      energy=red_acc[6][tid];                                               \
      ecoul=red_acc[7][tid];                                                \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]=energy*(acctyp)0.5;                                          \
      ei+=inum;                                                             \
      engv[ei]=e_coul*(acctyp)0.5;                                          \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }

#else

#define store_answers_tq(f, tor, energy, e_coul, virial, ii, inum, tid,     \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        tor.x += shfl_xor(tor.x, s, t_per_atom);                            \
        tor.y += shfl_xor(tor.y, s, t_per_atom);                            \
        tor.z += shfl_xor(tor.z, s, t_per_atom);                            \
        energy += shfl_xor(energy, s, t_per_atom);                          \
        e_coul += shfl_xor(e_coul, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]=energy*(acctyp)0.5;                                          \
      ei+=inum;                                                             \
      engv[ei]=e_coul*(acctyp)0.5;                                          \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }

#endif

__kernel void k_dipole_lj_sf(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict lj1,
                             const __global numtyp4 *restrict lj3,
                             const int lj_types,
                             const __global numtyp *restrict sp_lj_in,
                             const __global int *dev_nbor,
                             const __global int *dev_packed,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag, const int inum,
                             const int nbor_pitch,
                             const __global numtyp *restrict q_ ,
                             const __global numtyp4 *restrict mu_,
                             const __global numtyp *restrict cutsq,
                             const numtyp qqrd2e, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv;
        numtyp pre1, pre2, pre4;
        numtyp pdotp, pidotr, pjdotr;
        numtyp presf,afac,bfac,pqfac,qpfac,rcutlj2inv,rcutlj6inv,rcutcoul2inv;
        numtyp4 aforcecoul, bforcecoul;

        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          numtyp forceljcut = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;

          rcutlj2inv = ucl_recip(lj1[mtype].z);
          rcutlj6inv = rcutlj2inv * rcutlj2inv * rcutlj2inv;
          numtyp forceljsf = rcutlj6inv*(lj1[mtype].x*rcutlj6inv-lj1[mtype].y)*rcutlj2inv;

          force_lj = factor_lj * (forceljcut - forceljsf);
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);
          rcutcoul2inv = ucl_recip(lj1[mtype].w);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*rinv*(r2inv-rcutcoul2inv);

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;

            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            afac = (numtyp)1.0 - rsq*rsq * rcutcoul2inv*rcutcoul2inv;
            pre1 = afac * (pdotp - (numtyp)3.0*r2inv*pidotr*pjdotr);
            aforcecoul.x = pre1*delx;
            aforcecoul.y = pre1*dely;
            aforcecoul.z = pre1*delz;

            bfac = (numtyp)1.0-(numtyp)4.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv)+
              (numtyp)3.0*rsq*rsq*rcutcoul2inv*rcutcoul2inv;
            presf = (numtyp)2.0*r2inv*pidotr*pjdotr;
            bforcecoul.x = bfac * (pjdotr*mui.x+pidotr*muj.x-presf*delx);
            bforcecoul.y = bfac * (pjdotr*mui.y+pidotr*muj.y-presf*dely);
            bforcecoul.z = bfac * (pjdotr*mui.z+pidotr*muj.z-presf*delz);

            forcecoul.x += (numtyp)3.0*r5inv*(aforcecoul.x + bforcecoul.x);
            forcecoul.y += (numtyp)3.0*r5inv*(aforcecoul.y + bforcecoul.y);
            forcecoul.z += (numtyp)3.0*r5inv*(aforcecoul.z + bforcecoul.z);

            pre2 = (numtyp)3.0*bfac*r5inv*pjdotr;
            pre4 = -bfac*r3inv;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            rcutcoul2inv=ucl_recip(lj1[mtype].w);
            pre1 = (numtyp)3.0*qj*r5inv * pidotr*((numtyp)1.0-rsq*rcutcoul2inv);
            pqfac = (numtyp)1.0 - (numtyp)3.0*rsq*rcutcoul2inv +
              (numtyp)2.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv);
            pre2 = qj*r3inv * pqfac;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;
            rcutcoul2inv=ucl_recip(lj1[mtype].w);
            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr*((numtyp)1.0-rsq*rcutcoul2inv);
            qpfac = (numtyp)1.0 - (numtyp)3.0*rsq*rcutcoul2inv +
              (numtyp)2.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv);
            pre2 = qtmp*r3inv * qpfac;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;
        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (eflag>0) {
          acctyp e = (acctyp)0.0;
          if (rsq < lj1[mtype].w) {
            numtyp fac = (numtyp)1.0-ucl_sqrt(rsq*rcutcoul2inv);
            e = qtmp*qj*rinv*fac*fac;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += bfac* (r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr);
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr * pqfac;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr * qpfac;
              e *= fq;
          } else e = (acctyp)0.0;
          e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y) +
              rcutlj6inv*((numtyp)6.0*lj3[mtype].x*rcutlj6inv -
              (numtyp)3.0*lj3[mtype].y)*rsq*rcutlj2inv +
              rcutlj6inv*((numtyp)(-7.0)*lj3[mtype].x*rcutlj6inv +
              (numtyp)4.0*lj3[mtype].y);
            energy+=factor_lj*e;
          }
        }
        if (vflag>0) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }
    } // for nbor
    store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_dipole_lj_sf_fast(const __global numtyp4 *restrict x_,
                                  const __global numtyp4 *restrict lj1_in,
                                  const __global numtyp4 *restrict lj3_in,
                                  const __global numtyp *restrict sp_lj_in,
                                  const __global int *dev_nbor,
                                  const __global int *dev_packed,
                                  __global acctyp4 *restrict ans,
                                  __global acctyp *restrict engv,
                                  const int eflag, const int vflag,
                                  const int inum, const int nbor_pitch,
                                  const __global numtyp *restrict q_,
                                  const __global numtyp4 *restrict mu_,
                                  const __global numtyp *restrict _cutsq,
                                  const numtyp qqrd2e,
                                  const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    cutsq[tid]=_cutsq[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv;
        numtyp pre1, pre2, pre4;
        numtyp pdotp, pidotr, pjdotr;
        numtyp presf,afac,bfac,pqfac,qpfac,rcutlj2inv,rcutlj6inv,rcutcoul2inv;
        numtyp4 aforcecoul, bforcecoul;

        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          numtyp forceljcut = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;

          rcutlj2inv = ucl_recip(lj1[mtype].z);
          rcutlj6inv = rcutlj2inv * rcutlj2inv * rcutlj2inv;
          numtyp forceljsf = rcutlj6inv*(lj1[mtype].x*rcutlj6inv-lj1[mtype].y)*rcutlj2inv;

          force_lj = factor_lj * (forceljcut - forceljsf);
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);
          rcutcoul2inv = ucl_recip(lj1[mtype].w);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*rinv*(r2inv-rcutcoul2inv);

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;

            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            afac = (numtyp)1.0 - rsq*rsq * rcutcoul2inv*rcutcoul2inv;
            pre1 = afac * (pdotp - (numtyp)3.0*r2inv*pidotr*pjdotr);
            aforcecoul.x = pre1*delx;
            aforcecoul.y = pre1*dely;
            aforcecoul.z = pre1*delz;

            bfac = (numtyp)1.0-(numtyp)4.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv)+
              (numtyp)3.0*rsq*rsq*rcutcoul2inv*rcutcoul2inv;
            presf = (numtyp)2.0*r2inv*pidotr*pjdotr;
            bforcecoul.x = bfac * (pjdotr*mui.x+pidotr*muj.x-presf*delx);
            bforcecoul.y = bfac * (pjdotr*mui.y+pidotr*muj.y-presf*dely);
            bforcecoul.z = bfac * (pjdotr*mui.z+pidotr*muj.z-presf*delz);

            forcecoul.x += (numtyp)3.0*r5inv*(aforcecoul.x + bforcecoul.x);
            forcecoul.y += (numtyp)3.0*r5inv*(aforcecoul.y + bforcecoul.y);
            forcecoul.z += (numtyp)3.0*r5inv*(aforcecoul.z + bforcecoul.z);

            pre2 = (numtyp)3.0*bfac*r5inv*pjdotr;
            pre4 = -bfac*r3inv;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pre1 = (numtyp)3.0*qj*r5inv * pidotr*((numtyp)1.0-rsq*rcutcoul2inv);
            pqfac = (numtyp)1.0 - (numtyp)3.0*rsq*rcutcoul2inv +
              (numtyp)2.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv);
            pre2 = qj*r3inv * pqfac;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr*((numtyp)1.0-rsq*rcutcoul2inv);
            qpfac = (numtyp)1.0 - (numtyp)3.0*rsq*rcutcoul2inv +
              (numtyp)2.0*rsq*ucl_sqrt(rsq)*rcutcoul2inv*ucl_sqrt(rcutcoul2inv);
            pre2 = qtmp*r3inv * qpfac;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;
        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (eflag>0) {
          acctyp e = (acctyp)0.0;
          if (rsq < lj1[mtype].w) {
            numtyp fac = (numtyp)1.0-ucl_sqrt(rsq*rcutcoul2inv);
            e = qtmp*qj*rinv*fac*fac;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += bfac* (r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr);
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr * pqfac;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr * qpfac;
            e *= fq;
          } else e = (acctyp)0.0;
          e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y) +
              rcutlj6inv*((numtyp)6.0*lj3[mtype].x*rcutlj6inv -
              (numtyp)3.0*lj3[mtype].y)*rsq*rcutlj2inv +
              rcutlj6inv*((numtyp)(-7.0)*lj3[mtype].x*rcutlj6inv +
              (numtyp)4.0*lj3[mtype].y);
            energy+=factor_lj*e;
          }
        }
        if (vflag>0) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }

    } // for nbor
    store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

