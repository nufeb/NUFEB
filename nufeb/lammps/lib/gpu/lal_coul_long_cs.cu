#include "hip/hip_runtime.h"
// **************************************************************************
//                               coul_long_cs.cu
//                             -------------------
//                           Trung Nguyen (Northwestern)
//
//  Device code for acceleration of the coul/long/cs pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : June 2018
//    email                : ndactrung@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

// Note: EWALD_P is different from that in lal_preprocessor.h
//       acctyp is needed for these parameters
#define CS_EWALD_P (acctyp)9.95473818e-1
#define B0        (acctyp)-0.1335096380159268
#define B1        (acctyp)-2.57839507e-1
#define B2        (acctyp)-1.37203639e-1
#define B3        (acctyp)-8.88822059e-3
#define B4        (acctyp)-5.80844129e-3
#define B5        (acctyp)1.14652755e-1

#define EPSILON (acctyp)(1.0e-20)
#define EPS_EWALD (acctyp)(1.0e-6)
#define EPS_EWALD_SQR (acctyp)(1.0e-12)

#if (ARCH < 300)

#define store_answers_lq(f, e_coul, virial, ii, inum, tid,                  \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_PAIR];                                  \
                                                                            \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=e_coul;                                                 \
                                                                            \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
                                                                            \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    e_coul=red_acc[3][tid];                                                 \
                                                                            \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
                                                                            \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
                                                                            \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
                                                                            \
  if (offset==0) {                                                          \
    __global acctyp *ap1=engv+ii;                                           \
    if (eflag>0) {                                                          \
      *ap1=(acctyp)0;                                                       \
      ap1+=inum;                                                            \
      *ap1=e_coul*(acctyp)0.5;                                              \
      ap1+=inum;                                                            \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *ap1=virial[i]*(acctyp)0.5;                                         \
        ap1+=inum;                                                          \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#else

#define store_answers_lq(f, e_coul, virial, ii, inum, tid,                  \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        e_coul += shfl_xor(e_coul, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    __global acctyp *ap1=engv+ii;                                           \
    if (eflag>0) {                                                          \
      *ap1=(acctyp)0;                                                       \
      ap1+=inum;                                                            \
      *ap1=e_coul*(acctyp)0.5;                                              \
      ap1+=inum;                                                            \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *ap1=virial[i]*(acctyp)0.5;                                         \
        ap1+=inum;                                                          \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#endif

__kernel void k_coul_long_cs(const __global numtyp4 *restrict x_,
                          const __global numtyp *restrict scale,
                          const int lj_types,
                          const __global numtyp *restrict sp_cl_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const numtyp cut_coulsq, const numtyp qqrd2e,
                          const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_cl[4];
  sp_cl[0]=sp_cl_in[0];
  sp_cl[1]=sp_cl_in[1];
  sp_cl[2]=sp_cl_in[2];
  sp_cl[3]=sp_cl_in[3];

  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp qtmp; fetch(qtmp,i,q_tex);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq < cut_coulsq) {
        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;

        numtyp force,prefactor,_erfc;
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_rsqrt(r2inv);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp;
        if (factor_coul<(numtyp)1.0) {
          numtyp grij = g_ewald * (r+EPS_EWALD);
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= (r+EPS_EWALD);
          force = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
          // Additionally r2inv needs to be accordingly modified since the later
          // scaling of the overall force shall be consistent
          r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
          force *= r2inv;
        } else {
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= r;
          force = prefactor*(_erfc + EWALD_F*grij*expm2);
          force *= r2inv;
        }

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e = prefactor*_erfc;
          if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
          e_coul += e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_lq(f,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                     vflag,ans,engv);
  } // if ii
}

__kernel void k_coul_long_cs_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp *restrict scale_in,
                               const __global numtyp *restrict sp_cl_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp scale[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_cl[4];
  if (tid<4)
    sp_cl[tid]=sp_cl_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES)
    scale[tid]=scale_in[tid];

  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;

        numtyp force,prefactor,_erfc;
        numtyp r2inv = ucl_recip(rsq);
        numtyp r = ucl_rsqrt(r2inv);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp;
        if (factor_coul<(numtyp)1.0) {
          numtyp grij = g_ewald * (r+EPS_EWALD);
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= (r+EPS_EWALD);
          force = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
          // Additionally r2inv needs to be accordingly modified since the later
          // scaling of the overall force shall be consistent
          r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
        } else {
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
          numtyp u = (numtyp)1.0 - t;
          _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
          prefactor /= r;
          force = prefactor * (_erfc + EWALD_F*grij*expm2);
        }

        force *= r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e = prefactor*_erfc;
          if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
          e_coul += e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_lq(f,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                     vflag,ans,engv);
  } // if ii
}

