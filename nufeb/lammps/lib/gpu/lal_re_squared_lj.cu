#include "hip/hip_runtime.h"
// **************************************************************************
//                               re_squared_lj.cu
//                             -------------------
//                               W. Michael Brown
//
//  Device code for RE-Squared - Lennard-Jones potential acceleration
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Fri May 06 2011
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_ellipsoid_extra.h"
#endif

#if (ARCH < 300)

#define store_answers_rt(f, tor, energy, virial, ii, astride, tid,           \
                         t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                        \
    __local acctyp red_acc[7][BLOCK_PAIR];                                   \
    red_acc[0][tid]=f.x;                                                     \
    red_acc[1][tid]=f.y;                                                     \
    red_acc[2][tid]=f.z;                                                     \
    red_acc[3][tid]=tor.x;                                                   \
    red_acc[4][tid]=tor.y;                                                   \
    red_acc[5][tid]=tor.z;                                                   \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                          \
      if (offset < s) {                                                      \
        for (int r=0; r<6; r++)                                              \
          red_acc[r][tid] += red_acc[r][tid+s];                              \
      }                                                                      \
    }                                                                        \
    f.x=red_acc[0][tid];                                                     \
    f.y=red_acc[1][tid];                                                     \
    f.z=red_acc[2][tid];                                                     \
    tor.x=red_acc[3][tid];                                                   \
    tor.y=red_acc[4][tid];                                                   \
    tor.z=red_acc[5][tid];                                                   \
    if (eflag>0 || vflag>0) {                                                \
      for (int r=0; r<6; r++)                                                \
        red_acc[r][tid]=virial[r];                                           \
      red_acc[6][tid]=energy;                                                \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                        \
        if (offset < s) {                                                    \
          for (int r=0; r<7; r++)                                            \
            red_acc[r][tid] += red_acc[r][tid+s];                            \
        }                                                                    \
      }                                                                      \
      for (int r=0; r<6; r++)                                                \
        virial[r]=red_acc[r][tid];                                           \
      energy=red_acc[6][tid];                                                \
    }                                                                        \
  }                                                                          \
  if (offset==0) {                                                           \
    __global acctyp *ap1=engv+ii;                                            \
    if (eflag>0) {                                                           \
      *ap1+=energy*(acctyp)0.5;                                              \
      ap1+=astride;                                                          \
    }                                                                        \
    if (vflag>0) {                                                           \
      for (int i=0; i<6; i++) {                                              \
        *ap1+=virial[i]*(acctyp)0.5;                                         \
        ap1+=astride;                                                        \
      }                                                                      \
    }                                                                        \
    acctyp4 old=ans[ii];                                                     \
    old.x+=f.x;                                                              \
    old.y+=f.y;                                                              \
    old.z+=f.z;                                                              \
    ans[ii]=old;                                                             \
    old=ans[ii+astride];                                                     \
    old.x+=tor.x;                                                            \
    old.y+=tor.y;                                                            \
    old.z+=tor.z;                                                            \
    ans[ii+astride]=old;                                                     \
  }

#else

#define store_answers_rt(f, tor, energy, virial, ii, astride, tid,          \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        tor.x += shfl_xor(tor.x, s, t_per_atom);                            \
        tor.y += shfl_xor(tor.y, s, t_per_atom);                            \
        tor.z += shfl_xor(tor.z, s, t_per_atom);                            \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    __global acctyp *ap1=engv+ii;                                           \
    if (eflag>0) {                                                          \
      *ap1+=energy*(acctyp)0.5;                                             \
      ap1+=astride;                                                         \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *ap1+=virial[i]*(acctyp)0.5;                                        \
        ap1+=astride;                                                       \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
    old=ans[ii+astride];                                                    \
    old.x+=tor.x;                                                           \
    old.y+=tor.y;                                                           \
    old.z+=tor.z;                                                           \
    ans[ii+astride]=old;                                                    \
  }

#endif

__kernel void k_resquared_ellipsoid_sphere(const __global numtyp4 *restrict x_,
                                           const __global numtyp4 *restrict q,
                                           const __global numtyp4 *restrict shape,
                                           const __global numtyp4 *restrict well,
                                           const __global numtyp *restrict splj,
                                           const __global numtyp2 *restrict sig_eps,
                                           const int ntypes,
                                           const __global int *dev_nbor,
                                           const int stride,
                                           __global acctyp4 *restrict ans,
                                           const int astride,
                                           __global acctyp *restrict engv,
                                           __global int *restrict err_flag,
                                           const int eflag, const int vflag,
                                           const int inum,
                                           const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=splj[0];
  sp_lj[1]=splj[1];
  sp_lj[2]=splj[2];
  sp_lj[3]=splj[3];

  __local numtyp b_alpha, cr60, solv_f_a, solv_f_r;
  b_alpha=(numtyp)45.0/(numtyp)56.0;
  cr60=ucl_cbrt((numtyp)60.0);
  solv_f_a = (numtyp)3.0/((numtyp)16.0*ucl_atan((numtyp)1.0)*-(numtyp)36.0);
  solv_f_r = (numtyp)3.0/((numtyp)16.0*ucl_atan((numtyp)1.0)*(numtyp)2025.0);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_nbor,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int itype=ix.w;

    numtyp a[9];       // Rotation matrix (lab->body)
    numtyp aTe[9];     // A'*E
    numtyp lA_0[9], lA_1[9], lA_2[9]; // -A*rotation generator (x,y, or z)

    numtyp4 ishape;
    ishape=shape[itype];
    numtyp ilshape=ishape.x*ishape.y*ishape.z;

    {
      gpu_quat_to_mat_trans(q,i,a);
      gpu_transpose_times_diag3(a,well[itype],aTe);
      gpu_rotation_generator_x(a,lA_0);
      gpu_rotation_generator_y(a,lA_1);
      gpu_rotation_generator_z(a,lA_2);
    }

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_nbor[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int jtype=jx.w;

      // Compute r12
      numtyp r[3], rhat[3];
      numtyp rnorm;
      r[0] = jx.x-ix.x;
      r[1] = jx.y-ix.y;
      r[2] = jx.z-ix.z;
      rnorm = gpu_dot3(r,r);
      rnorm = ucl_rsqrt(rnorm);
      rhat[0] = r[0]*rnorm;
      rhat[1] = r[1]*rnorm;
      rhat[2] = r[2]*rnorm;

      numtyp sigma, epsilon;
      int mtype=fast_mul(ntypes,itype)+jtype;
      sigma = sig_eps[mtype].x;
      epsilon = sig_eps[mtype].y*factor_lj;

      numtyp aTs[9];
      numtyp4 scorrect;
      numtyp half_sigma=sigma*(numtyp)0.5;
      scorrect.x = ishape.x+half_sigma;
      scorrect.y = ishape.y+half_sigma;
      scorrect.z = ishape.z+half_sigma;
      scorrect.x = scorrect.x * scorrect.x * (numtyp)0.5;
      scorrect.y = scorrect.y * scorrect.y * (numtyp)0.5;
      scorrect.z = scorrect.z * scorrect.z * (numtyp)0.5;
      gpu_transpose_times_diag3(a,scorrect,aTs);

      // energy

      numtyp gamma[9], s[3];
      gpu_times3(aTs,a,gamma);
      gpu_mldivide3(gamma,rhat,s,err_flag);

      numtyp sigma12 = ucl_rsqrt((numtyp)0.5*gpu_dot3(s,rhat));
      numtyp temp[9], w[3];
      gpu_times3(aTe,a,temp);
      temp[0] += (numtyp)1.0;
      temp[4] += (numtyp)1.0;
      temp[8] += (numtyp)1.0;
      gpu_mldivide3(temp,rhat,w,err_flag);

      numtyp h12 = ucl_recip(rnorm)-sigma12;
      numtyp chi = (numtyp)2.0*gpu_dot3(rhat,w);
      numtyp sigh = sigma/h12;
      numtyp tprod = chi*sigh;

      numtyp Ua, Ur;
      numtyp h12p3 = h12*h12*h12;
      numtyp sigmap3 = sigma*sigma*sigma;
      numtyp stemp = h12*(numtyp)0.5;
      Ua = (ishape.x+stemp)*(ishape.y+stemp)*(ishape.z+stemp)*h12p3/(numtyp)8.0;
      Ua = ((numtyp)1.0+(numtyp)3.0*tprod)*ilshape/Ua;
      Ua = epsilon*Ua*sigmap3*solv_f_a;

      stemp = h12/cr60;
      Ur = (ishape.x+stemp)*(ishape.y+stemp)*(ishape.z+stemp)*h12p3/
           (numtyp)60.0;
      Ur = ((numtyp)1.0+b_alpha*tprod)*ilshape/Ur;
      numtyp sigh6=sigh*sigh*sigh;
      sigh6*=sigh6;
      Ur = epsilon*Ur*sigmap3*sigh6*solv_f_r;

      energy+=Ua+Ur;

      // force

      numtyp fourw[3], spr[3];
      numtyp sec = sigma*chi;
      numtyp sigma12p3 = sigma12*sigma12*sigma12;
      fourw[0] = (numtyp)4.0*w[0];
      fourw[1] = (numtyp)4.0*w[1];
      fourw[2] = (numtyp)4.0*w[2];
      spr[0] = (numtyp)0.5*sigma12p3*s[0];
      spr[1] = (numtyp)0.5*sigma12p3*s[1];
      spr[2] = (numtyp)0.5*sigma12p3*s[2];

      stemp = ucl_recip(ishape.x*(numtyp)2.0+h12)+
              ucl_recip(ishape.y*(numtyp)2.0+h12)+
              ucl_recip(ishape.z*(numtyp)2.0+h12)+
              (numtyp)3.0/h12;
      numtyp hsec = ucl_recip(h12+(numtyp)3.0*sec);
      numtyp dspu = ucl_recip(h12)-hsec+stemp;
      numtyp pbsu = (numtyp)3.0*sigma*hsec;

      stemp = ucl_recip(ishape.x*cr60+h12)+
              ucl_recip(ishape.y*cr60+h12)+
              ucl_recip(ishape.z*cr60+h12)+
              (numtyp)3.0/h12;
      hsec = ucl_recip(h12+b_alpha*sec);
      numtyp dspr = (numtyp)7.0/h12-hsec+stemp;
      numtyp pbsr = b_alpha*sigma*hsec;

      #pragma unroll
      for (int i=0; i<3; i++) {
        numtyp u[3];
        u[0] = -rhat[i]*rhat[0];
        u[1] = -rhat[i]*rhat[1];
        u[2] = -rhat[i]*rhat[2];
        u[i] += (numtyp)1.0;
        u[0] *= rnorm;
        u[1] *= rnorm;
        u[2] *= rnorm;
        numtyp dchi = gpu_dot3(u,fourw);
        numtyp dh12 = rhat[i]+gpu_dot3(u,spr);
        numtyp dUa = pbsu*dchi-dh12*dspu;
        numtyp dUr = pbsr*dchi-dh12*dspr;
        numtyp force=dUr*Ur+dUa*Ua;
        if (i==0) {
          f.x+=force;
          if (vflag>0)
            virial[0]+=-r[0]*force;
        } else if (i==1) {
          f.y+=force;
          if (vflag>0) {
            virial[1]+=-r[1]*force;
            virial[3]+=-r[0]*force;
          }
        } else {
          f.z+=force;
          if (vflag>0) {
            virial[2]+=-r[2]*force;
            virial[4]+=-r[0]*force;
            virial[5]+=-r[1]*force;
          }
        }

      }

      // torque on i
      numtyp fwae[3];
      gpu_row_times3(fourw,aTe,fwae);
      {
        numtyp tempv[3], p[3], lAtwo[9];
        gpu_times_column3(lA_0,rhat,p);
        gpu_times_column3(lA_0,w,tempv);
        numtyp dchi = -gpu_dot3(fwae,tempv);
        gpu_times3(aTs,lA_0,lAtwo);
        gpu_times_column3(lAtwo,spr,tempv);
        numtyp dh12 = -gpu_dot3(s,tempv);
        numtyp dUa = pbsu*dchi-dh12*dspu;
        numtyp dUr = pbsr*dchi-dh12*dspr;
        tor.x -= (dUa*Ua+dUr*Ur);
      }

      {
        numtyp tempv[3], p[3], lAtwo[9];
        gpu_times_column3(lA_1,rhat,p);
        gpu_times_column3(lA_1,w,tempv);
        numtyp dchi = -gpu_dot3(fwae,tempv);
        gpu_times3(aTs,lA_1,lAtwo);
        gpu_times_column3(lAtwo,spr,tempv);
        numtyp dh12 = -gpu_dot3(s,tempv);
        numtyp dUa = pbsu*dchi-dh12*dspu;
        numtyp dUr = pbsr*dchi-dh12*dspr;
        tor.y -= (dUa*Ua+dUr*Ur);
      }

      {
        numtyp tempv[3], p[3], lAtwo[9];
        gpu_times_column3(lA_2,rhat,p);
        gpu_times_column3(lA_2,w,tempv);
        numtyp dchi = -gpu_dot3(fwae,tempv);
        gpu_times3(aTs,lA_2,lAtwo);
        gpu_times_column3(lAtwo,spr,tempv);
        numtyp dh12 = -gpu_dot3(s,tempv);
        numtyp dUa = pbsu*dchi-dh12*dspu;
        numtyp dUr = pbsr*dchi-dh12*dspr;
        tor.z -= (dUa*Ua+dUr*Ur);
      }

    } // for nbor
    store_answers_rt(f,tor,energy,virial,ii,astride,tid,t_per_atom,offset,eflag,
                     vflag,ans,engv);
  } // if ii
}

__kernel void k_resquared_sphere_ellipsoid(const __global numtyp4 *restrict x_,
                                           const __global numtyp4 *restrict q,
                                           const __global numtyp4 *restrict shape,
                                           const __global numtyp4 *restrict well,
                                           const __global numtyp *restrict splj,
                                           const __global numtyp2 *restrict sig_eps,
                                           const int ntypes,
                                           const __global int *dev_nbor,
                                           const int stride,
                                           __global acctyp4 *restrict ans,
                                           __global acctyp *restrict engv,
                                           __global int *restrict err_flag,
                                           const int eflag, const int vflag,
                                           const int start, const int inum,
                                           const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  sp_lj[0]=splj[0];
  sp_lj[1]=splj[1];
  sp_lj[2]=splj[2];
  sp_lj[3]=splj[3];

  __local numtyp b_alpha, cr60, solv_f_a, solv_f_r;
  b_alpha=(numtyp)45.0/(numtyp)56.0;
  cr60=ucl_cbrt((numtyp)60.0);
  solv_f_a = (numtyp)3.0/((numtyp)16.0*ucl_atan((numtyp)1.0)*-(numtyp)36.0);
  solv_f_r = (numtyp)3.0/((numtyp)16.0*ucl_atan((numtyp)1.0)*(numtyp)2025.0);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int j, numj;
    __local int n_stride;
    nbor_info_e(dev_nbor,stride,t_per_atom,ii,offset,j,numj,
                n_stride,nbor_end,nbor);

    numtyp4 jx; fetch4(jx,j,pos_tex);
    int jtype=jx.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int i=dev_nbor[nbor];
      factor_lj = sp_lj[sbmask(i)];
      i &= NEIGHMASK;

      numtyp4 ix; fetch4(ix,i,pos_tex);
      int itype=ix.w;

      numtyp a[9];       // Rotation matrix (lab->body)
      numtyp aTe[9];     // A'*E
      numtyp4 ishape;

      ishape=shape[itype];
      gpu_quat_to_mat_trans(q,i,a);
      gpu_transpose_times_diag3(a,well[itype],aTe);

      // Compute r12
      numtyp r[3], rhat[3];
      numtyp rnorm;
      r[0] = ix.x-jx.x;
      r[1] = ix.y-jx.y;
      r[2] = ix.z-jx.z;
      rnorm = gpu_dot3(r,r);
      rnorm = ucl_rsqrt(rnorm);
      rhat[0] = r[0]*rnorm;
      rhat[1] = r[1]*rnorm;
      rhat[2] = r[2]*rnorm;

      numtyp sigma, epsilon;
      int mtype=fast_mul(ntypes,itype)+jtype;
      sigma = sig_eps[mtype].x;
      epsilon = sig_eps[mtype].y*factor_lj;

      numtyp aTs[9];
      numtyp4 scorrect;
      numtyp half_sigma=sigma * (numtyp)0.5;
      scorrect.x = ishape.x+half_sigma;
      scorrect.y = ishape.y+half_sigma;
      scorrect.z = ishape.z+half_sigma;
      scorrect.x = scorrect.x * scorrect.x * (numtyp)0.5;
      scorrect.y = scorrect.y * scorrect.y * (numtyp)0.5;
      scorrect.z = scorrect.z * scorrect.z * (numtyp)0.5;
      gpu_transpose_times_diag3(a,scorrect,aTs);

      // energy

      numtyp gamma[9], s[3];
      gpu_times3(aTs,a,gamma);
      gpu_mldivide3(gamma,rhat,s,err_flag);

      numtyp sigma12 = ucl_rsqrt((numtyp)0.5*gpu_dot3(s,rhat));
      numtyp temp[9], w[3];
      gpu_times3(aTe,a,temp);
      temp[0] += (numtyp)1.0;
      temp[4] += (numtyp)1.0;
      temp[8] += (numtyp)1.0;
      gpu_mldivide3(temp,rhat,w,err_flag);

      numtyp h12 = ucl_recip(rnorm)-sigma12;
      numtyp chi = (numtyp)2.0*gpu_dot3(rhat,w);
      numtyp sigh = sigma/h12;
      numtyp tprod = chi*sigh;

      numtyp Ua, Ur;
      numtyp h12p3 = h12*h12*h12;
      numtyp sigmap3 = sigma*sigma*sigma;
      numtyp stemp = h12/(numtyp)2.0;
      Ua = (ishape.x+stemp)*(ishape.y+stemp)*(ishape.z+stemp)*h12p3/(numtyp)8.0;
      numtyp ilshape=ishape.x*ishape.y*ishape.z;
      Ua = ((numtyp)1.0+(numtyp)3.0*tprod)*ilshape/Ua;
      Ua = epsilon*Ua*sigmap3*solv_f_a;

      stemp = h12/cr60;
      Ur = (ishape.x+stemp)*(ishape.y+stemp)*(ishape.z+stemp)*h12p3/
           (numtyp)60.0;
      Ur = ((numtyp)1.0+b_alpha*tprod)*ilshape/Ur;
      numtyp sigh6=sigh*sigh*sigh;
      sigh6*=sigh6;
      Ur = epsilon*Ur*sigmap3*sigh6*solv_f_r;

      energy+=Ua+Ur;

      // force

      numtyp fourw[3], spr[3];
      numtyp sec = sigma*chi;
      numtyp sigma12p3 = sigma12*sigma12*sigma12;
      fourw[0] = (numtyp)4.0*w[0];
      fourw[1] = (numtyp)4.0*w[1];
      fourw[2] = (numtyp)4.0*w[2];
      spr[0] = (numtyp)0.5*sigma12p3*s[0];
      spr[1] = (numtyp)0.5*sigma12p3*s[1];
      spr[2] = (numtyp)0.5*sigma12p3*s[2];

      stemp = ucl_recip(ishape.x*(numtyp)2.0+h12)+
              ucl_recip(ishape.y*(numtyp)2.0+h12)+
              ucl_recip(ishape.z*(numtyp)2.0+h12)+
              (numtyp)3.0/h12;
      numtyp hsec = ucl_recip(h12+(numtyp)3.0*sec);
      numtyp dspu = ucl_recip(h12)-hsec+stemp;
      numtyp pbsu = (numtyp)3.0*sigma*hsec;

      stemp = ucl_recip(ishape.x*cr60+h12)+
              ucl_recip(ishape.y*cr60+h12)+
              ucl_recip(ishape.z*cr60+h12)+
              (numtyp)3.0/h12;
      hsec = ucl_recip(h12+b_alpha*sec);
      numtyp dspr = (numtyp)7.0/h12-hsec+stemp;
      numtyp pbsr = b_alpha*sigma*hsec;

      #pragma unroll
      for (int i=0; i<3; i++) {
        numtyp u[3];
        u[0] = -rhat[i]*rhat[0];
        u[1] = -rhat[i]*rhat[1];
        u[2] = -rhat[i]*rhat[2];
        u[i] += (numtyp)1.0;
        u[0] *= rnorm;
        u[1] *= rnorm;
        u[2] *= rnorm;
        numtyp dchi = gpu_dot3(u,fourw);
        numtyp dh12 = rhat[i]+gpu_dot3(u,spr);
        numtyp dUa = pbsu*dchi-dh12*dspu;
        numtyp dUr = pbsr*dchi-dh12*dspr;
        numtyp force=dUr*Ur+dUa*Ua;
        if (i==0) {
          f.x+=force;
          if (vflag>0)
            virial[0]+=-r[0]*force;
        } else if (i==1) {
          f.y+=force;
          if (vflag>0) {
            virial[1]+=-r[1]*force;
            virial[3]+=-r[0]*force;
          }
        } else {
          f.z+=force;
          if (vflag>0) {
            virial[2]+=-r[2]*force;
            virial[4]+=-r[0]*force;
            virial[5]+=-r[1]*force;
          }
        }
      }
    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_resquared_lj(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict lj1,
                             const __global numtyp4 *restrict lj3,
                             const int lj_types,
                             const __global numtyp *restrict gum,
                             const int stride,
                             const __global int *dev_ij,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             __global int *restrict err_flag,
                             const int eflag, const int vflag, const int start,
                             const int inum, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  sp_lj[0]=gum[0];
  sp_lj[1]=gum[1];
  sp_lj[2]=gum[2];
  sp_lj[3]=gum[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_ij,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_ij[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      int ii=itype*lj_types+jtype;
      if (r2inv<lj1[ii].z && lj1[ii].w==SPHERE_SPHERE) {
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*r6inv*(lj1[ii].x*r6inv-lj1[ii].y);
        force*=factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[ii].x*r6inv-lj3[ii].y);
          energy+=factor_lj*(e-lj3[ii].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
    acc_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  } // if ii
}

__kernel void k_resquared_lj_fast(const __global numtyp4 *restrict x_,
                                  const __global numtyp4 *restrict lj1_in,
                                  const __global numtyp4 *restrict lj3_in,
                                  const __global numtyp *restrict gum,
                                  const int stride,
                                  const __global int *dev_ij,
                                  __global acctyp4 *restrict ans,
                                  __global acctyp *restrict engv,
                                  __global int *restrict err_flag,
                                  const int eflag, const int vflag,
                                  const int start, const int inum,
                                  const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<4)
    sp_lj[tid]=gum[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_ij,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_ij[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<lj1[mtype].z && lj1[mtype].w==SPHERE_SPHERE) {
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = factor_lj*r2inv*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    acc_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  } // if ii
}

