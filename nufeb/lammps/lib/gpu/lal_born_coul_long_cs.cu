#include "hip/hip_runtime.h"
// **************************************************************************
//                            born_coul_long_cs.cu
//                             -------------------
//                           Trung Dac Nguyen (Northwestern)
//
//  Device code for acceleration of the born/coul/long/cs pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : June 2018
//    email                : ndactrung@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

// Note: EWALD_P is different from that in lal_preprocessor.h
//       acctyp is needed for these parameters
#define CS_EWALD_P (acctyp)9.95473818e-1
#define B0        (acctyp)-0.1335096380159268
#define B1        (acctyp)-2.57839507e-1
#define B2        (acctyp)-1.37203639e-1
#define B3        (acctyp)-8.88822059e-3
#define B4        (acctyp)-5.80844129e-3
#define B5        (acctyp)1.14652755e-1

#define EPSILON (acctyp)(1.0e-20)
#define EPS_EWALD (acctyp)(1.0e-6)
#define EPS_EWALD_SQR (acctyp)(1.0e-12)

__kernel void k_born_coul_long_cs(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict coeff1,
                          const __global numtyp4 *restrict coeff2,
                          const int lj_types,
                          const __global numtyp *restrict sp_lj_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const __global numtyp4 *restrict cutsq_sigma,
                          const numtyp cut_coulsq, const numtyp qqrd2e,
                          const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq_sigma[mtype].x) { // cutsq 
        numtyp forcecoul,forceborn,force,r6inv,prefactor,_erfc,rexp;

        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;
        numtyp r2inv = ucl_recip(rsq);

        if (rsq < cut_coulsq) {
          numtyp r = ucl_sqrt(rsq);
          fetch(prefactor,j,q_tex);
          prefactor *= qqrd2e * qtmp;
          if (factor_coul<(numtyp)1.0) {
            numtyp grij = g_ewald * (r+EPS_EWALD);
            numtyp expm2 = ucl_exp(-grij*grij);
            acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
            numtyp u = (numtyp)1.0 - t;
            _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
            prefactor /= (r+EPS_EWALD);
            forcecoul = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
            // Additionally r2inv needs to be accordingly modified since the later
            // scaling of the overall force shall be consistent
            r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
          } else {
            numtyp grij = g_ewald * r;
            numtyp expm2 = ucl_exp(-grij*grij);
            acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
            numtyp u = (numtyp)1.0 - t;
            _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
            prefactor /= r;
            forcecoul = prefactor*(_erfc + EWALD_F*grij*expm2);
          }
        } else forcecoul = (numtyp)0.0;

        if (rsq < cutsq_sigma[mtype].y) { // cut_ljsq
          numtyp r = ucl_sqrt(rsq);
          rexp = ucl_exp((cutsq_sigma[mtype].z-r)*coeff1[mtype].x);
          r6inv = r2inv*r2inv*r2inv;
          forceborn = (coeff1[mtype].y*r*rexp - coeff1[mtype].z*r6inv
            + coeff1[mtype].w*r2inv*r6inv)*factor_lj;
        } else forceborn = (numtyp)0.0;

        force = (forcecoul + forceborn) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq) {
            numtyp e = prefactor*_erfc;
            if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
            e_coul += e;
          }
          if (rsq < cutsq_sigma[mtype].y) {
            numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv
              + coeff2[mtype].z*r2inv*r6inv;
            energy+=factor_lj*(e-coeff2[mtype].w);
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_born_coul_long_cs_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp4 *restrict coeff1_in,
                               const __global numtyp4 *restrict coeff2_in,
                               const __global numtyp *restrict sp_lj_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const __global numtyp4 *restrict cutsq_sigma,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff1[tid]=coeff1_in[tid];
    if (eflag>0)
      coeff2[tid]=coeff2_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_sigma[mtype].x) { // cutsq 
        numtyp forcecoul,forceborn,force,r6inv,prefactor,_erfc,rexp;

        rsq += EPSILON; // Add Epsilon for case: r = 0; Interaction must be removed by special bond;
        numtyp r2inv = ucl_recip(rsq);

        if (rsq < cut_coulsq) {
          numtyp r = ucl_sqrt(rsq);
          fetch(prefactor,j,q_tex);
          prefactor *= qqrd2e * qtmp;
          if (factor_coul<(numtyp)1.0) {
            numtyp grij = g_ewald * (r+EPS_EWALD);
            numtyp expm2 = ucl_exp(-grij*grij);
            acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
            numtyp u = (numtyp)1.0 - t;
            _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
            prefactor /= (r+EPS_EWALD);
            forcecoul = prefactor * (_erfc + EWALD_F*grij*expm2 - ((numtyp)1.0-factor_coul));
            // Additionally r2inv needs to be accordingly modified since the later
            // scaling of the overall force shall be consistent
            r2inv = ucl_recip(rsq + EPS_EWALD_SQR);
          } else {
            numtyp grij = g_ewald * r;
            numtyp expm2 = ucl_exp(-grij*grij);
            acctyp t = ucl_recip((numtyp)1.0 + CS_EWALD_P*grij);
            numtyp u = (numtyp)1.0 - t;
            _erfc = t * ((numtyp)1.0 + u*(B0+u*(B1+u*(B2+u*(B3+u*(B4+u*B5)))))) * expm2;
            prefactor /= r;
            forcecoul = prefactor*(_erfc + EWALD_F*grij*expm2);
          }
        } else forcecoul = (numtyp)0.0;

        if (rsq < cutsq_sigma[mtype].y) { // cut_ljsq
          numtyp r = ucl_sqrt(rsq);
          rexp = ucl_exp((cutsq_sigma[mtype].z-r)*coeff1[mtype].x);
          r6inv = r2inv*r2inv*r2inv;
          forceborn = (coeff1[mtype].y*r*rexp - coeff1[mtype].z*r6inv
            + coeff1[mtype].w*r2inv*r6inv)*factor_lj;
        } else forceborn = (numtyp)0.0;

        force = (forcecoul + forceborn) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq) {
            numtyp e = prefactor*_erfc;
            if (factor_coul<(numtyp)1.0) e -= ((numtyp)1.0-factor_coul)*prefactor;
            e_coul += e;
          }
          if (rsq < cutsq_sigma[mtype].y) {
            numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv
              + coeff2[mtype].z*r2inv*r6inv;
            energy+=factor_lj*(e-coeff2[mtype].w);
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

