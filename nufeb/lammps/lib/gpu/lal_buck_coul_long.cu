#include "hip/hip_runtime.h"
// **************************************************************************
//                              buck_coul_long.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the buck/coul/long pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

__kernel void k_buck_coul_long(const __global numtyp4 *restrict x_,
                               const __global numtyp4 *restrict coeff1,
                               const __global numtyp4 *restrict coeff2,
                               const int lj_types,
                               const __global numtyp *restrict sp_lj_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const __global numtyp *restrict cutsq,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force_lj, force, r6inv, prefactor, _erfc;
        numtyp rexp = (numtyp)0.0;

        if (rsq < coeff1[mtype].w) { // cut_ljsq
          numtyp r=ucl_sqrt(rsq);
          rexp = ucl_exp(-r*coeff1[mtype].x);
          r6inv = r2inv*r2inv*r2inv;
          force_lj = (coeff1[mtype].y*r*rexp
                  - coeff1[mtype].z*r6inv)*factor_lj;
        } else
          force_lj = (numtyp)0.0;

        if (rsq < cut_coulsq) {
          numtyp r = ucl_rsqrt(r2inv);
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
          fetch(prefactor,j,q_tex);
          prefactor *= qqrd2e * qtmp/r;
          forcecoul = prefactor * (_erfc + EWALD_F*grij*expm2-factor_coul);
        } else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq)
            e_coul += prefactor*(_erfc-factor_coul);
          if (rsq < coeff1[mtype].w) {
            numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv;
            energy+=factor_lj*(e-coeff2[mtype].z);
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_buck_coul_long_fast(const __global numtyp4 *restrict x_,
                                    const __global numtyp4 *restrict coeff1_in,
                                    const __global numtyp4 *restrict coeff2_in,
                                    const __global numtyp *restrict sp_lj_in,
                                    const __global int *dev_nbor,
                                    const __global int *dev_packed,
                                    __global acctyp4 *restrict ans,
                                    __global acctyp *restrict engv,
                                    const int eflag, const int vflag,
                                    const int inum, const int nbor_pitch,
                                    const __global numtyp *restrict q_,
                                    const __global numtyp *restrict cutsq,
                                    const numtyp cut_coulsq,
                                    const numtyp qqrd2e, const numtyp g_ewald,
                                    const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff1[tid]=coeff1_in[tid];
    if (eflag>0)
      coeff2[tid]=coeff2_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force_lj, force, r6inv, prefactor, _erfc;
        numtyp rexp = (numtyp)0.0;

        if (rsq < coeff1[mtype].w) {
          numtyp r=ucl_sqrt(rsq);
          rexp = ucl_exp(-r*coeff1[mtype].x);
          r6inv = r2inv*r2inv*r2inv;
          force_lj = (coeff1[mtype].y*r*rexp
                  - coeff1[mtype].z*r6inv)*factor_lj;
        } else
          force_lj = (numtyp)0.0;

        if (rsq < cut_coulsq) {
          numtyp r = ucl_rsqrt(r2inv);
          numtyp grij = g_ewald * r;
          numtyp expm2 = ucl_exp(-grij*grij);
          numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
          _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
          fetch(prefactor,j,q_tex);
          prefactor *= qqrd2e * qtmp/r;
          forcecoul = prefactor * (_erfc + EWALD_F*grij*expm2-factor_coul);
        } else
          forcecoul = (numtyp)0.0;

        force = (force_lj + forcecoul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq)
            e_coul += prefactor*(_erfc-factor_coul);
          if (rsq < coeff1[mtype].w) {
            numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv;
            energy+=factor_lj*(e-coeff2[mtype].z);
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

