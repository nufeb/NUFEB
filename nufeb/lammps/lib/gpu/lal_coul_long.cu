#include "hip/hip_runtime.h"
// **************************************************************************
//                               coul_long.cu
//                             -------------------
//                           Axel Kohlmeyer (Temple)
//
//  Device code for acceleration of the coul/long pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : July 2011
//    email                : a.kohlmeyer@temple.edu
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

#if (ARCH < 300)

#define store_answers_lq(f, e_coul, virial, ii, inum, tid,                  \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_PAIR];                                  \
                                                                            \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=e_coul;                                                 \
                                                                            \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
                                                                            \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    e_coul=red_acc[3][tid];                                                 \
                                                                            \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
                                                                            \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
                                                                            \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
                                                                            \
  if (offset==0) {                                                          \
    __global acctyp *ap1=engv+ii;                                           \
    if (eflag>0) {                                                          \
      *ap1=(acctyp)0;                                                       \
      ap1+=inum;                                                            \
      *ap1=e_coul*(acctyp)0.5;                                              \
      ap1+=inum;                                                            \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *ap1=virial[i]*(acctyp)0.5;                                         \
        ap1+=inum;                                                          \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#else

#define store_answers_lq(f, e_coul, virial, ii, inum, tid,                  \
                         t_per_atom, offset, eflag, vflag, ans, engv)       \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        e_coul += shfl_xor(e_coul, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    __global acctyp *ap1=engv+ii;                                           \
    if (eflag>0) {                                                          \
      *ap1=(acctyp)0;                                                       \
      ap1+=inum;                                                            \
      *ap1=e_coul*(acctyp)0.5;                                              \
      ap1+=inum;                                                            \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *ap1=virial[i]*(acctyp)0.5;                                         \
        ap1+=inum;                                                          \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#endif

__kernel void k_coul_long(const __global numtyp4 *restrict x_,
                          const __global numtyp *restrict scale,
                          const int lj_types,
                          const __global numtyp *restrict sp_cl_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const numtyp cut_coulsq, const numtyp qqrd2e,
                          const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_cl[4];
  sp_cl[0]=sp_cl_in[0];
  sp_cl[1]=sp_cl_in[1];
  sp_cl[2]=sp_cl_in[2];
  sp_cl[3]=sp_cl_in[3];

  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp qtmp; fetch(qtmp,i,q_tex);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = (numtyp)1.0-sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force, prefactor, _erfc;

        numtyp r = ucl_rsqrt(r2inv);
        numtyp grij = g_ewald * r;
        numtyp expm2 = ucl_exp(-grij*grij);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
        _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp/r;
        force = prefactor * (_erfc + EWALD_F*grij*expm2-factor_coul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          e_coul += prefactor*(_erfc-factor_coul);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_lq(f,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                     vflag,ans,engv);
  } // if ii
}

__kernel void k_coul_long_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp *restrict scale_in,
                               const __global numtyp *restrict sp_cl_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const numtyp cut_coulsq, const numtyp qqrd2e,
                               const numtyp g_ewald, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp scale[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_cl[4];
  if (tid<4)
    sp_cl[tid]=sp_cl_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES)
    scale[tid]=scale_in[tid];

  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul;
      factor_coul = (numtyp)1.0-sp_cl[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force, prefactor, _erfc;

        numtyp r = ucl_rsqrt(r2inv);
        numtyp grij = g_ewald * r;
        numtyp expm2 = ucl_exp(-grij*grij);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*grij);
        _erfc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * expm2;
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e * scale[mtype] * qtmp/r;
        force = prefactor*(_erfc + EWALD_F*grij*expm2-factor_coul) * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          e_coul += prefactor*(_erfc-factor_coul);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_lq(f,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                     vflag,ans,engv);
  } // if ii
}

