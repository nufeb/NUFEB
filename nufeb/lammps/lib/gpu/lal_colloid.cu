#include "hip/hip_runtime.h"
// **************************************************************************
//                                   colloid.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the colloid pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

__kernel void k_colloid(const __global numtyp4 *restrict x_,
                        const __global numtyp4 *restrict lj1,
                        const __global numtyp4 *restrict lj3,
                        const int lj_types,
                        const __global numtyp *restrict sp_lj_in,
                        const __global numtyp4 *restrict colloid1,
                        const __global numtyp4 *restrict colloid2,
                        const __global int *form,
                        const __global int *dev_nbor,
                        const __global int *dev_packed,
                        __global acctyp4 *restrict ans,
                        __global acctyp *restrict engv,
                        const int eflag, const int vflag, const int inum,
                        const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<lj1[mtype].z) {
        numtyp r,r2inv,r6inv;
        numtyp c1,c2,fR,evdwl;
        numtyp K[9],h[4],g[4];
        numtyp force = (numtyp)0;

        if (form[mtype]==0) { // SMALL_SMALL
          r2inv=ucl_recip(rsq);
          r6inv = r2inv*r2inv*r2inv;
          force = r2inv*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
          force*=factor_lj;
        } else if (form[mtype]==1) { // SMALL_LARGE
          c2 = colloid1[mtype].z;
          K[1] = c2*c2;
          K[2] = rsq;
          K[0] = K[1] - rsq;
          K[4] = rsq*rsq;
          K[3] = K[1] - K[2];
          K[3] *= K[3]*K[3];
          K[6] = K[3]*K[3];
          fR = colloid2[mtype].z*colloid1[mtype].x*c2*K[1]/K[3];
          force = (numtyp)4.0/(numtyp)15.0*fR *
             ((numtyp)2.0*(K[1]+K[2]) *
             (K[1]*((numtyp)5.0*K[1]+(numtyp)22.0*K[2])+(numtyp)5.0*K[4]) *
             colloid2[mtype].w/K[6]-(numtyp)5.0) / K[0];
          force*=factor_lj;
        } else if (form[mtype]==2) { // LARGE_LARGE
          r = ucl_sqrt(rsq);
          c1 = colloid1[mtype].y;
          c2 = colloid1[mtype].z;
          K[0] = c1*c2;
          K[1] = c1+c2;
          K[2] = c1-c2;
          K[3] = K[1]+r;
          K[4] = K[1]-r;
          K[5] = K[2]+r;
          K[6] = K[2]-r;
          K[7] = ucl_recip(K[3]*K[4]);
          K[8] = ucl_recip(K[5]*K[6]);
          g[0] = ucl_powr(K[3],(numtyp)-7.0);
          g[1] = -ucl_powr(-K[4],(numtyp)-7.0);
          g[2] = ucl_powr(K[5],(numtyp)-7.0);
          g[3] = -ucl_powr(-K[6],(numtyp)-7.0);
          h[0] = ((K[3]+(numtyp)5.0*K[1])*K[3]+(numtyp)30.0*K[0])*g[0];
          h[1] = ((K[4]+(numtyp)5.0*K[1])*K[4]+(numtyp)30.0*K[0])*g[1];
          h[2] = ((K[5]+(numtyp)5.0*K[2])*K[5]-(numtyp)30.0*K[0])*g[2];
          h[3] = ((K[6]+(numtyp)5.0*K[2])*K[6]-(numtyp)30.0*K[0])*g[3];
          g[0] *= (numtyp)42.0*K[0]/K[3]+(numtyp)6.0*K[1]+K[3];
          g[1] *= (numtyp)42.0*K[0]/K[4]+(numtyp)6.0*K[1]+K[4];
          g[2] *= (numtyp)-42.0*K[0]/K[5]+(numtyp)6.0*K[2]+K[5];
          g[3] *= (numtyp)-42.0*K[0]/K[6]+(numtyp)6.0*K[2]+K[6];

          fR = colloid1[mtype].x*colloid2[mtype].w/r/(numtyp)37800.0;
          evdwl = fR * (h[0]-h[1]-h[2]+h[3]);
          numtyp dUR = evdwl/r + (numtyp)5.0*fR*(g[0]+g[1]-g[2]-g[3]);
          numtyp dUA = -colloid1[mtype].x/(numtyp)3.0*r*
                       (((numtyp)2.0*K[0]*K[7]+(numtyp)1.0)*K[7] +
                       ((numtyp)2.0*K[0]*K[8]-(numtyp)1.0)*K[8]);
          force = factor_lj * (dUR+dUA)/r;
        }

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=(numtyp)0.0;
          if (form[mtype]==0) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          } else if (form[mtype]==1) {
            e=(numtyp)2.0/(numtyp)9.0*fR *
              ((numtyp)1.0-(K[1]*(K[1]*(K[1]/(numtyp)3.0+(numtyp)3.0*K[2]) +
              (numtyp)4.2*K[4])+K[2]*K[4]) * colloid2[mtype].w/K[6]);
          } else if (form[mtype]==2) {
            e=evdwl+colloid1[mtype].x/(numtyp)6.0 *
              ((numtyp)2.0*K[0]*(K[7]+K[8])-log(K[8]/K[7]));
          }
          energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_colloid_fast(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict lj1_in,
                             const __global numtyp4 *restrict lj3_in,
                             const __global numtyp *restrict sp_lj_in,
                             const __global numtyp4 *restrict colloid1_in,
                             const __global numtyp4 *restrict colloid2_in,
                             const __global int *form_in,
                             const __global int *dev_nbor,
                             const __global int *dev_packed,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag, const int inum,
                             const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 colloid1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 colloid2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local int form[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    colloid1[tid]=colloid1_in[tid];
    colloid2[tid]=colloid2_in[tid];
    form[tid]=form_in[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<lj1[mtype].z) {
        numtyp r,r2inv,r6inv;
        numtyp c1,c2,fR,evdwl;
        numtyp K[9],h[4],g[4];
        numtyp force = (numtyp)0;

        if (form[mtype]==0) { // SMALL_SMALL
          r2inv=ucl_recip(rsq);
          r6inv = r2inv*r2inv*r2inv;
          force = r2inv*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
          force*=factor_lj;
        } else if (form[mtype]==1) { // SMALL_LARGE
          c2 = colloid1[mtype].z;
          K[1] = c2*c2;
          K[2] = rsq;
          K[0] = K[1] - rsq;
          K[4] = rsq*rsq;
          K[3] = K[1] - K[2];
          K[3] *= K[3]*K[3];
          K[6] = K[3]*K[3];
          fR = colloid2[mtype].z*colloid1[mtype].x*c2*K[1]/K[3];
          force = (numtyp)4.0/(numtyp)15.0*fR *
            ((numtyp)2.0*(K[1]+K[2]) *
            (K[1]*((numtyp)5.0*K[1]+(numtyp)22.0*K[2])+(numtyp)5.0*K[4]) *
            colloid2[mtype].w/K[6]-(numtyp)5.0) / K[0];
          force*=factor_lj;
        } else if (form[mtype]==2) { // LARGE_LARGE
          r = ucl_sqrt(rsq);
          c1 = colloid1[mtype].y;
          c2 = colloid1[mtype].z;
          K[0] = c1*c2;
          K[1] = c1+c2;
          K[2] = c1-c2;
          K[3] = K[1]+r;
          K[4] = K[1]-r;
          K[5] = K[2]+r;
          K[6] = K[2]-r;
          K[7] = ucl_recip(K[3]*K[4]);
          K[8] = ucl_recip(K[5]*K[6]);
          g[0] = ucl_powr(K[3],(numtyp)-7.0);
          g[1] = -ucl_powr(-K[4],(numtyp)-7.0);
          g[2] = ucl_powr(K[5],(numtyp)-7.0);
          g[3] = -ucl_powr(-K[6],(numtyp)-7.0);
          h[0] = ((K[3]+(numtyp)5.0*K[1])*K[3]+(numtyp)30.0*K[0])*g[0];
          h[1] = ((K[4]+(numtyp)5.0*K[1])*K[4]+(numtyp)30.0*K[0])*g[1];
          h[2] = ((K[5]+(numtyp)5.0*K[2])*K[5]-(numtyp)30.0*K[0])*g[2];
          h[3] = ((K[6]+(numtyp)5.0*K[2])*K[6]-(numtyp)30.0*K[0])*g[3];
          g[0] *= (numtyp)42.0*K[0]/K[3]+(numtyp)6.0*K[1]+K[3];
          g[1] *= (numtyp)42.0*K[0]/K[4]+(numtyp)6.0*K[1]+K[4];
          g[2] *= (numtyp)-42.0*K[0]/K[5]+(numtyp)6.0*K[2]+K[5];
          g[3] *= (numtyp)-42.0*K[0]/K[6]+(numtyp)6.0*K[2]+K[6];

          fR = colloid1[mtype].x*colloid2[mtype].w/r/(numtyp)37800.0;
          evdwl = fR * (h[0]-h[1]-h[2]+h[3]);
          numtyp dUR = evdwl/r + (numtyp)5.0*fR*(g[0]+g[1]-g[2]-g[3]);
          numtyp dUA = -colloid1[mtype].x/(numtyp)3.0*r*
            (((numtyp)2.0*K[0]*K[7]+(numtyp)1.0)*K[7] +
            ((numtyp)2.0*K[0]*K[8]-(numtyp)1.0)*K[8]);
          force = factor_lj * (dUR+dUA)/r;
        } else force = (numtyp)0.0;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=(numtyp)0.0;
          if (form[mtype]==0) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          } else if (form[mtype]==1) {
            e=(numtyp)2.0/(numtyp)9.0*fR *
              ((numtyp)1.0-(K[1]*(K[1]*(K[1]/(numtyp)3.0+
              (numtyp)3.0*K[2])+(numtyp)4.2*K[4])+K[2]*K[4])*
              colloid2[mtype].w/K[6]);
          } else if (form[mtype]==2) {
            e=evdwl+colloid1[mtype].x/(numtyp)6.0 *
              ((numtyp)2.0*K[0]*(K[7]+K[8])-log(K[8]/K[7]));
          }
          energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

