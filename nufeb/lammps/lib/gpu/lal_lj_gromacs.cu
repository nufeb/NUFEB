#include "hip/hip_runtime.h"
// **************************************************************************
//                               lj_gromacs.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the gromacs/coul/long pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif

#else
#define pos_tex x_
#endif

__kernel void k_lj_gromacs(const __global numtyp4 *restrict x_,
                           const __global numtyp4 *restrict lj1,
                           const __global numtyp4 *restrict lj3,
                           const __global numtyp4 *restrict ljsw,
                           const int lj_types,
                           const __global numtyp *restrict sp_lj_in,
                           const __global int *dev_nbor,
                           const __global int *dev_packed,
                           __global acctyp4 *restrict ans,
                           __global acctyp *restrict engv,
                           const int eflag, const int vflag, const int inum,
                           const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<lj1[mtype].z) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, force, r6inv, t;

        r6inv = r2inv*r2inv*r2inv;
        force_lj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        if (rsq > lj1[mtype].w) {
          numtyp r = ucl_sqrt(rsq);
          t = r - lj3[mtype].z;
          numtyp fswitch = r*t*t*(ljsw[mtype].x + ljsw[mtype].y*t);
          force_lj += fswitch;
        }

        force = factor_lj*force_lj * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          e += lj3[mtype].w;
          if (rsq > lj1[mtype].w) {
            numtyp eswitch = t*t*t*(ljsw[mtype].z + ljsw[mtype].w*t);
            e += eswitch;
          }
          energy+=factor_lj*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
  } // if ii
}

__kernel void k_lj_gromacs_fast(const __global numtyp4 *restrict x_,
                                const __global numtyp4 *restrict lj1_in,
                                const __global numtyp4 *restrict lj3_in,
                                const __global numtyp4 *restrict ljsw_in,
                                const __global numtyp *restrict sp_lj_in,
                                const __global int *dev_nbor,
                                const __global int *dev_packed,
                                __global acctyp4 *restrict ans,
                                __global acctyp *restrict engv,
                                const int eflag, const int vflag, const int inum,
                                const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 ljsw[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    lj3[tid]=lj3_in[tid];
    ljsw[tid]=ljsw_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<lj1[mtype].z) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, force, r6inv, t;

        r6inv = r2inv*r2inv*r2inv;
        force_lj = r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);
        if (rsq > lj1[mtype].w) {
          numtyp r = ucl_sqrt(rsq);
          t = r - lj3[mtype].z;
          numtyp fswitch = r*t*t*(ljsw[mtype].x + ljsw[mtype].y*t);
          force_lj += fswitch;
        }

        force = factor_lj*force_lj * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          e += lj3[mtype].w;
          if (rsq > lj1[mtype].w) {
            numtyp eswitch = t*t*t*(ljsw[mtype].z + ljsw[mtype].w*t);
            e += eswitch;
          }
          energy+=factor_lj*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,
                  vflag,ans,engv);
  } // if ii
}

