#include "hip/hip_runtime.h"
// **************************************************************************
//                                re_squared.cu
//                             -------------------
//                               W. Michael Brown
//
//  Device code for RE-Squared potential acceleration
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Fri May 06 2011
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_ellipsoid_extra.h"
#endif

ucl_inline numtyp det_prime(const numtyp m[9], const numtyp m2[9])
{
  numtyp ans;
  ans = m2[0]*m[4]*m[8] - m2[0]*m[5]*m[7] -
        m[3]*m2[1]*m[8] + m[3]*m2[2]*m[7] +
        m[6]*m2[1]*m[5] - m[6]*m2[2]*m[4] +
        m[0]*m2[4]*m[8] - m[0]*m2[5]*m[7] -
        m2[3]*m[1]*m[8] + m2[3]*m[2]*m[7] +
        m[6]*m[1]*m2[5] - m[6]*m[2]*m2[4] +
        m[0]*m[4]*m2[8] - m[0]*m[5]*m2[7] -
        m[3]*m[1]*m2[8] + m[3]*m[2]*m2[7] +
        m2[6]*m[1]*m[5] - m2[6]*m[2]*m[4];
  return ans;
}

__kernel void k_resquared(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict q,
                          const __global numtyp4 *restrict shape,
                          const __global numtyp4 *restrict well,
                          const __global numtyp *restrict splj,
                          const __global numtyp2 *restrict sig_eps,
                          const int ntypes,
                          const __global int *dev_nbor,
                          const int stride,
                          __global acctyp4 *restrict ans,
                          const int astride,
                          __global acctyp *restrict engv,
                          __global int *restrict err_flag,
                          const int eflag, const int vflag, const int inum,
                          const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=splj[0];
  sp_lj[1]=splj[1];
  sp_lj[2]=splj[2];
  sp_lj[3]=splj[3];

  __local numtyp b_alpha, cr60;
  b_alpha=(numtyp)45.0/(numtyp)56.0;
  cr60=ucl_cbrt((numtyp)60.0);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_nbor,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int itype=ix.w;

    numtyp a1[9];       // Rotation matrix (lab->body)
    numtyp aTe1[9];     // A'*E
    numtyp gamma1[9];   // A'*S^2*A
    numtyp sa1[9];      // S^2*A;
    numtyp lA1_0[9], lA1_1[9], lA1_2[9]; // -A*rotation generator (x,y, or z)
    numtyp lAtwo1_0[9], lAtwo1_1[9], lAtwo1_2[9];  // A'*S^2*lA
    numtyp lAsa1_0[9], lAsa1_1[9], lAsa1_2[9];   // lAtwo+lA'*sa
    numtyp4 ishape;

    ishape=shape[itype];
    numtyp4 ishape2;
    ishape2.x=ishape.x*ishape.x;
    ishape2.y=ishape.y*ishape.y;
    ishape2.z=ishape.z*ishape.z;
    numtyp ilshape = ishape.x*ishape.y*ishape.z;

    {
      numtyp aTs[9];    // A1'*S1^2
      gpu_quat_to_mat_trans(q,i,a1);
      gpu_transpose_times_diag3(a1,well[itype],aTe1);
      gpu_transpose_times_diag3(a1,ishape2,aTs);
      gpu_diag_times3(ishape2,a1,sa1);
      gpu_times3(aTs,a1,gamma1);
      gpu_rotation_generator_x(a1,lA1_0);
      gpu_rotation_generator_y(a1,lA1_1);
      gpu_rotation_generator_z(a1,lA1_2);
      gpu_times3(aTs,lA1_0,lAtwo1_0);
      gpu_transpose_times3(lA1_0,sa1,lAsa1_0);
      gpu_plus3(lAsa1_0,lAtwo1_0,lAsa1_0);
      gpu_times3(aTs,lA1_1,lAtwo1_1);
      gpu_transpose_times3(lA1_1,sa1,lAsa1_1);
      gpu_plus3(lAsa1_1,lAtwo1_1,lAsa1_1);
      gpu_times3(aTs,lA1_2,lAtwo1_2);
      gpu_transpose_times3(lA1_2,sa1,lAsa1_2);
      gpu_plus3(lAsa1_2,lAtwo1_2,lAsa1_2);
    }
    ishape2.x=ucl_recip(ishape2.x);
    ishape2.y=ucl_recip(ishape2.y);
    ishape2.z=ucl_recip(ishape2.z);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_nbor[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int jtype=jx.w;

      // Compute r12
      numtyp r[3], rhat[3];
      numtyp rnorm;
      r[0] = jx.x-ix.x;
      r[1] = jx.y-ix.y;
      r[2] = jx.z-ix.z;
      rnorm = gpu_dot3(r,r);
      rnorm = ucl_rsqrt(rnorm);
      rhat[0] = r[0]*rnorm;
      rhat[1] = r[1]*rnorm;
      rhat[2] = r[2]*rnorm;


      numtyp a2[9];       // Rotation matrix (lab->body)
      numtyp gamma2[9];   // A'*S^2*A
      numtyp4 jshape;

      jshape=shape[jtype];
      numtyp4 jshape2;
      jshape2.x=jshape.x*jshape.x;
      jshape2.y=jshape.y*jshape.y;
      jshape2.z=jshape.z*jshape.z;
      {
        numtyp aTs[9];    // A1'*S1^2
        gpu_quat_to_mat_trans(q,j,a2);
        gpu_transpose_times_diag3(a2,jshape2,aTs);
        gpu_times3(aTs,a2,gamma2);
      }

      numtyp temp[9], s[3], z1[3], z2[3], v1[3], v2[3];
      numtyp sigma12, sigma1, sigma2;
      gpu_plus3(gamma1,gamma2,temp);
      gpu_mldivide3(temp,rhat,s,err_flag);
      sigma12 = ucl_rsqrt((numtyp)0.5*gpu_dot3(s,rhat));
      gpu_times_column3(a1,rhat,z1);
      gpu_times_column3(a2,rhat,z2);
      v1[0] = z1[0]*ishape2.x;
      v1[1] = z1[1]*ishape2.y;
      v1[2] = z1[2]*ishape2.z;
      v2[0] = z2[0]/jshape2.x;
      v2[1] = z2[1]/jshape2.y;
      v2[2] = z2[2]/jshape2.z;
      sigma1 = ucl_sqrt(gpu_dot3(z1,v1));
      sigma2 = ucl_sqrt(gpu_dot3(z2,v2));

      numtyp H12[9];
      numtyp dH;
      H12[0] = gamma1[0]*sigma1+gamma2[0]*sigma2;
      H12[1] = gamma1[1]*sigma1+gamma2[1]*sigma2;
      H12[2] = gamma1[2]*sigma1+gamma2[2]*sigma2;
      H12[3] = gamma1[3]*sigma1+gamma2[3]*sigma2;
      H12[4] = gamma1[4]*sigma1+gamma2[4]*sigma2;
      H12[5] = gamma1[5]*sigma1+gamma2[5]*sigma2;
      H12[6] = gamma1[6]*sigma1+gamma2[6]*sigma2;
      H12[7] = gamma1[7]*sigma1+gamma2[7]*sigma2;
      H12[8] = gamma1[8]*sigma1+gamma2[8]*sigma2;
      dH=gpu_det3(H12);

      numtyp sigma1p2, sigma2p2, lambda, nu;
      sigma1p2 = sigma1*sigma1;
      sigma2p2 = sigma2*sigma2;
      numtyp jlshape = jshape.x*jshape.y*jshape.z;
      lambda = ilshape*sigma1p2 + jlshape*sigma2p2;


      sigma1=ucl_recip(sigma1);
      sigma2=ucl_recip(sigma2);

      nu = ucl_rsqrt((sigma1+sigma2)/dH);
      gpu_times3(aTe1,a1,temp);

      numtyp sigma, epsilon;
      int mtype=fast_mul(ntypes,itype)+jtype;
      sigma = sig_eps[mtype].x;
      epsilon = sig_eps[mtype].y*factor_lj;

      numtyp w[3], temp2[9];
      numtyp h12,eta,chi,sprod,sigh,tprod;
      numtyp aTe2[9];     // A'*E
      gpu_transpose_times_diag3(a2,well[jtype],aTe2);
      gpu_times3(aTe2,a2,temp2);
      gpu_plus3(temp,temp2,temp);
      gpu_mldivide3(temp,rhat,w,err_flag);
      h12 = ucl_recip(rnorm)-sigma12;
      eta = lambda/nu;
      chi = (numtyp)2.0*gpu_dot3(rhat,w);
      sprod = ilshape * jlshape;
      sigh = sigma/h12;
      tprod = eta*chi*sigh;

      numtyp stemp, Ua;
      stemp = h12*(numtyp)0.5;
      Ua = (ishape.x+stemp)*(ishape.y+stemp)*
           (ishape.z+stemp)*(jshape.x+stemp)*
           (jshape.y+stemp)*(jshape.z+stemp);
      Ua = ((numtyp)1.0+(numtyp)3.0*tprod)*sprod/Ua;
      Ua = epsilon*Ua/(numtyp)-36.0;

      numtyp Ur;
      stemp = h12/cr60;
      Ur = (ishape.x+stemp)*(ishape.y+stemp)*
           (ishape.z+stemp)*(jshape.x+stemp)*
           (jshape.y+stemp)*(jshape.z+stemp);
      Ur = ((numtyp)1.0+b_alpha*tprod)*sprod/Ur;
      numtyp sigh6=sigh*sigh*sigh;
      sigh6*=sigh6;
      Ur = epsilon*Ur*sigh6/(numtyp)2025.0;

      energy+=Ua+Ur;

      // force

      numtyp vsigma1[3], vsigma2[3], gsigma1[9], gsigma2[9];
      numtyp sec, sigma12p3, sigma1p3, sigma2p3;
      sec = sigma*eta*chi;
      sigma12p3 = sigma12*sigma12*sigma12;
      sigma1p3 = sigma1/sigma1p2;
      sigma2p3 = sigma2/sigma2p2;
      vsigma1[0] = -sigma1p3*v1[0];
      vsigma1[1] = -sigma1p3*v1[1];
      vsigma1[2] = -sigma1p3*v1[2];
      vsigma2[0] = -sigma2p3*v2[0];
      vsigma2[1] = -sigma2p3*v2[1];
      vsigma2[2] = -sigma2p3*v2[2];
      gsigma1[0] = -gamma1[0]*sigma1p2;
      gsigma1[1] = -gamma1[1]*sigma1p2;
      gsigma1[2] = -gamma1[2]*sigma1p2;
      gsigma1[3] = -gamma1[3]*sigma1p2;
      gsigma1[4] = -gamma1[4]*sigma1p2;
      gsigma1[5] = -gamma1[5]*sigma1p2;
      gsigma1[6] = -gamma1[6]*sigma1p2;
      gsigma1[7] = -gamma1[7]*sigma1p2;
      gsigma1[8] = -gamma1[8]*sigma1p2;
      gsigma2[0] = -gamma2[0]*sigma2p2;
      gsigma2[1] = -gamma2[1]*sigma2p2;
      gsigma2[2] = -gamma2[2]*sigma2p2;
      gsigma2[3] = -gamma2[3]*sigma2p2;
      gsigma2[4] = -gamma2[4]*sigma2p2;
      gsigma2[5] = -gamma2[5]*sigma2p2;
      gsigma2[6] = -gamma2[6]*sigma2p2;
      gsigma2[7] = -gamma2[7]*sigma2p2;
      gsigma2[8] = -gamma2[8]*sigma2p2;

      numtyp tsig1sig2, tdH, teta1, teta2;
      numtyp fourw[3], spr[3];
      tsig1sig2 = eta/((numtyp)2.0*(sigma1+sigma2));
      tdH = eta/((numtyp)2.0*dH);
      teta1 = (numtyp)2.0*eta/lambda;
      teta2 = teta1*jlshape/sigma2p3;
      teta1 = teta1*ilshape/sigma1p3;
      fourw[0] = (numtyp)4.0*w[0];
      fourw[1] = (numtyp)4.0*w[1];
      fourw[2] = (numtyp)4.0*w[2];
      spr[0] = (numtyp)0.5*sigma12p3*s[0];
      spr[1] = (numtyp)0.5*sigma12p3*s[1];
      spr[2] = (numtyp)0.5*sigma12p3*s[2];

      numtyp hsec, dspu, pbsu;
      stemp = ucl_recip(ishape.x*(numtyp)2.0+h12)+
              ucl_recip(ishape.y*(numtyp)2.0+h12)+
              ucl_recip(ishape.z*(numtyp)2.0+h12)+
              ucl_recip(jshape.x*(numtyp)2.0+h12)+
              ucl_recip(jshape.y*(numtyp)2.0+h12)+
              ucl_recip(jshape.z*(numtyp)2.0+h12);
      hsec = ucl_recip(h12+(numtyp)3.0*sec);
      dspu = ucl_recip(h12)-hsec+stemp;
      pbsu = (numtyp)3.0*sigma*hsec;

      numtyp dspr, pbsr;
      stemp = ucl_recip(ishape.x*cr60+h12)+
              ucl_recip(ishape.y*cr60+h12)+
              ucl_recip(ishape.z*cr60+h12)+
              ucl_recip(jshape.x*cr60+h12)+
              ucl_recip(jshape.y*cr60+h12)+
              ucl_recip(jshape.z*cr60+h12);
      hsec = ucl_recip(h12+b_alpha*sec);
      dspr = (numtyp)7.0/h12-hsec+stemp;
      pbsr = b_alpha*sigma*hsec;

      numtyp dH12[9];
      numtyp dUa, dUr, deta, dchi, ddH, dh12;
      numtyp dsigma1, dsigma2;

      #pragma unroll
      for (int i=0; i<3; i++) {
        numtyp u[3], u1[3], u2[3];
        u[0] = -rhat[i]*rhat[0];
        u[1] = -rhat[i]*rhat[1];
        u[2] = -rhat[i]*rhat[2];
        u[i] += (numtyp)1.0;
        u[0] *= rnorm;
        u[1] *= rnorm;
        u[2] *= rnorm;
        gpu_times_column3(a1,u,u1);
        gpu_times_column3(a2,u,u2);
        dsigma1=gpu_dot3(u1,vsigma1);
        dsigma2=gpu_dot3(u2,vsigma2);
        dH12[0] = dsigma1*gsigma1[0]+dsigma2*gsigma2[0];
        dH12[1] = dsigma1*gsigma1[1]+dsigma2*gsigma2[1];
        dH12[2] = dsigma1*gsigma1[2]+dsigma2*gsigma2[2];
        dH12[3] = dsigma1*gsigma1[3]+dsigma2*gsigma2[3];
        dH12[4] = dsigma1*gsigma1[4]+dsigma2*gsigma2[4];
        dH12[5] = dsigma1*gsigma1[5]+dsigma2*gsigma2[5];
        dH12[6] = dsigma1*gsigma1[6]+dsigma2*gsigma2[6];
        dH12[7] = dsigma1*gsigma1[7]+dsigma2*gsigma2[7];
        dH12[8] = dsigma1*gsigma1[8]+dsigma2*gsigma2[8];
        ddH = det_prime(H12,dH12);
        deta = (dsigma1+dsigma2)*tsig1sig2;
        deta -= ddH*tdH;
        deta -= dsigma1*teta1+dsigma2*teta2;
        dchi = gpu_dot3(u,fourw);
        dh12 = rhat[i]+gpu_dot3(u,spr);
        dUa = pbsu*(eta*dchi+deta*chi)-dh12*dspu;
        dUr = pbsr*(eta*dchi+deta*chi)-dh12*dspr;
        numtyp force=dUr*Ur+dUa*Ua;
        if (i==0) {
          f.x+=force;
          if (vflag>0)
            virial[0]+=-r[0]*force;
        } else if (i==1) {
          f.y+=force;
          if (vflag>0) {
            virial[1]+=-r[1]*force;
            virial[3]+=-r[0]*force;
          }
        } else {
          f.z+=force;
          if (vflag>0) {
            virial[2]+=-r[2]*force;
            virial[4]+=-r[0]*force;
            virial[5]+=-r[1]*force;
          }
        }
      }

      // torque on i
      sigma1=ucl_recip(sigma1);

      numtyp fwae[3], p[3];
      gpu_row_times3(fourw,aTe1,fwae);

      {
        gpu_times_column3(lA1_0,rhat,p);
        dsigma1 = gpu_dot3(p,vsigma1);
        dH12[0] = lAsa1_0[0]*sigma1+dsigma1*gsigma1[0];
        dH12[1] = lAsa1_0[1]*sigma1+dsigma1*gsigma1[1];
        dH12[2] = lAsa1_0[2]*sigma1+dsigma1*gsigma1[2];
        dH12[3] = lAsa1_0[3]*sigma1+dsigma1*gsigma1[3];
        dH12[4] = lAsa1_0[4]*sigma1+dsigma1*gsigma1[4];
        dH12[5] = lAsa1_0[5]*sigma1+dsigma1*gsigma1[5];
        dH12[6] = lAsa1_0[6]*sigma1+dsigma1*gsigma1[6];
        dH12[7] = lAsa1_0[7]*sigma1+dsigma1*gsigma1[7];
        dH12[8] = lAsa1_0[8]*sigma1+dsigma1*gsigma1[8];
        ddH = det_prime(H12,dH12);
        deta = tsig1sig2*dsigma1-tdH*ddH;
        deta -= teta1*dsigma1;
        numtyp tempv[3];
        gpu_times_column3(lA1_0,w,tempv);
        dchi = -gpu_dot3(fwae,tempv);
        gpu_times_column3(lAtwo1_0,spr,tempv);
        dh12 = -gpu_dot3(s,tempv);

        dUa = pbsu*(eta*dchi + deta*chi)-dh12*dspu;
        dUr = pbsr*(eta*dchi + deta*chi)-dh12*dspr;
        tor.x -= (dUa*Ua+dUr*Ur);
      }

      {
        gpu_times_column3(lA1_1,rhat,p);
        dsigma1 = gpu_dot3(p,vsigma1);
        dH12[0] = lAsa1_1[0]*sigma1+dsigma1*gsigma1[0];
        dH12[1] = lAsa1_1[1]*sigma1+dsigma1*gsigma1[1];
        dH12[2] = lAsa1_1[2]*sigma1+dsigma1*gsigma1[2];
        dH12[3] = lAsa1_1[3]*sigma1+dsigma1*gsigma1[3];
        dH12[4] = lAsa1_1[4]*sigma1+dsigma1*gsigma1[4];
        dH12[5] = lAsa1_1[5]*sigma1+dsigma1*gsigma1[5];
        dH12[6] = lAsa1_1[6]*sigma1+dsigma1*gsigma1[6];
        dH12[7] = lAsa1_1[7]*sigma1+dsigma1*gsigma1[7];
        dH12[8] = lAsa1_1[8]*sigma1+dsigma1*gsigma1[8];
        ddH = det_prime(H12,dH12);
        deta = tsig1sig2*dsigma1-tdH*ddH;
        deta -= teta1*dsigma1;
        numtyp tempv[3];
        gpu_times_column3(lA1_1,w,tempv);
        dchi = -gpu_dot3(fwae,tempv);
        gpu_times_column3(lAtwo1_1,spr,tempv);
        dh12 = -gpu_dot3(s,tempv);

        dUa = pbsu*(eta*dchi + deta*chi)-dh12*dspu;
        dUr = pbsr*(eta*dchi + deta*chi)-dh12*dspr;
        tor.y -= (dUa*Ua+dUr*Ur);
      }

      {
        gpu_times_column3(lA1_2,rhat,p);
        dsigma1 = gpu_dot3(p,vsigma1);
        dH12[0] = lAsa1_2[0]*sigma1+dsigma1*gsigma1[0];
        dH12[1] = lAsa1_2[1]*sigma1+dsigma1*gsigma1[1];
        dH12[2] = lAsa1_2[2]*sigma1+dsigma1*gsigma1[2];
        dH12[3] = lAsa1_2[3]*sigma1+dsigma1*gsigma1[3];
        dH12[4] = lAsa1_2[4]*sigma1+dsigma1*gsigma1[4];
        dH12[5] = lAsa1_2[5]*sigma1+dsigma1*gsigma1[5];
        dH12[6] = lAsa1_2[6]*sigma1+dsigma1*gsigma1[6];
        dH12[7] = lAsa1_2[7]*sigma1+dsigma1*gsigma1[7];
        dH12[8] = lAsa1_2[8]*sigma1+dsigma1*gsigma1[8];
        ddH = det_prime(H12,dH12);
        deta = tsig1sig2*dsigma1-tdH*ddH;
        deta -= teta1*dsigma1;
        numtyp tempv[3];
        gpu_times_column3(lA1_2,w,tempv);
        dchi = -gpu_dot3(fwae,tempv);
        gpu_times_column3(lAtwo1_2,spr,tempv);
        dh12 = -gpu_dot3(s,tempv);

        dUa = pbsu*(eta*dchi + deta*chi)-dh12*dspu;
        dUr = pbsr*(eta*dchi + deta*chi)-dh12*dspr;
        tor.z -= (dUa*Ua+dUr*Ur);
      }

    } // for nbor
    store_answers_t(f,tor,energy,virial,ii,astride,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

