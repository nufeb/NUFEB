#include "hip/hip_runtime.h"
// **************************************************************************
//                                   eam.cu
//                             -------------------
//                   Trung Dac Nguyen, W. Michael Brown (ORNL)
//
//  Device code for acceleration of the eam pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> fp_tex;
texture<float4> rhor_sp1_tex;
texture<float4> rhor_sp2_tex;
texture<float4> frho_sp1_tex;
texture<float4> frho_sp2_tex;
texture<float4> z2r_sp1_tex;
texture<float4> z2r_sp2_tex;
#else
texture<int4> pos_tex;
texture<int2> fp_tex;
texture<int4> rhor_sp1_tex;
texture<int4> rhor_sp2_tex;
texture<int4> frho_sp1_tex;
texture<int4> frho_sp2_tex;
texture<int4> z2r_sp1_tex;
texture<int4> z2r_sp2_tex;
#endif

#else

#define pos_tex x_
#define fp_tex fp_
#define rhor_sp1_tex rhor_spline1
#define rhor_sp2_tex rhor_spline2
#define frho_sp1_tex frho_spline1
#define frho_sp2_tex frho_spline2
#define z2r_sp1_tex z2r_spline1
#define z2r_sp2_tex z2r_spline2

#endif

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MAX(A,B) ((A) > (B) ? (A) : (B))

#if (ARCH < 300)

#define store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,           \
                        eflag,vflag,engv,rdrho,nrho,i,rhomax)               \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[BLOCK_PAIR];                                     \
    red_acc[tid]=rho;                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s)                                                       \
         red_acc[tid] += red_acc[tid+s];                                    \
      }                                                                     \
      rho=red_acc[tid];                                                     \
  }                                                                         \
  if (offset==0) {                                                          \
    numtyp p = rho*rdrho + (numtyp)1.0;                                     \
    int m=p;                                                                \
    m = MAX(1,MIN(m,nrho-1));                                               \
    p -= m;                                                                 \
    p = MIN(p,(numtyp)1.0);                                                 \
    int index = type2frho[itype]*(nrho+1)+m;                                \
    numtyp4 coeff; fetch4(coeff,index,frho_sp1_tex);                        \
    numtyp fp = (coeff.x*p + coeff.y)*p + coeff.z;                          \
    fp_[i]=fp;                                                              \
    if (eflag>0) {                                                          \
      fetch4(coeff,index,frho_sp2_tex);                                     \
      energy = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;             \
      if (rho > rhomax) energy += fp*(rho-rhomax);                          \
      engv[ii]=energy;                                                      \
    }                                                                       \
  }

#define store_answers_eam(f, energy, virial, ii, inum, tid, t_per_atom,     \
                      offset, elag, vflag, ans, engv)                       \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_PAIR];                                  \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#else

#define store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,           \
                        eflag,vflag,engv,rdrho,nrho,i,rhomax)               \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1)                           \
        rho += shfl_xor(rho, s, t_per_atom);                                \
  }                                                                         \
  if (offset==0) {                                                          \
    numtyp p = rho*rdrho + (numtyp)1.0;                                     \
    int m=p;                                                                \
    m = MAX(1,MIN(m,nrho-1));                                               \
    p -= m;                                                                 \
    p = MIN(p,(numtyp)1.0);                                                 \
    int index = type2frho[itype]*(nrho+1)+m;                                \
    numtyp4 coeff; fetch4(coeff,index,frho_sp1_tex);                        \
    numtyp fp = (coeff.x*p + coeff.y)*p + coeff.z;                          \
    fp_[i]=fp;                                                              \
    if (eflag>0) {                                                          \
      fetch4(coeff,index,frho_sp2_tex);                                     \
      energy = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;             \
      if (rho > rhomax) energy += fp*(rho-rhomax);                          \
      engv[ii]=energy;                                          \
    }                                                                       \
  }

#define store_answers_eam(f, energy, virial, ii, inum, tid, t_per_atom,     \
                          offset, eflag, vflag, ans, engv)                  \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                     \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

#endif

__kernel void k_energy(const __global numtyp4 *restrict x_,
                       const __global int2 *restrict type2rhor_z2r,
                       const __global int *restrict type2frho,
                       const __global numtyp4 *restrict rhor_spline2,
                       const __global numtyp4 *restrict frho_spline1,
                       const __global numtyp4 *restrict frho_spline2,
                       const __global int *dev_nbor,
                       const __global int *dev_packed,
                       __global numtyp *restrict fp_,
                       __global acctyp *restrict engv,
                       const int eflag, const int inum, const int nbor_pitch,
                       const int ntypes,  const numtyp cutforcesq,
                       const numtyp rdr, const numtyp rdrho,
                       const numtyp rhomax, const int nrho,
                       const int nr, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp rho = (acctyp)0;
  acctyp energy = (acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        int mtype = jtype*ntypes+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        numtyp4 coeff; fetch4(coeff,index,rhor_sp2_tex);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor

    store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
        eflag,vflag,engv,rdrho,nrho,i,rhomax);
  } // if ii
}

__kernel void k_energy_fast(const __global numtyp4 *restrict x_,
                            const __global int2 *restrict type2rhor_z2r_in,
                            const __global int *restrict type2frho_in,
                            const __global numtyp4 *restrict rhor_spline2,
                            const __global numtyp4 *restrict frho_spline1,
                            const __global numtyp4 *restrict frho_spline2,
                            const __global int *dev_nbor,
                            const __global int *dev_packed,
                            __global numtyp *restrict fp_,
                            __global acctyp *restrict engv,
                            const int eflag,  const int inum,
                            const int nbor_pitch, const int ntypes,
                            const numtyp cutforcesq,  const numtyp rdr,
                            const numtyp rdrho, const numtyp rhomax,
                            const int nrho, const int nr,
                            const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local int type2frho[MAX_SHARED_TYPES];

  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }

  if (tid<MAX_SHARED_TYPES) {
    type2frho[tid]=type2frho_in[tid];
  }

  acctyp rho = (acctyp)0;
  acctyp energy = (acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        int jtype=fast_mul((int)MAX_SHARED_TYPES,jx.w);
        int mtype = jtype+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        numtyp4 coeff; fetch4(coeff,index,rhor_sp2_tex);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor

    store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
                    eflag,vflag,engv,rdrho,nrho,i,rhomax);
  } // if ii
}

__kernel void k_eam(const __global numtyp4 *restrict x_,
                    const __global numtyp *fp_,
                    const __global int2 *type2rhor_z2r,
                    const __global numtyp4 *rhor_spline1,
                    const __global numtyp4 *z2r_spline1,
                    const __global numtyp4 *z2r_spline2,
                    const __global int *dev_nbor,
                    const __global int *dev_packed,
                    __global acctyp4 *ans,
                    __global acctyp *engv,
                    const int eflag, const int vflag,  const int inum,
                    const int nbor_pitch, const int ntypes,
                    const numtyp cutforcesq,  const numtyp rdr, const int nr,
                    const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp ifp; fetch(ifp,i,fp_tex);  //fp_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        int mtype,index;
        numtyp4 coeff;

        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = jtype*ntypes+itype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        fetch4(coeff,index,z2r_sp1_tex);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        fetch4(coeff,index,z2r_sp2_tex);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;

        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip;
        fetch(psip,j,fp_tex);
        psip = ifp*rhojp + psip*rhoip + phip;
        numtyp force = -psip*recip;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          energy += phi;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
    store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

__kernel void k_eam_fast(const __global numtyp4 *x_,
                         const __global numtyp *fp_,
                         const __global int2 *type2rhor_z2r_in,
                         const __global numtyp4 *rhor_spline1,
                         const __global numtyp4 *z2r_spline1,
                         const __global numtyp4 *z2r_spline2,
                         const __global int *dev_nbor,
                         const __global int *dev_packed,
                         __global acctyp4 *ans,
                         __global acctyp *engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch, const numtyp cutforcesq,
                         const numtyp rdr, const int nr, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];

  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp ifp; fetch(ifp,i,fp_tex); //fp_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jw=jx.w;
      int jtype=fast_mul((int)MAX_SHARED_TYPES,jw);

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);

        numtyp4 coeff;
        int mtype,index;

        mtype = itype+jw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = jtype+iw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        fetch4(coeff,index,rhor_sp1_tex);
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = itype+jw;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        fetch4(coeff,index,z2r_sp1_tex);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        fetch4(coeff,index,z2r_sp2_tex);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;

        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip;
        fetch(psip,j,fp_tex);
        psip = ifp*rhojp + psip*rhoip + phip;
        numtyp force = -psip*recip;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          energy += phi;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
    store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

