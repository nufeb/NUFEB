#include "hip/hip_runtime.h"
// **************************************************************************
//                              ellipsoid_nbor.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for Ellipsoid neighbor routines
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_preprocessor.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

// ---------------------------------------------------------------------------
// Unpack neighbors from dev_ij array into dev_nbor matrix for coalesced access
// -- Only unpack neighbors matching the specified inclusive range of forms
// -- Only unpack neighbors within cutoff
// ---------------------------------------------------------------------------
__kernel void kernel_nbor(const __global numtyp4 *restrict x_,
                          const __global numtyp2 *restrict cut_form,
                          const int ntypes,
                          __global int *dev_nbor,
                          const int nbor_pitch, const int start, const int inum,
                          const __global int *dev_ij,
                          const int form_low, const int form_high) {

  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X+start;

  if (ii<inum) {
    int i=dev_ij[ii];
    int nbor=ii+nbor_pitch;
    int numj=dev_ij[nbor];
    nbor+=nbor_pitch;
    int nbor_end=nbor+fast_mul(numj,nbor_pitch);
    int packed=ii+nbor_pitch+nbor_pitch;

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul(iw,ntypes);
    int newj=0;
    for ( ; nbor<nbor_end; nbor+=nbor_pitch) {
      int j=dev_ij[nbor];
      j &= NEIGHMASK;
      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      int mtype=itype+jtype;
      numtyp2 cf=cut_form[mtype];
      if (cf.y>=form_low && cf.y<=form_high) {
        // Compute r12;
        numtyp rsq=jx.x-ix.x;
        rsq*=rsq;
        numtyp t=jx.y-ix.y;
        rsq+=t*t;
        t=jx.z-ix.z;
        rsq+=t*t;

        if (rsq<cf.x) {
          dev_nbor[packed]=j;
          packed+=nbor_pitch;
          newj++;
        }
      }
    }
    dev_nbor[ii+nbor_pitch]=newj;
  }
}

// ---------------------------------------------------------------------------
// Unpack neighbors from dev_ij array into dev_nbor matrix for coalesced access
// -- Only unpack neighbors matching the specified inclusive range of forms
// -- Only unpack neighbors within cutoff
// -- Fast version of routine that uses shared memory for LJ constants
// ---------------------------------------------------------------------------
__kernel void kernel_nbor_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp2 *restrict cut_form,
                               __global int *dev_nbor,
                               const int nbor_pitch, const int start,
                               const int inum,
                               const __global int *dev_ij,
                               const int form_low, const int form_high) {

  int ii=THREAD_ID_X;
  __local int form[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (ii<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    cutsq[ii]=cut_form[ii].x;
    form[ii]=cut_form[ii].y;
  }
  ii+=fast_mul((int)BLOCK_SIZE_X,(int)BLOCK_ID_X)+start;
  __syncthreads();

  if (ii<inum) {
    int i=dev_ij[ii];
    int nbor=ii+nbor_pitch;
    int numj=dev_ij[nbor];
    nbor+=nbor_pitch;
    int nbor_end=nbor+fast_mul(numj,nbor_pitch);
    int packed=ii+nbor_pitch+nbor_pitch;

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    int newj=0;
    for ( ; nbor<nbor_end; nbor+=nbor_pitch) {
      int j=dev_ij[nbor];
      j &= NEIGHMASK;
      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      int mtype=itype+jtype;

      if (form[mtype]>=form_low && form[mtype]<=form_high) {
        // Compute r12;
        numtyp rsq=jx.x-ix.x;
        rsq*=rsq;
        numtyp t=jx.y-ix.y;
        rsq+=t*t;
        t=jx.z-ix.z;
        rsq+=t*t;

        if (rsq<cutsq[mtype]) {
          dev_nbor[packed]=j;
          packed+=nbor_pitch;
          newj++;
        }
      }
    }
    dev_nbor[ii+nbor_pitch]=newj;
  }
}
