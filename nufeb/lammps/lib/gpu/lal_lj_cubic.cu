#include "hip/hip_runtime.h"
// **************************************************************************
//                                  lj_cubic.cu
//                             -------------------
//                               Trung Dac Nguyen
//
//  Device code for acceleration of the lj/cubic pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : ndactrung@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

// LJ quantities scaled by epsilon and rmin = sigma*2^1/6 (see src/pair_lj_cubic.h)

#define _RT6TWO (numtyp)1.1224621
#define _PHIS (numtyp)-0.7869823  /* energy at s */
#define _DPHIDS (numtyp)2.6899009 /* gradient at s */
#define _A3 (numtyp)27.93357 /* cubic coefficient */

__kernel void k_lj_cubic(const __global numtyp4 *restrict x_,
                         const __global numtyp4 *restrict lj1,
                         const __global numtyp4 *restrict lj2,
                         const __global numtyp2 *restrict lj3,
                         const int lj_types,
                         const __global numtyp *restrict sp_lj,
                         const __global int * dev_nbor,
                         const __global int * dev_packed,
                         __global acctyp4 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<lj1[mtype].z) {
        numtyp r2inv,r6inv,force,t;
        r2inv=ucl_recip(rsq);
        if (rsq <= lj2[mtype].x) {
          r6inv = r2inv*r2inv*r2inv;
          force = r6inv * (lj1[mtype].x*r6inv - lj1[mtype].y);
        } else {
          numtyp r = ucl_sqrt(rsq);
          numtyp rmin = lj2[mtype].z*_RT6TWO;
          t = (r - lj2[mtype].y)/rmin;
          force = lj2[mtype].w*(-_DPHIDS + _A3*t*t/2.0)*r/rmin;
        }

        force*=factor_lj*r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e;
          if (rsq <= lj2[mtype].x)
            e = r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          else
            e = lj2[mtype].w*(_PHIS + _DPHIDS*t - _A3*t*t*t/6.0);
          energy+=factor_lj*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_lj_cubic_fast(const __global numtyp4 *restrict x_,
                              const __global numtyp4 *restrict lj1_in,
                              const __global numtyp4 *restrict lj2_in,
                              const __global numtyp2 *restrict lj3_in,
                              const __global numtyp *restrict sp_lj_in,
                              const __global int * dev_nbor,
                              const __global int * dev_packed,
                              __global acctyp4 *restrict ans,
                              __global acctyp *restrict engv,
                              const int eflag, const int vflag, const int inum,
                              const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp2 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    lj2[tid]=lj2_in[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<lj1[mtype].z) {
        numtyp r2inv,r6inv,force,t;
        r2inv=ucl_recip(rsq);
        if (rsq <= lj2[mtype].x) {
          r6inv = r2inv*r2inv*r2inv;
          force = r6inv * (lj1[mtype].x*r6inv - lj1[mtype].y);
        } else {
          numtyp r = ucl_sqrt(rsq);
          numtyp rmin = lj2[mtype].z*_RT6TWO;
          t = (r - lj2[mtype].y)/rmin;
          force = lj2[mtype].w*(-_DPHIDS + _A3*t*t/2.0)*r/rmin;
        }

        force*=factor_lj*r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e;
          if (rsq <= lj2[mtype].x)
            e = r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          else
            e = lj2[mtype].w*(_PHIS + _DPHIDS*t - _A3*t*t*t/6.0);
          energy+=factor_lj*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

