#include "hip/hip_runtime.h"
// **************************************************************************
//                                coul_dsf.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the coul/dsf pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : 8/15/2012
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

#define MY_PIS (acctyp)1.77245385090551602729

__kernel void k_coul_dsf(const __global numtyp4 *restrict x_,
                         const int lj_types,
                         const __global numtyp *restrict sp_lj_in,
                         const __global int *dev_nbor,
                         const __global int *dev_packed,
                         __global acctyp4 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch,
                         const __global numtyp *restrict q_ ,
                         const numtyp cut_coulsq, const numtyp qqrd2e,
                         const numtyp e_shift, const numtyp f_shift,
                         const numtyp alpha, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);

    if (eflag>0) {
      acctyp e_self = -((acctyp)0.5*e_shift + alpha/MY_PIS) *
        qtmp*qtmp*qqrd2e/(acctyp)t_per_atom;
      e_coul += (acctyp)2.0*e_self;
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul, r, prefactor, erfcc;
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force;

        r = ucl_sqrt(rsq);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e*qtmp/r;
        numtyp erfcd = ucl_exp(-alpha*alpha*rsq);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*alpha*r);
        erfcc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * erfcd;
        forcecoul = prefactor * (erfcc + (numtyp)2.0*alpha/MY_PIS*r*erfcd +
          rsq*f_shift-factor_coul);

        force = forcecoul * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=prefactor*(erfcc-r*e_shift-rsq*f_shift-factor_coul);
          e_coul += e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_coul_dsf_fast(const __global numtyp4 *restrict x_,
                              const __global numtyp *restrict sp_lj_in,
                              const __global int *dev_nbor,
                              const __global int *dev_packed,
                              __global acctyp4 *restrict ans,
                              __global acctyp *restrict engv,
                              const int eflag, const int vflag, const int inum,
                              const int nbor_pitch,
                              const __global numtyp *restrict q_,
                              const numtyp cut_coulsq, const numtyp qqrd2e,
                              const numtyp e_shift, const numtyp f_shift,
                              const numtyp alpha, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);

    if (eflag>0) {
      acctyp e_self = -((acctyp)0.5*e_shift + alpha/MY_PIS) *
        qtmp*qtmp*qqrd2e/(acctyp)t_per_atom;
      e_coul += (acctyp)2.0*e_self;
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_coul, r, prefactor, erfcc;
      factor_coul = (numtyp)1.0-sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force;

        r = ucl_sqrt(rsq);
        fetch(prefactor,j,q_tex);
        prefactor *= qqrd2e*qtmp/r;
        numtyp erfcd = ucl_exp(-alpha*alpha*rsq);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*alpha*r);
        erfcc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * erfcd;
        forcecoul = prefactor * (erfcc + (numtyp)2.0*alpha/MY_PIS*r*erfcd +
          rsq*f_shift-factor_coul);

        force = forcecoul * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=prefactor*(erfcc-r*e_shift-rsq*f_shift-factor_coul);
          e_coul += e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

