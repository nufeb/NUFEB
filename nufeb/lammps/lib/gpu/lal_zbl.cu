#include "hip/hip_runtime.h"
// **************************************************************************
//                                   zbl.cu
//                             -------------------
//                              Trung Dac Nguyen
//
//  Device code for acceleration of the zbl pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : ndactrung@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

// ZBL constants

#define c1 (numtyp)0.02817
#define c2 (numtyp)0.28022
#define c3 (numtyp)0.50986
#define c4 (numtyp)0.18175

/* ----------------------------------------------------------------------
   compute ZBL pair energy
------------------------------------------------------------------------- */

ucl_inline numtyp e_zbl(numtyp r, numtyp d1aij, numtyp d2aij,
                      numtyp d3aij, numtyp d4aij, numtyp zzeij) {

  numtyp rinv = ucl_recip(r);

  numtyp sum = c1*ucl_exp(-d1aij*r);
  sum += c2*ucl_exp(-d2aij*r);
  sum += c3*ucl_exp(-d3aij*r);
  sum += c4*ucl_exp(-d4aij*r);

  numtyp result = zzeij*sum*rinv;

  return result;
};

/* ----------------------------------------------------------------------
   compute ZBL first derivative
------------------------------------------------------------------------- */

ucl_inline numtyp dzbldr(numtyp r, numtyp d1aij, numtyp d2aij,
                         numtyp d3aij, numtyp d4aij, numtyp zzeij) {
  numtyp rinv = ucl_recip(r);

  numtyp e1 = ucl_exp(-d1aij*r);
  numtyp e2 = ucl_exp(-d2aij*r);
  numtyp e3 = ucl_exp(-d3aij*r);
  numtyp e4 = ucl_exp(-d4aij*r);

  numtyp sum = c1*e1;
  sum += c2*e2;
  sum += c3*e3;
  sum += c4*e4;

  numtyp sum_p = -c1*d1aij*e1;
  sum_p -= c2*d2aij*e2;
  sum_p -= c3*d3aij*e3;
  sum_p -= c4*d4aij*e4;

  numtyp result = zzeij*(sum_p - sum*rinv)*rinv;

  return result;
};

__kernel void k_zbl(const __global numtyp4 *restrict x_,
                    const __global numtyp4 *restrict coeff1,
                    const __global numtyp4 *restrict coeff2,
                    const __global numtyp4 *restrict coeff3,
                    const double cut_globalsq,
                    const double cut_innersq,
                    const double cut_inner,
                    const int lj_types,
                    const __global int *dev_nbor,
                    const __global int *dev_packed,
                    __global acctyp4 *restrict ans,
                    __global acctyp *restrict engv,
                    const int eflag, const int vflag, const int inum,
                    const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cut_globalsq) {
        numtyp r, t, force;
        r = ucl_sqrt(rsq);
        force = dzbldr(r, coeff2[mtype].x, coeff2[mtype].y,
                       coeff2[mtype].z, coeff2[mtype].w, coeff1[mtype].z);
        if (rsq>cut_innersq) {
          t = r - cut_inner;
          force = t*t * (coeff1[mtype].x + coeff1[mtype].y*t);
        }
        force *= (numtyp)-1.0*ucl_recip(r);

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=e_zbl(r, coeff2[mtype].x, coeff2[mtype].y,
                         coeff2[mtype].z, coeff2[mtype].w, coeff1[mtype].z);
          e += coeff3[mtype].z;
          if (rsq > cut_innersq) {
            e += t*t*t * (coeff3[mtype].x + coeff3[mtype].y*t);
          }
          energy+=e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_zbl_fast(const __global numtyp4 *restrict x_,
                         const __global numtyp4 *restrict coeff1_in,
                         const __global numtyp4 *restrict coeff2_in,
                         const __global numtyp4 *restrict coeff3_in,
                         const double cut_globalsq,
                         const double cut_innersq,
                         const double cut_inner,
                         const __global int *dev_nbor,
                         const __global int *dev_packed,
                         __global acctyp4 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff1[tid]=coeff1_in[tid];
    coeff2[tid]=coeff2_in[tid];
    coeff3[tid]=coeff3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cut_globalsq) {
        numtyp r, t, force;
        r = ucl_sqrt(rsq);
        force = dzbldr(r, coeff2[mtype].x, coeff2[mtype].y,
                       coeff2[mtype].z, coeff2[mtype].w, coeff1[mtype].z);
        if (rsq>cut_innersq) {
          t = r - cut_inner;
          force += t*t * (coeff1[mtype].x + coeff1[mtype].y*t);
        }

        force *= (numtyp)-1.0*ucl_recip(r);

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=e_zbl(r, coeff2[mtype].x, coeff2[mtype].y,
                         coeff2[mtype].z, coeff2[mtype].w, coeff1[mtype].z);
          e += coeff3[mtype].z;
          if (rsq > cut_innersq) {
            e += t*t*t * (coeff3[mtype].x + coeff3[mtype].y*t);
          }
          energy+=e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

