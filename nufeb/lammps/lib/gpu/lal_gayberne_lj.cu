#include "hip/hip_runtime.h"
// **************************************************************************
//                                gayberne_lj.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for Gay-Berne - Lennard-Jones potential acceleration
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_ellipsoid_extra.h"
#endif

__kernel void k_gayberne_sphere_ellipsoid(const __global numtyp4 *restrict x_,
                                          const __global numtyp4 *restrict q,
                                          const __global numtyp4 *restrict shape,
                                          const __global numtyp4 *restrict well,
                                          const __global numtyp *restrict gum,
                                          const __global numtyp2 *restrict sig_eps,
                                          const int ntypes,
                                          const __global numtyp *restrict lshape,
                                          const __global int *dev_nbor,
                                          const int stride,
                                          __global acctyp4 *restrict ans,
                                          __global acctyp *restrict engv,
                                          __global int *restrict err_flag,
                                          const int eflag, const int vflag,
                                          const int start, const int inum,
                                          const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  sp_lj[0]=gum[3];
  sp_lj[1]=gum[4];
  sp_lj[2]=gum[5];
  sp_lj[3]=gum[6];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_nbor,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int itype=ix.w;

    numtyp oner=shape[itype].x;
    numtyp one_well=well[itype].x;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_nbor[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int jtype=jx.w;

      // Compute r12
      numtyp r12[3];
      r12[0] = jx.x-ix.x;
      r12[1] = jx.y-ix.y;
      r12[2] = jx.z-ix.z;
      numtyp ir = gpu_dot3(r12,r12);

      ir = ucl_rsqrt(ir);
      numtyp r = ucl_recip(ir);

      numtyp r12hat[3];
      r12hat[0]=r12[0]*ir;
      r12hat[1]=r12[1]*ir;
      r12hat[2]=r12[2]*ir;

      numtyp a2[9];
      gpu_quat_to_mat_trans(q,j,a2);

      numtyp u_r, dUr[3], eta;
      { // Compute U_r, dUr, eta, and teta
        // Compute g12
        numtyp g12[9];
        {
          {
            numtyp g2[9];
            gpu_diag_times3(shape[jtype],a2,g12);
            gpu_transpose_times3(a2,g12,g2);
            g12[0]=g2[0]+oner;
            g12[4]=g2[4]+oner;
            g12[8]=g2[8]+oner;
            g12[1]=g2[1];
            g12[2]=g2[2];
            g12[3]=g2[3];
            g12[5]=g2[5];
            g12[6]=g2[6];
            g12[7]=g2[7];
          }

          { // Compute U_r and dUr

            // Compute kappa
            numtyp kappa[3];
            gpu_mldivide3(g12,r12,kappa,err_flag);

            // -- kappa is now / r
            kappa[0]*=ir;
            kappa[1]*=ir;
            kappa[2]*=ir;

            // energy

            // compute u_r and dUr
            numtyp uslj_rsq;
            {
              // Compute distance of closest approach
              numtyp h12, sigma12;
              sigma12 = gpu_dot3(r12hat,kappa);
              sigma12 = ucl_rsqrt((numtyp)0.5*sigma12);
              h12 = r-sigma12;

              // -- kappa is now ok
              kappa[0]*=r;
              kappa[1]*=r;
              kappa[2]*=r;

              int mtype=fast_mul(ntypes,itype)+jtype;
              numtyp sigma = sig_eps[mtype].x;
              numtyp epsilon = sig_eps[mtype].y;
              numtyp varrho = sigma/(h12+gum[0]*sigma);
              numtyp varrho6 = varrho*varrho*varrho;
              varrho6*=varrho6;
              numtyp varrho12 = varrho6*varrho6;
              u_r = (numtyp)4.0*epsilon*(varrho12-varrho6);

              numtyp temp1 = ((numtyp)2.0*varrho12*varrho-varrho6*varrho)/sigma;
              temp1 = temp1*(numtyp)24.0*epsilon;
              uslj_rsq = temp1*sigma12*sigma12*sigma12*(numtyp)0.5;
              numtyp temp2 = gpu_dot3(kappa,r12hat);
              uslj_rsq = uslj_rsq*ir*ir;

              dUr[0] = temp1*r12hat[0]+uslj_rsq*(kappa[0]-temp2*r12hat[0]);
              dUr[1] = temp1*r12hat[1]+uslj_rsq*(kappa[1]-temp2*r12hat[1]);
              dUr[2] = temp1*r12hat[2]+uslj_rsq*(kappa[2]-temp2*r12hat[2]);
            }
          }
        }

        // Compute eta
        {
          eta = (numtyp)2.0*lshape[itype]*lshape[jtype];
          numtyp det_g12 = gpu_det3(g12);
          eta = ucl_powr(eta/det_g12,gum[1]);
        }
      }

      numtyp chi, dchi[3];
      { // Compute chi and dchi

        // Compute b12
        numtyp b12[9];
        {
          numtyp b2[9];
          gpu_diag_times3(well[jtype],a2,b12);
          gpu_transpose_times3(a2,b12,b2);
          b12[0]=b2[0]+one_well;
          b12[4]=b2[4]+one_well;
          b12[8]=b2[8]+one_well;
          b12[1]=b2[1];
          b12[2]=b2[2];
          b12[3]=b2[3];
          b12[5]=b2[5];
          b12[6]=b2[6];
          b12[7]=b2[7];
        }

        // compute chi_12
        numtyp iota[3];
        gpu_mldivide3(b12,r12,iota,err_flag);
        // -- iota is now iota/r
        iota[0]*=ir;
        iota[1]*=ir;
        iota[2]*=ir;
        chi = gpu_dot3(r12hat,iota);
        chi = ucl_powr(chi*(numtyp)2.0,gum[2]);

        // -- iota is now ok
        iota[0]*=r;
        iota[1]*=r;
        iota[2]*=r;

        numtyp temp1 = gpu_dot3(iota,r12hat);
        numtyp temp2 = (numtyp)-4.0*ir*ir*gum[2]*ucl_powr(chi,(gum[2]-(numtyp)1.0)/
                                                     gum[2]);
        dchi[0] = temp2*(iota[0]-temp1*r12hat[0]);
        dchi[1] = temp2*(iota[1]-temp1*r12hat[1]);
        dchi[2] = temp2*(iota[2]-temp1*r12hat[2]);
      }

      numtyp temp2 = factor_lj*eta*chi;
      if (eflag>0)
        energy+=u_r*temp2;
      numtyp temp1 = -eta*u_r*factor_lj;
      if (vflag>0) {
        r12[0]*=-1;
        r12[1]*=-1;
        r12[2]*=-1;
        numtyp ft=temp1*dchi[0]-temp2*dUr[0];
        f.x+=ft;
        virial[0]+=r12[0]*ft;
        ft=temp1*dchi[1]-temp2*dUr[1];
        f.y+=ft;
        virial[1]+=r12[1]*ft;
        virial[3]+=r12[0]*ft;
        ft=temp1*dchi[2]-temp2*dUr[2];
        f.z+=ft;
        virial[2]+=r12[2]*ft;
        virial[4]+=r12[0]*ft;
        virial[5]+=r12[1]*ft;
      } else {
        f.x+=temp1*dchi[0]-temp2*dUr[0];
        f.y+=temp1*dchi[1]-temp2*dUr[1];
        f.z+=temp1*dchi[2]-temp2*dUr[2];
      }
    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_gayberne_lj(const __global numtyp4 *restrict x_,
                            const __global numtyp4 *restrict lj1,
                            const __global numtyp4 *restrict lj3,
                            const int lj_types,
                            const __global numtyp *restrict gum,
                            const int stride,
                            const __global int *dev_ij,
                            __global acctyp4 *restrict ans,
                            __global acctyp *restrict engv,
                            __global int *restrict err_flag,
                            const int eflag, const int vflag, const int start,
                            const int inum, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  sp_lj[0]=gum[3];
  sp_lj[1]=gum[4];
  sp_lj[2]=gum[5];
  sp_lj[3]=gum[6];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_ij,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_ij[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      int ii=itype*lj_types+jtype;
      if (r2inv<lj1[ii].z && lj1[ii].w==SPHERE_SPHERE) {
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*r6inv*(lj1[ii].x*r6inv-lj1[ii].y);
        force*=factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[ii].x*r6inv-lj3[ii].y);
          energy+=factor_lj*(e-lj3[ii].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    acc_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  } // if ii
}

__kernel void k_gayberne_lj_fast(const __global numtyp4 *restrict x_,
                                 const __global numtyp4 *restrict lj1_in,
                                 const __global numtyp4 *restrict lj3_in,
                                 const __global numtyp *restrict gum,
                                 const int stride,
                                 const __global int *dev_ij,
                                 __global acctyp4 *restrict ans,
                                 __global acctyp *restrict engv,
                                 __global int *restrict err_flag,
                                 const int eflag, const int vflag,
                                 const int start, const int inum,
                                 const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  ii+=start;

  __local numtyp sp_lj[4];
  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<4)
    sp_lj[tid]=gum[tid+3];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info_e(dev_ij,stride,t_per_atom,ii,offset,i,numj,
                n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_ij[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex);
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<lj1[mtype].z && lj1[mtype].w==SPHERE_SPHERE) {
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = factor_lj*r2inv*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y);

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
          energy+=factor_lj*(e-lj3[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    acc_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
  } // if ii
}

