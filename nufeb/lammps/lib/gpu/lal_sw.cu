#include "hip/hip_runtime.h"
// **************************************************************************
//                                   sw.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the sw pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Tue March 26, 2013
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> sw1_tex;
texture<float4> sw2_tex;
texture<float4> sw3_tex;
#else
texture<int4,1> pos_tex;
texture<int4> sw1_tex;
texture<int4> sw2_tex;
texture<int4> sw3_tex;
#endif

#else
#define pos_tex x_
#define sw1_tex sw1
#define sw2_tex sw2
#define sw3_tex sw3
#endif

#define THIRD (numtyp)0.66666666666666666667

//#define THREE_CONCURRENT

#if (ARCH < 300)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_ELLIPSE];                               \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif

__kernel void k_sw_short_nbor(const __global numtyp4 *restrict x_,
                           const __global numtyp4 *restrict sw3,
                           const __global int *restrict map,
                           const __global int *restrict elem2param,
                           const int nelements,
                           const __global int * dev_nbor,
                           const __global int * dev_packed,
                           __global int * dev_short_nbor,
                           const int inum, const int nbor_pitch, const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    int ncount = 0;
    int m = nbor;
    dev_short_nbor[m] = 0;
    int nbor_short = nbor+n_stride;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      int nj = j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<sw3[ijparam].y) { // sw_cutsq = sw3[ijparam].y
        dev_short_nbor[nbor_short] = nj;
        nbor_short += n_stride;
        ncount++;
      }
    } // for nbor

    // store the number of neighbors for each thread
    dev_short_nbor[m] = ncount;

  } // if ii
}

__kernel void k_sw(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict sw1,
                   const __global numtyp4 *restrict sw2,
                   const __global numtyp4 *restrict sw3,
                   const __global int *restrict map,
                   const __global int *restrict elem2param,
                   const int nelements,
                   const __global int * dev_nbor,
                   const __global int * dev_packed,
                   const __global int * dev_short_nbor,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end, i, numj;
    const __global int* nbor_mem = dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=nbor_mem[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<sw3[ijparam].y) { // sw_cutsq = sw3[ijparam].y
        numtyp4 sw1_ijparam; fetch4(sw1_ijparam,ijparam,sw1_tex);
        numtyp sw_epsilon=sw1_ijparam.x;
        numtyp sw_sigma=sw1_ijparam.y;
        numtyp4 sw2_ijparam; fetch4(sw2_ijparam,ijparam,sw2_tex);
        numtyp sw_biga=sw2_ijparam.x;
        numtyp sw_bigb=sw2_ijparam.y;
        numtyp sw_powerp=sw2_ijparam.z;
        numtyp sw_powerq=sw2_ijparam.w;
        numtyp4 sw3_ijparam; fetch4(sw3_ijparam,ijparam,sw3_tex);
        numtyp sw_cut=sw3_ijparam.x;
        numtyp pre_sw_c1=sw_biga*sw_epsilon*sw_powerp*sw_bigb*
            pow(sw_sigma,sw_powerp);
        numtyp pre_sw_c2=sw_biga*sw_epsilon*sw_powerq*
            pow(sw_sigma,sw_powerq);
        numtyp pre_sw_c3=sw_biga*sw_epsilon*sw_bigb*
            pow(sw_sigma,sw_powerp+(numtyp)1.0);
        numtyp pre_sw_c4=sw_biga*sw_epsilon*
            pow(sw_sigma,sw_powerq+(numtyp)1.0);
        numtyp pre_sw_c5=sw_biga*sw_epsilon*sw_bigb*
            pow(sw_sigma,sw_powerp);
        numtyp pre_sw_c6=sw_biga*sw_epsilon*
            pow(sw_sigma,sw_powerq);

        numtyp r=ucl_sqrt(rsq);
        numtyp rp=ucl_powr(r,-sw_powerp);
        numtyp rq=ucl_powr(r,-sw_powerq);
        numtyp rainv=ucl_recip(r-sw_cut);
        numtyp expsrainv=ucl_exp(sw_sigma*rainv);
        rainv*=rainv*r;
        numtyp force = (pre_sw_c1*rp-pre_sw_c2*rq +
                       (pre_sw_c3*rp-pre_sw_c4*rq) * rainv)*
                       expsrainv*ucl_recip(rsq);

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0)
          energy+=(pre_sw_c5*rp - pre_sw_c6*rq) * expsrainv;

        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor

    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

#define threebody(delr1x, delr1y, delr1z, eflag, energy)                     \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_cut_ij);                                 \
  numtyp gsrainv1 = sw_sigma_gamma_ij * rainv1;                              \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rinvsq2 = ucl_recip(rsq2);                                          \
  numtyp rainv2 = ucl_recip(r2 - sw_cut_ik);                                 \
  numtyp gsrainv2 = sw_sigma_gamma_ik * rainv2;                              \
  numtyp gsrainvsq2 = gsrainv2*rainv2/r2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta_ijk;                                       \
  numtyp delcssq = delcs*delcs;                                              \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_lambda_epsilon_ijk * facexp*delcssq;                    \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp frad2 = facrad*gsrainvsq2;                                          \
  numtyp facang = sw_lambda_epsilon2_ijk * facexp*delcs;                     \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
                                                                             \
  numtyp csfac2 = rinvsq2*csfacang;                                          \
                                                                             \
  fkx = delr2x*(frad2+csfac2)-delr1x*facang12;                               \
  fky = delr2y*(frad2+csfac2)-delr1y*facang12;                               \
  fkz = delr2z*(frad2+csfac2)-delr1z*facang12;                               \
                                                                             \
  if (eflag>0)                                                               \
    energy+=facrad;                                                          \
  if (vflag>0) {                                                             \
    virial[0] += delr1x*fjx + delr2x*fkx;                                    \
    virial[1] += delr1y*fjy + delr2y*fky;                                    \
    virial[2] += delr1z*fjz + delr2z*fkz;                                    \
    virial[3] += delr1x*fjy + delr2x*fky;                                    \
    virial[4] += delr1x*fjz + delr2x*fkz;                                    \
    virial[5] += delr1y*fjz + delr2y*fkz;                                    \
  }                                                                          \
}

#define threebody_half(delr1x, delr1y, delr1z)                               \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_cut_ij);                                 \
  numtyp gsrainv1 = sw_sigma_gamma_ij * rainv1;                              \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rainv2 = ucl_recip(r2 - sw_cut_ik);                                 \
  numtyp gsrainv2 = sw_sigma_gamma_ik * rainv2;                              \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta_ijk;                                       \
  numtyp delcssq = delcs*delcs;                                              \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_lambda_epsilon_ijk * facexp*delcssq;                    \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp facang = sw_lambda_epsilon2_ijk * facexp*delcs;                     \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
}

__kernel void k_sw_three_center(const __global numtyp4 *restrict x_,
                                const __global numtyp4 *restrict sw1,
                                const __global numtyp4 *restrict sw2,
                                const __global numtyp4 *restrict sw3,
                                const __global int *restrict map,
                                const __global int *restrict elem2param,
                                const int nelements,
                                const __global int * dev_nbor,
                                const __global int * dev_packed,
                                const __global int * dev_short_nbor,
                                __global acctyp4 *restrict ans,
                                __global acctyp *restrict engv,
                                const int eflag, const int vflag,
                                const int inum,  const int nbor_pitch,
                                const int t_per_atom, const int evatom) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_sigma_gamma_ij, sw_cut_ij, sw_sigma_gamma_ik, sw_cut_ik;
  numtyp sw_costheta_ijk, sw_lambda_epsilon_ijk, sw_lambda_epsilon2_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }
    int nborj_start = nbor_j;

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = jx.x-ix.x;
      numtyp delr1y = jx.y-ix.y;
      numtyp delr1z = jx.z-ix.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 sw3_ijparam; fetch4(sw3_ijparam,ijparam,sw3_tex);

      if (rsq1 > sw3_ijparam.y) continue;

      numtyp4 sw1_ijparam; fetch4(sw1_ijparam,ijparam,sw1_tex);
      sw_sigma_gamma_ij=sw1_ijparam.y*sw1_ijparam.w; //sw_sigma*sw_gamma;
      sw_cut_ij=sw3_ijparam.x;

      int nbor_k,k_end;
      if (dev_packed==dev_nbor) {
        nbor_k=nborj_start-offset_j+offset_k;
        int numk = dev_short_nbor[nbor_k-n_stride];
        k_end = nbor_k+fast_mul(numk,n_stride);
      } else {
        nbor_k = nbor_j-offset_j+offset_k;
        if (nbor_k<=nbor_j) nbor_k += n_stride;
        k_end = nbor_end;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (dev_packed==dev_nbor && k <= j) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[itype*nelements*nelements+ktype*nelements+ktype];
        numtyp4 sw3_ikparam; fetch4(sw3_ikparam,ikparam,sw3_tex);

        numtyp delr2x = kx.x-ix.x;
        numtyp delr2y = kx.y-ix.y;
        numtyp delr2z = kx.z-ix.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        if (rsq2 < sw3_ikparam.y) {   // sw_cutsq=sw3[ikparam].y;
          numtyp4 sw1_ikparam; fetch4(sw1_ikparam,ikparam,sw1_tex);
          sw_sigma_gamma_ik=sw1_ikparam.y*sw1_ikparam.w; //sw_sigma*sw_gamma;
          sw_cut_ik=sw3_ikparam.x;

          int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];
          numtyp4 sw1_ijkparam; fetch4(sw1_ijkparam,ijkparam,sw1_tex);
          sw_lambda_epsilon_ijk=sw1_ijkparam.x*sw1_ijkparam.z; //sw_lambda*sw_epsilon;
          sw_lambda_epsilon2_ijk=(numtyp)2.0*sw_lambda_epsilon_ijk;
          numtyp4 sw3_ijkparam; fetch4(sw3_ijkparam,ijkparam,sw3_tex);
          sw_costheta_ijk=sw3_ijkparam.z;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x -= fjx + fkx;
          f.y -= fjy + fky;
          f.z -= fjz + fkz;
        }
      }
    } // for nbor

    numtyp pre;
    if (evatom==1)
      pre=THIRD;
    else
      pre=(numtyp)2.0;
    energy*=pre;
    for (int i=0; i<6; i++)
      virial[i]*=pre;

    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);

  } // if ii
}

__kernel void k_sw_three_end(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict sw1,
                             const __global numtyp4 *restrict sw2,
                             const __global numtyp4 *restrict sw3,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_ilist,
                             const __global int * dev_short_nbor,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_sigma_gamma_ij, sw_cut_ij, sw_sigma_gamma_ik, sw_cut_ik;
  numtyp sw_costheta_ijk, sw_lambda_epsilon_ijk, sw_lambda_epsilon2_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 sw3_ijparam; fetch4(sw3_ijparam,ijparam,sw3_tex);

      if (rsq1 > sw3_ijparam.y) continue;

      numtyp4 sw1_ijparam; fetch4(sw1_ijparam,ijparam,sw1_tex);
      sw_sigma_gamma_ij=sw1_ijparam.y*sw1_ijparam.w; //sw_sigma*sw_gamma;
      sw_cut_ij=sw3_ijparam.x;

      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; //jk

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        numtyp4 sw3_ikparam; fetch4(sw3_ikparam,ikparam,sw3_tex);

        if (rsq2 < sw3_ikparam.y) {
          numtyp4 sw1_ikparam; fetch4(sw1_ikparam,ikparam,sw1_tex);
          sw_sigma_gamma_ik=sw1_ikparam.y*sw1_ikparam.w; //sw_sigma*sw_gamma;
          sw_cut_ik=sw3_ikparam.x;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; //jik
          numtyp4 sw1_ijkparam; fetch4(sw1_ijkparam,ijkparam,sw1_tex);
          sw_lambda_epsilon_ijk=sw1_ijkparam.x*sw1_ijkparam.z; //sw_lambda*sw_epsilon;
          sw_lambda_epsilon2_ijk=(numtyp)2.0*sw_lambda_epsilon_ijk;
          numtyp4 sw3_ijkparam; fetch4(sw3_ijkparam,ijkparam,sw3_tex);
          sw_costheta_ijk=sw3_ijkparam.z;

          numtyp fjx, fjy, fjz;
          //if (evatom==0) {
            threebody_half(delr1x,delr1y,delr1z);
          //} else {
          //  numtyp fkx, fky, fkz;
          //  threebody(delr1x,delr1y,delr1z,eflag,energy);
          //}

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

__kernel void k_sw_three_end_vatom(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict sw1,
                             const __global numtyp4 *restrict sw2,
                             const __global numtyp4 *restrict sw3,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_ilist,
                             const __global int * dev_short_nbor,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_sigma_gamma_ij, sw_cut_ij, sw_sigma_gamma_ik, sw_cut_ik;
  numtyp sw_costheta_ijk, sw_lambda_epsilon_ijk, sw_lambda_epsilon2_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 sw3_ijparam; fetch4(sw3_ijparam,ijparam,sw3_tex);

      if (rsq1 > sw3_ijparam.y) continue;

      numtyp4 sw1_ijparam; fetch4(sw1_ijparam,ijparam,sw1_tex);
      sw_sigma_gamma_ij=sw1_ijparam.y*sw1_ijparam.w; //sw_sigma*sw_gamma;
      sw_cut_ij=sw3_ijparam.x;

      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; // jk
        numtyp4 sw3_ikparam; fetch4(sw3_ikparam,ikparam,sw3_tex);

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;

        if (rsq2 < sw3_ikparam.y) {
          numtyp4 sw1_ikparam; fetch4(sw1_ikparam,ikparam,sw1_tex);
          sw_sigma_gamma_ik=sw1_ikparam.y*sw1_ikparam.w; //sw_sigma*sw_gamma;
          sw_cut_ik=sw3_ikparam.x;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; // jik
          numtyp4 sw1_ijkparam; fetch4(sw1_ijkparam,ijkparam,sw1_tex);
          sw_lambda_epsilon_ijk=sw1_ijkparam.x*sw1_ijkparam.z; //sw_lambda*sw_epsilon;
          sw_lambda_epsilon2_ijk=(numtyp)2.0*sw_lambda_epsilon_ijk;
          numtyp4 sw3_ijkparam; fetch4(sw3_ijkparam,ijkparam,sw3_tex);
          sw_costheta_ijk=sw3_ijkparam.z;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    energy*=THIRD;
    for (int i=0; i<6; i++)
      virial[i]*=THIRD;
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

