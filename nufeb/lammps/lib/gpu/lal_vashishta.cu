#include "hip/hip_runtime.h"
// **************************************************************************
//                                 vashishta.cu
//                             -------------------
//                           Anders Hafreager (UiO)
//
//  Device code for acceleration of the vashishta pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Mon June 12, 2017
//    email                : andershaf@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> param1_tex;
texture<float4> param2_tex;
texture<float4> param3_tex;
texture<float4> param4_tex;
texture<float4> param5_tex;
#else
texture<int4,1> pos_tex;
texture<int4> param1_tex;
texture<int4> param2_tex;
texture<int4> param3_tex;
texture<int4> param4_tex;
texture<int4> param5_tex;
#endif

#else
#define pos_tex x_
#define param1_tex param1
#define param2_tex param2
#define param3_tex param3
#define param4_tex param4
#define param5_tex param5
#endif

#define THIRD (numtyp)0.66666666666666666667

//#define THREE_CONCURRENT

#if (ARCH < 300)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_ELLIPSE];                               \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif

__kernel void k_vashishta_short_nbor(const __global numtyp4 *restrict x_,
                                     const __global numtyp4 *restrict param4,
                                     const __global int *restrict map,
                                     const __global int *restrict elem2param,
                                     const int nelements, const int nparams,
                                     const __global int * dev_nbor,
                                     const __global int * dev_packed,
                                     __global int * dev_short_nbor,
                                     const int inum, const int nbor_pitch,
                                     const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    int ncount = 0;
    int m = nbor;
    dev_short_nbor[m] = 0;
    int nbor_short = nbor+n_stride;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      int nj = j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<param4[ijparam].x) { //param4[ijparam].x = r0sq; //param4[ijparam].z=cutsq
        dev_short_nbor[nbor_short] = nj;
        nbor_short += n_stride;
        ncount++;
      }
    } // for nbor

    // store the number of neighbors for each thread
    dev_short_nbor[m] = ncount;

  } // if ii
}

__kernel void k_vashishta(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict param1,
                   const __global numtyp4 *restrict param2,
                   const __global numtyp4 *restrict param3,
                   const __global numtyp4 *restrict param4,
                   const __global numtyp4 *restrict param5,
                   const __global int *restrict map,
                   const __global int *restrict elem2param,
                   const int nelements,
                   const __global int * dev_nbor,
                   const __global int * dev_packed,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end, i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<param4[ijparam].z) { // cutsq = param4[ijparam].z
        numtyp4 param1_ijparam; fetch4(param1_ijparam,ijparam,param1_tex);
        numtyp param1_eta=param1_ijparam.x;
        numtyp param1_lam1inv=param1_ijparam.y;
        numtyp param1_lam4inv=param1_ijparam.z;
        numtyp param1_zizj=param1_ijparam.w;

        numtyp4 param2_ijparam; fetch4(param2_ijparam,ijparam,param2_tex);
        numtyp param2_mbigd=param2_ijparam.x;
        numtyp param2_dvrc =param2_ijparam.y;
        numtyp param2_big6w=param2_ijparam.z;
        numtyp param2_heta =param2_ijparam.w;

        numtyp4 param3_ijparam; fetch4(param3_ijparam,ijparam,param3_tex);
        numtyp param3_bigh=param3_ijparam.x;
        numtyp param3_bigw=param3_ijparam.y;
        numtyp param3_dvrc=param3_ijparam.z;
        numtyp param3_c0  =param3_ijparam.w;

        numtyp r=ucl_sqrt(rsq);
        numtyp rinvsq=1.0/rsq;
        numtyp r4inv = rinvsq*rinvsq;
        numtyp r6inv = rinvsq*r4inv;

        numtyp reta = pow(r,-param1_eta);
        numtyp lam1r = r*param1_lam1inv;
        numtyp lam4r = r*param1_lam4inv;
        numtyp vc2 = param1_zizj * ucl_exp(-lam1r)/r;
        numtyp vc3 = param2_mbigd * r4inv*ucl_exp(-lam4r);

        numtyp force = (param2_dvrc*r
            - (4.0*vc3 + lam4r*vc3+param2_big6w*r6inv
               - param2_heta*reta - vc2 - lam1r*vc2)
            ) * rinvsq;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0)
          energy += (param3_bigh*reta+vc2-vc3-param3_bigw*r6inv-r*param3_dvrc+param3_c0);
          
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor

    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

#define threebody(delr1x, delr1y, delr1z, eflag, energy)                     \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - param_r0_ij);                               \
  numtyp gsrainv1 = param_gamma_ij * rainv1;                                 \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rinvsq2 = ucl_recip(rsq2);                                          \
  numtyp rainv2 = ucl_recip(r2 - param_r0_ik);                               \
  numtyp gsrainv2 = param_gamma_ik * rainv2;                                 \
  numtyp gsrainvsq2 = gsrainv2*rainv2/r2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - param_costheta_ijk;                                    \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = param_bigc_ijk*delcssq+1.0;                                \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = param_bigb_ijk * facexp*pcs;                               \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp frad2 = facrad*gsrainvsq2;                                          \
  numtyp facang = param_big2b_ijk * facexp*delcs/pcsinvsq;                   \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
                                                                             \
  numtyp csfac2 = rinvsq2*csfacang;                                          \
                                                                             \
  fkx = delr2x*(frad2+csfac2)-delr1x*facang12;                               \
  fky = delr2y*(frad2+csfac2)-delr1y*facang12;                               \
  fkz = delr2z*(frad2+csfac2)-delr1z*facang12;                               \
                                                                             \
  if (eflag>0)                                                               \
    energy+=facrad;                                                          \
  if (vflag>0) {                                                             \
    virial[0] += delr1x*fjx + delr2x*fkx;                                    \
    virial[1] += delr1y*fjy + delr2y*fky;                                    \
    virial[2] += delr1z*fjz + delr2z*fkz;                                    \
    virial[3] += delr1x*fjy + delr2x*fky;                                    \
    virial[4] += delr1x*fjz + delr2x*fkz;                                    \
    virial[5] += delr1y*fjz + delr2y*fkz;                                    \
  }                                                                          \
}

#define threebody_half(delr1x, delr1y, delr1z)                               \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - param_r0_ij);                               \
  numtyp gsrainv1 = param_gamma_ij * rainv1;                                 \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rainv2 = ucl_recip(r2 - param_r0_ik);                               \
  numtyp gsrainv2 = param_gamma_ik * rainv2;                                 \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - param_costheta_ijk;                                    \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = param_bigc_ijk*delcssq+1.0;                                \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = param_bigb_ijk * facexp*pcs;                               \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp facang = param_big2b_ijk * facexp*delcs/pcsinvsq;                   \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
}

__kernel void k_vashishta_three_center(const __global numtyp4 *restrict x_,
                                const __global numtyp4 *restrict param1,
                                const __global numtyp4 *restrict param2,
                                const __global numtyp4 *restrict param3,
                                const __global numtyp4 *restrict param4,
                                const __global numtyp4 *restrict param5,
                                const __global int *restrict map,
                                const __global int *restrict elem2param,
                                const int nelements,
                                const __global int * dev_nbor,
                                const __global int * dev_packed,
                                const __global int * dev_short_nbor,
                                __global acctyp4 *restrict ans,
                                __global acctyp *restrict engv,
                                const int eflag, const int vflag,
                                const int inum,  const int nbor_pitch,
                                const int t_per_atom, const int evatom) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }
    int nborj_start = nbor_j;

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = jx.x-ix.x;
      numtyp delr1y = jx.y-ix.y;
      numtyp delr1z = jx.z-ix.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      
      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij=param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue; // still keep this for neigh no and tpa > 1
      param_gamma_ij=param4_ijparam.y;
      param_r0_ij=param4_ijparam.w;
      
      int nbor_k,k_end;
      if (dev_packed==dev_nbor) {
        nbor_k=nborj_start-offset_j+offset_k;
        int numk = dev_short_nbor[nbor_k-n_stride];
        k_end = nbor_k+fast_mul(numk,n_stride);
      } else {
        nbor_k = nbor_j-offset_j+offset_k;
        if (nbor_k<=nbor_j) nbor_k += n_stride;
        k_end = nbor_end;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (dev_packed==dev_nbor && k <= j) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[itype*nelements*nelements+ktype*nelements+ktype];
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);

        numtyp delr2x = kx.x-ix.x;
        numtyp delr2y = kx.y-ix.y;
        numtyp delr2z = kx.z-ix.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;

        param_r0sq_ik=param4_ikparam.x;
        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;

          int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;
          param_costheta_ijk=param5_ijkparam.y;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x -= fjx + fkx;
          f.y -= fjy + fky;
          f.z -= fjz + fkz;
        }
      }
    } // for nbor

    numtyp pre;
    if (evatom==1)
      pre=THIRD;
    else
      pre=(numtyp)2.0;
    energy*=pre;
    for (int i=0; i<6; i++)
      virial[i]*=pre;

    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);

  } // if ii
}

__kernel void k_vashishta_three_end(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict param1,
                             const __global numtyp4 *restrict param2,
                             const __global numtyp4 *restrict param3,
                             const __global numtyp4 *restrict param4,
                             const __global numtyp4 *restrict param5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_ilist,
                             const __global int * dev_short_nbor,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij = param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue; // still keep this for neigh no and tpa > 1

      param_gamma_ij=param4_ijparam.y;
      param_r0_ij = param4_ijparam.w;
      
      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; //jk

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);
        param_r0sq_ik=param4_ikparam.x;

        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;
          
          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; //jik
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_costheta_ijk=param5_ijkparam.y;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;
          
          numtyp fjx, fjy, fjz;
          //if (evatom==0) {
            threebody_half(delr1x,delr1y,delr1z);
          //} else {
          //  numtyp fkx, fky, fkz;
          //  threebody(delr1x,delr1y,delr1z,eflag,energy);
          //}

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

__kernel void k_vashishta_three_end_vatom(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict param1,
                             const __global numtyp4 *restrict param2,
                             const __global numtyp4 *restrict param3,
                             const __global numtyp4 *restrict param4,
                             const __global numtyp4 *restrict param5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_ilist,
                             const __global int * dev_short_nbor,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp param_gamma_ij, param_r0sq_ij, param_r0_ij, param_gamma_ik, param_r0sq_ik, param_r0_ik;
  numtyp param_costheta_ijk, param_bigc_ijk, param_bigb_ijk, param_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 param4_ijparam; fetch4(param4_ijparam,ijparam,param4_tex);
      param_r0sq_ij=param4_ijparam.x;
      if (rsq1 > param_r0sq_ij) continue;  // still keep this for neigh no and tpa > 1

      param_gamma_ij=param4_ijparam.y;
      param_r0_ij=param4_ijparam.w;
      
      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; // jk
        numtyp4 param4_ikparam; fetch4(param4_ikparam,ikparam,param4_tex);

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        param_r0sq_ik=param4_ikparam.x;

        if (rsq2 < param_r0sq_ik) {
          param_gamma_ik=param4_ikparam.y;
          param_r0_ik=param4_ikparam.w;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; // jik
          numtyp4 param5_ijkparam; fetch4(param5_ijkparam,ijkparam,param5_tex);
          param_bigc_ijk=param5_ijkparam.x;
          param_costheta_ijk=param5_ijkparam.y;
          param_bigb_ijk=param5_ijkparam.z;
          param_big2b_ijk=param5_ijkparam.w;
          
          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    energy*=THIRD;
    for (int i=0; i<6; i++)
      virial[i]*=THIRD;
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

