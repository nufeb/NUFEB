#include "hip/hip_runtime.h"
// **************************************************************************
//                                   buck.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the buck pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

__kernel void k_buck(const __global numtyp4 *restrict x_,
                     const __global numtyp4 *restrict coeff1,
                     const __global numtyp4 *restrict coeff2,
                     const int lj_types,
                     const __global numtyp *restrict sp_lj_in,
                     const __global int *dev_nbor,
                     const __global int *dev_packed,
                     __global acctyp4 *restrict ans,
                     __global acctyp *restrict engv,
                     const int eflag,  const int vflag, const int inum,
                     const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (r2inv<coeff1[mtype].w) {
        numtyp r=ucl_sqrt(r2inv);
        numtyp rexp = ucl_exp(-r*coeff1[mtype].x);
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*(coeff1[mtype].y*r*rexp
                - coeff1[mtype].z*r6inv);
        force*=factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv;
          energy+=factor_lj*(e-coeff2[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_buck_fast(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict coeff1_in,
                          const __global numtyp4 *restrict coeff2_in,
                          const __global numtyp *restrict sp_lj_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff1[tid]=coeff1_in[tid];
    if (eflag>0)
      coeff2[tid]=coeff2_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r2inv = delx*delx+dely*dely+delz*delz;

      if (r2inv<coeff1[mtype].w) {
        numtyp r=ucl_sqrt(r2inv);
        numtyp rexp = ucl_exp(-r*coeff1[mtype].x);
        r2inv=ucl_recip(r2inv);
        numtyp r6inv = r2inv*r2inv*r2inv;
        numtyp force = r2inv*(coeff1[mtype].y*r*rexp
                - coeff1[mtype].z*r6inv);
        force*=factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=coeff2[mtype].x*rexp - coeff2[mtype].y*r6inv;
          energy+=factor_lj*(e-coeff2[mtype].z);
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

