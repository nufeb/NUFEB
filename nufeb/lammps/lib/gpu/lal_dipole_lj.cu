#include "hip/hip_runtime.h"
// **************************************************************************
//                                dipole_lj.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the dipole/cut pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
texture<float4> mu_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
texture<int4,1> mu_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#define mu_tex mu_
#endif

#if (ARCH < 300)

#define store_answers_tq(f, tor, energy, ecoul, virial, ii, inum, tid,      \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[8][BLOCK_PAIR];                                  \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=tor.x;                                                  \
    red_acc[4][tid]=tor.y;                                                  \
    red_acc[5][tid]=tor.z;                                                  \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<6; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    tor.x=red_acc[3][tid];                                                  \
    tor.y=red_acc[4][tid];                                                  \
    tor.z=red_acc[5][tid];                                                  \
    if (eflag>0 || vflag>0) {                                               \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      red_acc[6][tid]=energy;                                               \
      red_acc[7][tid]=ecoul;                                                \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<8; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
      energy=red_acc[6][tid];                                               \
      ecoul=red_acc[7][tid];                                                \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]=energy*(acctyp)0.5;                                             \
      ei+=inum;                                                           \
      engv[ei]=e_coul*(acctyp)0.5;                                             \
      ei+=inum;                                                           \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                        \
        ei+=inum;                                                         \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }

#else

#define store_answers_tq(f, tor, energy, e_coul, virial, ii, inum, tid,     \
                        t_per_atom, offset, eflag, vflag, ans, engv)        \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        tor.x += shfl_xor(tor.x, s, t_per_atom);                            \
        tor.y += shfl_xor(tor.y, s, t_per_atom);                            \
        tor.z += shfl_xor(tor.z, s, t_per_atom);                            \
        energy += shfl_xor(energy, s, t_per_atom);                          \
        e_coul += shfl_xor(e_coul, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]=energy*(acctyp)0.5;                                             \
      ei+=inum;                                                           \
      engv[ei]=e_coul*(acctyp)0.5;                                             \
      ei+=inum;                                                           \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]=virial[i]*(acctyp)0.5;                                        \
        ei+=inum;                                                         \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
    ans[ii+inum]=tor;                                                       \
  }

#endif

__kernel void k_dipole_lj(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict lj1,
                          const __global numtyp4 *restrict lj3,
                          const int lj_types,
                          const __global numtyp *restrict sp_lj_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp *restrict q_,
                          const __global numtyp4 *restrict mu_,
                          const __global numtyp *restrict cutsq,
                          const numtyp qqrd2e, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[8];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];
  sp_lj[4]=sp_lj_in[4];
  sp_lj[5]=sp_lj_in[5];
  sp_lj[6]=sp_lj_in[6];
  sp_lj[7]=sp_lj_in[7];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int itype=ix.w;

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv, r7inv;
        numtyp pre1, pre2, pre3, pre4;
        numtyp pdotp, pidotr, pjdotr;
        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*r3inv;

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
                  r7inv = r5inv*r2inv;
            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            pre1 = (numtyp)3.0*r5inv*pdotp - (numtyp)15.0*r7inv*pidotr*pjdotr;
            pre2 = (numtyp)3.0*r5inv*pjdotr;
            pre3 = (numtyp)3.0*r5inv*pidotr;
            pre4 = (numtyp)(-1.0)*r3inv;

            forcecoul.x += pre1*delx + pre2*mui.x + pre3*muj.x;
            forcecoul.y += pre1*dely + pre2*mui.y + pre3*muj.y;
            forcecoul.z += pre1*delz + pre2*mui.z + pre3*muj.z;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pre1 = (numtyp)3.0*qj*r5inv * pidotr;
            pre2 = qj*r3inv;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;
            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr;
            pre2 = qtmp*r3inv;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;
        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (eflag>0) {
          acctyp e = (acctyp)0.0;
          if (rsq < lj1[mtype].w) {
            e = qtmp*qj*rinv;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr;
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr;
            e *= fq;
          } else e = (acctyp)0.0;
            e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          }
        }
        if (vflag>0) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }

    } // for nbor
    store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_dipole_lj_fast(const __global numtyp4 *restrict x_,
                               const __global numtyp4 *restrict lj1_in,
                               const __global numtyp4 *restrict lj3_in,
                               const __global numtyp *restrict sp_lj_in,
                               const __global int *dev_nbor,
                               const __global int *dev_packed,
                               __global acctyp4 *restrict ans,
                               __global acctyp *restrict engv,
                               const int eflag, const int vflag, const int inum,
                               const int nbor_pitch,
                               const __global numtyp *restrict q_,
                               const __global numtyp4 *restrict mu_,
                               const __global numtyp *restrict _cutsq,
                               const numtyp qqrd2e, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 lj1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 lj3[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp cutsq[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[8];
  if (tid<8)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    lj1[tid]=lj1_in[tid];
    cutsq[tid]=_cutsq[tid];
    if (eflag>0)
      lj3[tid]=lj3_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp4 tor;
  tor.x=(acctyp)0;
  tor.y=(acctyp)0;
  tor.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
    numtyp4 mui; fetch4(mui,i,mu_tex); //mu_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<nbor_end; nbor+=n_stride) {
      int j=dev_packed[nbor];

      numtyp factor_lj, factor_coul;
      factor_lj = sp_lj[sbmask(j)];
      factor_coul = sp_lj[sbmask(j)+4];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp qj; fetch(qj,j,q_tex);
      numtyp4 muj; fetch4(muj,j,mu_tex); //mu_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[mtype]) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp force_lj, r6inv;
        numtyp rinv, r3inv, r5inv, r7inv;
        numtyp pre1, pre2, pre3, pre4;
        numtyp pdotp, pidotr, pjdotr;
        acctyp4 forcecoul, ticoul;
        acctyp4 force;

        forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
        ticoul.x = ticoul.y = ticoul.z = (acctyp)0;

        if (rsq < lj1[mtype].z) {
          r6inv = r2inv*r2inv*r2inv;
          force_lj = factor_lj*r6inv*(lj1[mtype].x*r6inv-lj1[mtype].y)*r2inv;
        } else force_lj = (numtyp)0.0;

        if (rsq < lj1[mtype].w) {
          rinv = ucl_rsqrt(rsq);

          // charge-charge
          if (qtmp != (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            pre1 = qtmp*qj*r3inv;

            forcecoul.x += pre1*delx;
            forcecoul.y += pre1*dely;
            forcecoul.z += pre1*delz;
          }

          // dipole-dipole
          if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            r7inv = r5inv*r2inv;
            pdotp  = mui.x*muj.x + mui.y*muj.y + mui.z*muj.z;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;

            pre1 = (numtyp)3.0*r5inv*pdotp - (numtyp)15.0*r7inv*pidotr*pjdotr;
            pre2 = (numtyp)3.0*r5inv*pjdotr;
            pre3 = (numtyp)3.0*r5inv*pidotr;
            pre4 = (numtyp)(-1.0)*r3inv;

            forcecoul.x += pre1*delx + pre2*mui.x + pre3*muj.x;
            forcecoul.y += pre1*dely + pre2*mui.y + pre3*muj.y;
            forcecoul.z += pre1*delz + pre2*mui.z + pre3*muj.z;

            numtyp crossx = pre4 * (mui.y*muj.z - mui.z*muj.y);
            numtyp crossy = pre4 * (mui.z*muj.x - mui.x*muj.z);
            numtyp crossz = pre4 * (mui.x*muj.y - mui.y*muj.x);

            ticoul.x += crossx + pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += crossy + pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += crossz + pre2 * (mui.x*dely - mui.y*delx);
          }

          // dipole-charge
          if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pidotr = mui.x*delx + mui.y*dely + mui.z*delz;
            pre1 = (numtyp)3.0*qj*r5inv * pidotr;
            pre2 = qj*r3inv;

            forcecoul.x += pre2*mui.x - pre1*delx;
            forcecoul.y += pre2*mui.y - pre1*dely;
            forcecoul.z += pre2*mui.z - pre1*delz;
            ticoul.x += pre2 * (mui.y*delz - mui.z*dely);
            ticoul.y += pre2 * (mui.z*delx - mui.x*delz);
            ticoul.z += pre2 * (mui.x*dely - mui.y*delx);
          }

          // charge-dipole
          if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0) {
            r3inv = r2inv*rinv;
            r5inv = r3inv*r2inv;
            pjdotr = muj.x*delx + muj.y*dely + muj.z*delz;
            pre1 = (numtyp)3.0*qtmp*r5inv * pjdotr;
            pre2 = qtmp*r3inv;

            forcecoul.x += pre1*delx - pre2*muj.x;
            forcecoul.y += pre1*dely - pre2*muj.y;
            forcecoul.z += pre1*delz - pre2*muj.z;
          }
        } else {
          forcecoul.x = forcecoul.y = forcecoul.z = (acctyp)0;
          ticoul.x = ticoul.y = ticoul.z = (acctyp)0;
        }

        numtyp fq = factor_coul*qqrd2e;
        force.x = fq*forcecoul.x + delx*force_lj;
        force.y = fq*forcecoul.y + dely*force_lj;
        force.z = fq*forcecoul.z + delz*force_lj;

        f.x+=force.x;
        f.y+=force.y;
        f.z+=force.z;
        tor.x+=fq*ticoul.x;
        tor.y+=fq*ticoul.y;
        tor.z+=fq*ticoul.z;

        if (eflag>0) {
          acctyp e = (acctyp)0;
          if (rsq < lj1[mtype].w) {
            e = qtmp*qj*rinv;
            if (mui.w > (numtyp)0.0 && muj.w > (numtyp)0.0)
              e += r3inv*pdotp - (numtyp)3.0*r5inv*pidotr*pjdotr;
            if (mui.w > (numtyp)0.0 && qj != (numtyp)0.0)
              e += -qj*r3inv*pidotr;
            if (muj.w > (numtyp)0.0 && qtmp != (numtyp)0.0)
              e += qtmp*r3inv*pjdotr;
            e *= fq;
          } else e = (acctyp)0;
          e_coul += e;

          if (rsq < lj1[mtype].z) {
            e=r6inv*(lj3[mtype].x*r6inv-lj3[mtype].y);
            energy+=factor_lj*(e-lj3[mtype].z);
          }
        }
        if (vflag>0) {
          virial[0] += delx*force.x;
          virial[1] += dely*force.y;
          virial[2] += delz*force.z;
          virial[3] += delx*force.y;
          virial[4] += delx*force.z;
          virial[5] += dely*force.z;
        }
      }

    } // for nbor
    store_answers_tq(f,tor,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

