#include "hip/hip_runtime.h"
// **************************************************************************
//                                 tersoff.cu
//                             -------------------
//                              Trung Dac Nguyen
//
//  Device code for acceleration of the tersoff pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//       begin                : Thu April 17, 2014
//       email                : ndactrung@gmail.com
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_tersoff_extra.h"

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> ts1_tex;
texture<float4> ts2_tex;
texture<float4> ts3_tex;
texture<float4> ts4_tex;
texture<float4> ts5_tex;
#else
texture<int4,1> pos_tex;
texture<int4> ts1_tex;
texture<int4> ts2_tex;
texture<int4> ts3_tex;
texture<int4> ts4_tex;
texture<int4> ts5_tex;
#endif

#else
#define pos_tex x_
#define ts1_tex ts1
#define ts2_tex ts2
#define ts3_tex ts3
#define ts4_tex ts4
#define ts5_tex ts5
#endif

//#define THREE_CONCURRENT

#define TWOTHIRD (numtyp)0.66666666666666666667

#define zeta_idx(nbor_mem, packed_mem, nbor_pitch, n_stride, t_per_atom,    \
                 i, nbor_j, offset_j, idx)                                  \
  if (nbor_mem==packed_mem) {                                               \
    int jj = (nbor_j-offset_j-2*nbor_pitch)/n_stride;                       \
    idx = jj*n_stride + i*t_per_atom + offset_j;                            \
  } else {                                                                  \
    idx = nbor_j;                                                           \
  }

#if (ARCH < 300)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom,       \
                        offset, eflag, vflag, ans, engv)                    \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_PAIR];                                  \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#define acc_zeta(z, tid, t_per_atom, offset)                                \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[BLOCK_PAIR];                                     \
    red_acc[tid]=z;                                                         \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        red_acc[tid] += red_acc[tid+s];                                     \
      }                                                                     \
    }                                                                       \
    z=red_acc[tid];                                                         \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom,       \
                        offset, eflag, vflag, ans, engv)                    \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      f.x += shfl_xor(f.x, s, t_per_atom);                                  \
      f.y += shfl_xor(f.y, s, t_per_atom);                                  \
      f.z += shfl_xor(f.z, s, t_per_atom);                                  \
      energy += shfl_xor(energy, s, t_per_atom);                            \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#define acc_zeta(z, tid, t_per_atom, offset)                                \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      z += shfl_xor(z, s, t_per_atom);                                      \
    }                                                                       \
  }

#endif

__kernel void k_tersoff_short_nbor(const __global numtyp4 *restrict x_,
                                   const __global numtyp *restrict cutsq,
                                   const __global int *restrict map,
                                   const __global int *restrict elem2param,
                                   const int nelements, const int nparams,
                                   const __global int * dev_nbor,
                                   const __global int * dev_packed,
                                   __global int * dev_short_nbor,
                                   const int inum, const int nbor_pitch,
                                   const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    int ncount = 0;
    int m = nbor;
    dev_short_nbor[m] = 0;
    int nbor_short = nbor+n_stride;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      int nj = j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq[ijparam]) {
        dev_short_nbor[nbor_short] = nj;
        nbor_short += n_stride;
        ncount++;
      }
    } // for nbor

    // store the number of neighbors for each thread
    dev_short_nbor[m] = ncount;

  } // if ii
}

// Tersoff is currently used for 3 elements at most: 3*3*3 = 27 entries
// while the block size should never be less than 32.
// SHARED_SIZE = 32 for now to reduce the pressure on the shared memory per block
// must be increased if there will be more than 3 elements in the future.

#define SHARED_SIZE 32

__kernel void k_tersoff_zeta(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict ts1_in,
                             const __global numtyp4 *restrict ts2_in,
                             const __global numtyp4 *restrict ts3_in,
                             const __global numtyp4 *restrict ts4_in,
                             const __global numtyp4 *restrict ts5_in,
                             const __global numtyp *restrict cutsq,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements, const int nparams,
                             __global acctyp4 * zetaij,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_short_nbor,
                             const int eflag, const int inum,
                             const int nbor_pitch, const int t_per_atom) {
  __local int tpa_sq,n_stride;
  tpa_sq = fast_mul(t_per_atom,t_per_atom);

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  // must be increased if there will be more than 3 elements in the future.
  __local numtyp4 ts1[SHARED_SIZE];
  __local numtyp4 ts2[SHARED_SIZE];
  __local numtyp4 ts3[SHARED_SIZE];
  __local numtyp4 ts4[SHARED_SIZE];
  __local numtyp4 ts5[SHARED_SIZE];
  if (tid<nparams) {
    ts1[tid]=ts1_in[tid];
    ts2[tid]=ts2_in[tid];
    ts3[tid]=ts3_in[tid];
    ts4[tid]=ts4_in[tid];
    ts5[tid]=ts5_in[tid];
  }

  acctyp z = (acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor_j, nbor_end, i, numj;
    const __global int* nbor_mem=dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }
    int nborj_start = nbor_j;

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute rij
      numtyp4 delr1, delr2;
      delr1.x = jx.x-ix.x;
      delr1.y = jx.y-ix.y;
      delr1.z = jx.z-ix.z;
      numtyp rsq1 = delr1.x*delr1.x+delr1.y*delr1.y+delr1.z*delr1.z;

      // compute zeta_ij
      z = (acctyp)0;

      int nbor_k = nborj_start-offset_j+offset_k;
      int k_end = nbor_end;
      if (dev_packed==dev_nbor) {
        int numk = dev_short_nbor[nbor_k-n_stride];
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k < k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == j) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex); //x_[k];
        int ktype=kx.w;
        ktype=map[ktype];
        int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];

        // Compute rik
        delr2.x = kx.x-ix.x;
        delr2.y = kx.y-ix.y;
        delr2.z = kx.z-ix.z;
        numtyp rsq2 = delr2.x*delr2.x+delr2.y*delr2.y+delr2.z*delr2.z;

        if (rsq2 > cutsq[ijkparam]) continue;

        numtyp4 ts1_ijkparam = ts1[ijkparam]; //fetch4(ts1_ijkparam,ijkparam,ts1_tex);
        numtyp ijkparam_lam3 = ts1_ijkparam.z;
        numtyp ijkparam_powermint = ts1_ijkparam.w;
        numtyp4 ts2_ijkparam = ts2[ijkparam]; //fetch4(ts2_ijkparam,ijkparam,ts2_tex);
        numtyp ijkparam_bigr = ts2_ijkparam.z;
        numtyp ijkparam_bigd = ts2_ijkparam.w;
        numtyp4 ts4_ijkparam = ts4[ijkparam]; //fetch4(ts4_ijkparam,ijkparam,ts4_tex);
        numtyp ijkparam_c = ts4_ijkparam.x;
        numtyp ijkparam_d = ts4_ijkparam.y;
        numtyp ijkparam_h = ts4_ijkparam.z;
        numtyp ijkparam_gamma = ts4_ijkparam.w;
        z += zeta(ijkparam_powermint, ijkparam_lam3, ijkparam_bigr, ijkparam_bigd,
                  ijkparam_c, ijkparam_d, ijkparam_h, ijkparam_gamma,
                  rsq1, rsq2, delr1, delr2);
      }

      // idx to zetaij is shifted by n_stride relative to nbor_j in dev_short_nbor
      int idx = nbor_j;
      if (dev_packed==dev_nbor) idx -= n_stride;
      acc_zeta(z, tid, t_per_atom, offset_k);

      numtyp4 ts1_ijparam = ts1[ijparam]; //fetch4(ts1_ijparam,ijparam,ts1_tex);
      numtyp ijparam_lam2 = ts1_ijparam.y;
      numtyp4 ts2_ijparam = ts2[ijparam]; //fetch4(ts2_ijparam,ijparam,ts2_tex);
      numtyp ijparam_bigb = ts2_ijparam.y;
      numtyp ijparam_bigr = ts2_ijparam.z;
      numtyp ijparam_bigd = ts2_ijparam.w;
      numtyp4 ts3_ijparam = ts3[ijparam]; //fetch4(ts3_ijparam,ijparam,ts3_tex);
      numtyp ijparam_c1 = ts3_ijparam.x;
      numtyp ijparam_c2 = ts3_ijparam.y;
      numtyp ijparam_c3 = ts3_ijparam.z;
      numtyp ijparam_c4 = ts3_ijparam.w;
      numtyp4 ts5_ijparam = ts5[ijparam]; //fetch4(ts5_ijparam,ijparam,ts5_tex);
      numtyp ijparam_beta = ts5_ijparam.x;
      numtyp ijparam_powern = ts5_ijparam.y;

      if (offset_k == 0) {
        numtyp fpfeng[4];
        force_zeta(ijparam_bigb, ijparam_bigr, ijparam_bigd, ijparam_lam2,
                   ijparam_beta, ijparam_powern, ijparam_c1, ijparam_c2, ijparam_c3,
                   ijparam_c4, rsq1, z, eflag, fpfeng);
        acctyp4 zij;
        zij.x = fpfeng[0];
        zij.y = fpfeng[1];
        zij.z = fpfeng[2];
        zij.w = z;
        zetaij[idx] = zij;
      }

    } // for nbor
  } // if ii
}

__kernel void k_tersoff_repulsive(const __global numtyp4 *restrict x_,
                                  const __global numtyp4 *restrict ts1_in,
                                  const __global numtyp4 *restrict ts2_in,
                                  const __global numtyp *restrict cutsq,
                                  const __global int *restrict map,
                                  const __global int *restrict elem2param,
                                  const int nelements, const int nparams,
                                  const __global int * dev_nbor,
                                  const __global int * dev_packed,
                                  const __global int * dev_short_nbor,
                                  __global acctyp4 *restrict ans,
                                  __global acctyp *restrict engv,
                                  const int eflag, const int vflag,
                                  const int inum, const int nbor_pitch,
                                  const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 ts1[SHARED_SIZE];
  __local numtyp4 ts2[SHARED_SIZE];
  if (tid<nparams) {
    ts1[tid]=ts1_in[tid];
    ts2[tid]=ts2_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end, i, numj;
    const __global int* nbor_mem=dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=nbor_mem[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12

      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      // rsq<cutsq[ijparam]
      numtyp feng[2];
      numtyp ijparam_lam1 = ts1[ijparam].x;
      numtyp4 ts2_ijparam = ts2[ijparam];
      numtyp ijparam_biga = ts2_ijparam.x;
      numtyp ijparam_bigr = ts2_ijparam.z;
      numtyp ijparam_bigd = ts2_ijparam.w;

      repulsive(ijparam_bigr, ijparam_bigd, ijparam_lam1, ijparam_biga,
                rsq, eflag, feng);

      numtyp force = feng[0];
      f.x+=delx*force;
      f.y+=dely*force;
      f.z+=delz*force;

      if (eflag>0)
        energy+=feng[1];
      if (vflag>0) {
        virial[0] += delx*delx*force;
        virial[1] += dely*dely*force;
        virial[2] += delz*delz*force;
        virial[3] += delx*dely*force;
        virial[4] += delx*delz*force;
        virial[5] += dely*delz*force;
      }
    } // for nbor

    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

__kernel void k_tersoff_three_center(const __global numtyp4 *restrict x_,
                                     const __global numtyp4 *restrict ts1_in,
                                     const __global numtyp4 *restrict ts2_in,
                                     const __global numtyp4 *restrict ts4_in,
                                     const __global numtyp *restrict cutsq,
                                     const __global int *restrict map,
                                     const __global int *restrict elem2param,
                                     const int nelements, const int nparams,
                                     const __global acctyp4 *restrict zetaij,
                                     const __global int * dev_nbor,
                                     const __global int * dev_packed,
                                     const __global int * dev_short_nbor,
                                     __global acctyp4 *restrict ans,
                                     __global acctyp *restrict engv,
                                     const int eflag, const int vflag,
                                     const int inum,  const int nbor_pitch,
                                     const int t_per_atom, const int evatom) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp lam3, powermint, bigr, bigd, c, d, h, gamma;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset); // offset ranges from 0 to tpa_sq-1

  __local numtyp4 ts1[SHARED_SIZE];
  __local numtyp4 ts2[SHARED_SIZE];
  __local numtyp4 ts4[SHARED_SIZE];
  if (tid<nparams) {
    ts1[tid]=ts1_in[tid];
    ts2[tid]=ts2_in[tid];
    ts4[tid]=ts4_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  numtyp tpainv = ucl_recip((numtyp)t_per_atom);

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end;
    const __global int* nbor_mem=dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }
    int nborj_start = nbor_j;

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delr1[3];
      delr1[0] = jx.x-ix.x;
      delr1[1] = jx.y-ix.y;
      delr1[2] = jx.z-ix.z;
      numtyp rsq1 = delr1[0]*delr1[0] + delr1[1]*delr1[1] + delr1[2]*delr1[2];

      numtyp r1 = ucl_sqrt(rsq1);
      numtyp r1inv = ucl_rsqrt(rsq1);

      // look up for zeta_ij
      // idx to zetaij is shifted by n_stride relative to nbor_j in dev_short_nbor
      int idx = nbor_j;
      if (dev_packed==dev_nbor) idx -= n_stride;
      acctyp4 zeta_ij = zetaij[idx]; // fetch(zeta_ij,idx,zeta_tex);
      numtyp force = zeta_ij.x*tpainv;
      numtyp prefactor = zeta_ij.y;
      f.x += delr1[0]*force;
      f.y += delr1[1]*force;
      f.z += delr1[2]*force;

      if (eflag>0) {
        energy+=zeta_ij.z*tpainv;
      }
      if (vflag>0) {
        numtyp mforce = -force;
        virial[0] += delr1[0]*delr1[0]*mforce;
        virial[1] += delr1[1]*delr1[1]*mforce;
        virial[2] += delr1[2]*delr1[2]*mforce;
        virial[3] += delr1[0]*delr1[1]*mforce;
        virial[4] += delr1[0]*delr1[2]*mforce;
        virial[5] += delr1[1]*delr1[2]*mforce;
      }

      int nbor_k = nborj_start-offset_j+offset_k;
      int k_end = nbor_end;
      if (dev_packed==dev_nbor) {
        int numk = dev_short_nbor[nbor_k-n_stride];
        k_end = nbor_k+fast_mul(numk,n_stride);
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (j == k) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];

        numtyp delr2[3];
        delr2[0] = kx.x-ix.x;
        delr2[1] = kx.y-ix.y;
        delr2[2] = kx.z-ix.z;
        numtyp rsq2 = delr2[0]*delr2[0] + delr2[1]*delr2[1] + delr2[2]*delr2[2];

        if (rsq2 > cutsq[ijkparam]) continue;
        numtyp r2 = ucl_sqrt(rsq2);
        numtyp r2inv = ucl_rsqrt(rsq2);

        numtyp fi[3], fj[3], fk[3];
        numtyp4 ts1_ijkparam = ts1[ijkparam]; //fetch4(ts1_ijkparam,ijkparam,ts1_tex);
        lam3 = ts1_ijkparam.z;
        powermint = ts1_ijkparam.w;
        numtyp4 ts2_ijkparam = ts2[ijkparam]; //fetch4(ts2_ijkparam,ijkparam,ts2_tex);
        bigr = ts2_ijkparam.z;
        bigd = ts2_ijkparam.w;
        numtyp4 ts4_ijkparam = ts4[ijkparam]; //fetch4(ts4_ijkparam,ijkparam,ts4_tex);
        c = ts4_ijkparam.x;
        d = ts4_ijkparam.y;
        h = ts4_ijkparam.z;
        gamma = ts4_ijkparam.w;
        if (vflag>0)
          attractive(bigr, bigd, powermint, lam3, c, d, h, gamma,
                     prefactor, r1, r1inv, r2, r2inv, delr1, delr2, fi, fj, fk);
        else
          attractive_fi(bigr, bigd, powermint, lam3, c, d, h, gamma,
                        prefactor, r1, r1inv, r2, r2inv, delr1, delr2, fi);
        f.x += fi[0];
        f.y += fi[1];
        f.z += fi[2];

        if (vflag>0) {
          acctyp v[6];
          numtyp pre = (numtyp)2.0;
          if (evatom==1) pre = TWOTHIRD;
          v[0] = pre*(delr1[0]*fj[0] + delr2[0]*fk[0]);
          v[1] = pre*(delr1[1]*fj[1] + delr2[1]*fk[1]);
          v[2] = pre*(delr1[2]*fj[2] + delr2[2]*fk[2]);
          v[3] = pre*(delr1[0]*fj[1] + delr2[0]*fk[1]);
          v[4] = pre*(delr1[0]*fj[2] + delr2[0]*fk[2]);
          v[5] = pre*(delr1[1]*fj[2] + delr2[1]*fk[2]);

          virial[0] += v[0]; virial[1] += v[1]; virial[2] += v[2];
          virial[3] += v[3]; virial[4] += v[4]; virial[5] += v[5];
        }
      } // nbor_k
    } // for nbor_j

    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,
                     offset,eflag,vflag,ans,engv);
  } // if ii
}

__kernel void k_tersoff_three_end(const __global numtyp4 *restrict x_,
                                  const __global numtyp4 *restrict ts1_in,
                                  const __global numtyp4 *restrict ts2_in,
                                  const __global numtyp4 *restrict ts4_in,
                                  const __global numtyp *restrict cutsq,
                                  const __global int *restrict map,
                                  const __global int *restrict elem2param,
                                  const int nelements, const int nparams,
                                  const __global acctyp4 *restrict zetaij,
                                  const __global int * dev_nbor,
                                  const __global int * dev_packed,
                                  const __global int * dev_ilist,
                                  const __global int * dev_short_nbor,
                                  __global acctyp4 *restrict ans,
                                  __global acctyp *restrict engv,
                                  const int eflag, const int vflag,
                                  const int inum,  const int nbor_pitch,
                                  const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp lam3, powermint, bigr, bigd, c, d, h, gamma;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  __local numtyp4 ts1[SHARED_SIZE];
  __local numtyp4 ts2[SHARED_SIZE];
  __local numtyp4 ts4[SHARED_SIZE];
  if (tid<nparams) {
    ts1[tid]=ts1_in[tid];
    ts2[tid]=ts2_in[tid];
    ts4[tid]=ts4_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __local int red_acc[2*BLOCK_PAIR];

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem=dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    numtyp tpainv = ucl_recip((numtyp)t_per_atom);

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delr1[3];
      delr1[0] = jx.x-ix.x;
      delr1[1] = jx.y-ix.y;
      delr1[2] = jx.z-ix.z;
      numtyp rsq1 = delr1[0]*delr1[0] + delr1[1]*delr1[1] + delr1[2]*delr1[2];

      numtyp mdelr1[3];
      mdelr1[0] = -delr1[0];
      mdelr1[1] = -delr1[1];
      mdelr1[2] = -delr1[2];

      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }
      int nbork_start = nbor_k;

      // look up for zeta_ji: find i in the j's neighbor list
      int m = tid / t_per_atom;
      int ijnum = -1;
      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;
        if (k == i) {
          ijnum = nbor_k;
          red_acc[2*m+0] = ijnum;
          red_acc[2*m+1] = offset_k;
          break;
        }
      }

      numtyp r1 = ucl_sqrt(rsq1);
      numtyp r1inv = ucl_rsqrt(rsq1);
      int offset_kf;
      if (ijnum >= 0) {
        offset_kf = offset_k;
      } else {
        ijnum = red_acc[2*m+0];
        offset_kf = red_acc[2*m+1];
      }

      // idx to zetaij is shifted by n_stride relative to ijnum in dev_short_nbor
      int idx = ijnum;
      if (dev_packed==dev_nbor) idx -= n_stride;
      acctyp4 zeta_ji = zetaij[idx]; // fetch(zeta_ji,idx,zeta_tex);
      numtyp force = zeta_ji.x*tpainv;
      numtyp prefactor_ji = zeta_ji.y;
      f.x += delr1[0]*force;
      f.y += delr1[1]*force;
      f.z += delr1[2]*force;

      if (eflag>0) {
        energy+=zeta_ji.z*tpainv;
      }
      if (vflag>0) {
        numtyp mforce = -force;
        virial[0] += mdelr1[0]*mdelr1[0]*mforce;
        virial[1] += mdelr1[1]*mdelr1[1]*mforce;
        virial[2] += mdelr1[2]*mdelr1[2]*mforce;
        virial[3] += mdelr1[0]*mdelr1[1]*mforce;
        virial[4] += mdelr1[0]*mdelr1[2]*mforce;
        virial[5] += mdelr1[1]*mdelr1[2]*mforce;
      }

      // attractive forces
      for (nbor_k = nbork_start ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int jikparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype];

        numtyp delr2[3];
        delr2[0] = kx.x-jx.x;
        delr2[1] = kx.y-jx.y;
        delr2[2] = kx.z-jx.z;
        numtyp rsq2 = delr2[0]*delr2[0] + delr2[1]*delr2[1] + delr2[2]*delr2[2];

        if (rsq2 > cutsq[jikparam]) continue;
        numtyp r2 = ucl_sqrt(rsq2);
        numtyp r2inv = ucl_rsqrt(rsq2);
        numtyp4 ts1_param, ts2_param, ts4_param;
        numtyp fi[3];

        ts1_param = ts1[jikparam]; //fetch4(ts1_jikparam,jikparam,ts1_tex);
        lam3 = ts1_param.z;
        powermint = ts1_param.w;
        ts2_param = ts2[jikparam]; //fetch4(ts2_jikparam,jikparam,ts2_tex);
        bigr = ts2_param.z;
        bigd = ts2_param.w;
        ts4_param = ts4[jikparam]; //fetch4(ts4_jikparam,jikparam,ts4_tex);
        c = ts4_param.x;
        d = ts4_param.y;
        h = ts4_param.z;
        gamma = ts4_param.w;
        attractive_fj(bigr, bigd, powermint, lam3, c, d, h, gamma,
                      prefactor_ji, r1, r1inv, r2, r2inv, mdelr1, delr2, fi);
        f.x += fi[0];
        f.y += fi[1];
        f.z += fi[2];

        // idx to zetaij is shifted by n_stride relative to nbor_k in dev_short_nbor
        int idx = nbor_k;
        if (dev_packed==dev_nbor) idx -= n_stride;

        acctyp4 zeta_jk = zetaij[idx]; // fetch(zeta_jk,idx,zeta_tex);
        numtyp prefactor_jk = zeta_jk.y;
        int jkiparam=elem2param[jtype*nelements*nelements+ktype*nelements+itype];
        ts1_param = ts1[jkiparam]; //fetch4(ts1_jkiparam,jkiparam,ts1_tex);
        lam3 = ts1_param.z;
        powermint = ts1_param.w;
        ts2_param = ts2[jkiparam]; //fetch4(ts2_jkiparam,jkiparam,ts2_tex);
        bigr = ts2_param.z;
        bigd = ts2_param.w;
        ts4_param = ts4[jkiparam]; //fetch4(ts4_jkiparam,jkiparam,ts4_tex);
        c = ts4_param.x;
        d = ts4_param.y;
        h = ts4_param.z;
        gamma = ts4_param.w;
        attractive_fk(bigr, bigd, powermint, lam3, c, d, h, gamma,
                      prefactor_jk, r2, r2inv, r1, r1inv, delr2, mdelr1, fi);
        f.x += fi[0];
        f.y += fi[1];
        f.z += fi[2];
      } // for nbor_k
    } // for nbor_j

    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

__kernel void k_tersoff_three_end_vatom(const __global numtyp4 *restrict x_,
                                        const __global numtyp4 *restrict ts1_in,
                                        const __global numtyp4 *restrict ts2_in,
                                        const __global numtyp4 *restrict ts4_in,
                                        const __global numtyp *restrict cutsq,
                                        const __global int *restrict map,
                                        const __global int *restrict elem2param,
                                        const int nelements, const int nparams,
                                        const __global acctyp4 *restrict zetaij,
                                        const __global int * dev_nbor,
                                        const __global int * dev_packed,
                                        const __global int * dev_ilist,
                                        const __global int * dev_short_nbor,
                                        __global acctyp4 *restrict ans,
                                        __global acctyp *restrict engv,
                                        const int eflag, const int vflag,
                                        const int inum,  const int nbor_pitch,
                                        const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp lam3, powermint, bigr, bigd, c, d, h, gamma;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  __local numtyp4 ts1[SHARED_SIZE];
  __local numtyp4 ts2[SHARED_SIZE];
  __local numtyp4 ts4[SHARED_SIZE];
  if (tid<nparams) {
    ts1[tid]=ts1_in[tid];
    ts2[tid]=ts2_in[tid];
    ts4[tid]=ts4_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __local int red_acc[2*BLOCK_PAIR];

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;
    const __global int* nbor_mem = dev_packed;
    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    numtyp tpainv = ucl_recip((numtyp)t_per_atom);

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor_j];
      nbor_j += n_stride;
      nbor_end = nbor_j+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=nbor_mem[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];
      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delr1[3];
      delr1[0] = jx.x-ix.x;
      delr1[1] = jx.y-ix.y;
      delr1[2] = jx.z-ix.z;
      numtyp rsq1 = delr1[0]*delr1[0] + delr1[1]*delr1[1] + delr1[2]*delr1[2];

      numtyp mdelr1[3];
      mdelr1[0] = -delr1[0];
      mdelr1[1] = -delr1[1];
      mdelr1[2] = -delr1[2];

      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_ilist[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      // recalculate numk and k_end for the use of short neighbor list
      if (dev_packed==dev_nbor) {
        numk = dev_short_nbor[nbor_k];
        nbor_k += n_stride;
        k_end = nbor_k+fast_mul(numk,n_stride);
      }
      int nbork_start = nbor_k;

      // look up for zeta_ji
      int m = tid / t_per_atom;
      int ijnum = -1;
      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;
        if (k == i) {
          ijnum = nbor_k;
          red_acc[2*m+0] = ijnum;
          red_acc[2*m+1] = offset_k;
          break;
        }
      }

      numtyp r1 = ucl_sqrt(rsq1);
      numtyp r1inv = ucl_rsqrt(rsq1);
      int offset_kf;
      if (ijnum >= 0) {
        offset_kf = offset_k;
      } else {
        ijnum = red_acc[2*m+0];
        offset_kf = red_acc[2*m+1];
      }

      // idx to zetaij is shifted by n_stride relative to ijnum in dev_short_nbor
      int idx = ijnum;
      if (dev_packed==dev_nbor) idx -= n_stride;
      acctyp4 zeta_ji = zetaij[idx]; //  fetch(zeta_ji,idx,zeta_tex);
      numtyp force = zeta_ji.x*tpainv;
      numtyp prefactor_ji = zeta_ji.y;
      f.x += delr1[0]*force;
      f.y += delr1[1]*force;
      f.z += delr1[2]*force;

      if (eflag>0) {
        energy+=zeta_ji.z*tpainv;
      }
      if (vflag>0) {
        numtyp mforce = -force;
        virial[0] += mdelr1[0]*mdelr1[0]*mforce;
        virial[1] += mdelr1[1]*mdelr1[1]*mforce;
        virial[2] += mdelr1[2]*mdelr1[2]*mforce;
        virial[3] += mdelr1[0]*mdelr1[1]*mforce;
        virial[4] += mdelr1[0]*mdelr1[2]*mforce;
        virial[5] += mdelr1[1]*mdelr1[2]*mforce;
      }

      // attractive forces
      for (nbor_k = nbork_start; nbor_k<k_end; nbor_k+=n_stride) {
        int k=nbor_mem[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int jikparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype];

        numtyp delr2[3];
        delr2[0] = kx.x-jx.x;
        delr2[1] = kx.y-jx.y;
        delr2[2] = kx.z-jx.z;
        numtyp rsq2 = delr2[0]*delr2[0] + delr2[1]*delr2[1] + delr2[2]*delr2[2];

        if (rsq2 > cutsq[jikparam]) continue;
        numtyp r2 = ucl_sqrt(rsq2);
        numtyp r2inv = ucl_rsqrt(rsq2);

        numtyp fi[3], fj[3], fk[3];
        numtyp4 ts1_param, ts2_param, ts4_param;
        ts1_param = ts1[jikparam]; //fetch4(ts1_jikparam,jikparam,ts1_tex);
        lam3 = ts1_param.z;
        powermint = ts1_param.w;
        ts2_param = ts2[jikparam]; //fetch4(ts2_jikparam,jikparam,ts2_tex);
        bigr = ts2_param.z;
        bigd = ts2_param.w;
        ts4_param = ts4[jikparam]; //fetch4(ts4_jikparam,jikparam,ts4_tex);
        c = ts4_param.x;
        d = ts4_param.y;
        h = ts4_param.z;
        gamma = ts4_param.w;
        attractive(bigr, bigd, powermint, lam3, c, d, h, gamma,
                   prefactor_ji, r1, r1inv, r2, r2inv, mdelr1, delr2, fi, fj, fk);
        f.x += fj[0];
        f.y += fj[1];
        f.z += fj[2];

        virial[0] += TWOTHIRD*(mdelr1[0]*fj[0] + delr2[0]*fk[0]);
        virial[1] += TWOTHIRD*(mdelr1[1]*fj[1] + delr2[1]*fk[1]);
        virial[2] += TWOTHIRD*(mdelr1[2]*fj[2] + delr2[2]*fk[2]);
        virial[3] += TWOTHIRD*(mdelr1[0]*fj[1] + delr2[0]*fk[1]);
        virial[4] += TWOTHIRD*(mdelr1[0]*fj[2] + delr2[0]*fk[2]);
        virial[5] += TWOTHIRD*(mdelr1[1]*fj[2] + delr2[1]*fk[2]);

        // idx to zetaij is shifted by n_stride relative to nbor_k in dev_short_nbor
        int idx = nbor_k;
        if (dev_packed==dev_nbor) idx -= n_stride;
        acctyp4 zeta_jk = zetaij[idx]; // fetch(zeta_jk,idx,zeta_tex);
        numtyp prefactor_jk = zeta_jk.y;

        int jkiparam=elem2param[jtype*nelements*nelements+ktype*nelements+itype];
        ts1_param = ts1[jkiparam]; //fetch4(ts1_jkiparam,jkiparam,ts1_tex);
        lam3 = ts1_param.z;
        powermint = ts1_param.w;
        ts2_param = ts2[jkiparam]; //fetch4(ts2_jkiparam,jkiparam,ts2_tex);
        bigr = ts2_param.z;
        bigd = ts2_param.w;
        ts4_param = ts4[jkiparam]; //fetch4(ts4_jkiparam,jkiparam,ts4_tex);
        c = ts4_param.x;
        d = ts4_param.y;
        h = ts4_param.z;
        gamma = ts4_param.w;
        attractive(bigr, bigd, powermint, lam3, c, d, h, gamma,
                   prefactor_jk, r2, r2inv, r1, r1inv, delr2, mdelr1, fi, fj, fk);
        f.x += fk[0];
        f.y += fk[1];
        f.z += fk[2];

        virial[0] += TWOTHIRD*(delr2[0]*fj[0] + mdelr1[0]*fk[0]);
        virial[1] += TWOTHIRD*(delr2[1]*fj[1] + mdelr1[1]*fk[1]);
        virial[2] += TWOTHIRD*(delr2[2]*fj[2] + mdelr1[2]*fk[2]);
        virial[3] += TWOTHIRD*(delr2[0]*fj[1] + mdelr1[0]*fk[1]);
        virial[4] += TWOTHIRD*(delr2[0]*fj[2] + mdelr1[0]*fk[2]);
        virial[5] += TWOTHIRD*(delr2[1]*fj[2] + mdelr1[1]*fk[2]);
      }
    } // for nbor

    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

