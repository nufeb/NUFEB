#include "hip/hip_runtime.h"
// **************************************************************************
//                                  morse.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the morse pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif

#else
#define pos_tex x_
#endif

__kernel void k_morse(const __global numtyp4 *restrict x_,
                      const __global numtyp4 *restrict mor1,
                      const __global numtyp2 *restrict mor2,
                      const int lj_types,
                      const __global numtyp *restrict sp_lj_in,
                      const __global int *dev_nbor,
                      const __global int *dev_packed,
                      __global acctyp4 *restrict ans,
                      __global acctyp *restrict engv,
                      const int eflag, const int vflag, const int inum,
                      const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (r<mor1[mtype].x) {
        r=ucl_sqrt(r);
        numtyp dexp=r-mor1[mtype].z;
        dexp=ucl_exp(-mor1[mtype].w*dexp);
        numtyp dm=dexp*dexp-dexp;
        numtyp force = mor1[mtype].y*dm/r*factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=mor2[mtype].x*(dexp*dexp - 2.0*dexp) - mor2[mtype].y;
          energy+=e*factor_lj;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_morse_fast(const __global numtyp4 *restrict x_,
                           const __global numtyp4 *restrict mor1_in,
                           const __global numtyp2 *restrict mor2_in,
                           const __global numtyp *restrict sp_lj_in,
                           const __global int *dev_nbor,
                           const __global int *dev_packed,
                           __global acctyp4 *restrict ans,
                           __global acctyp *restrict engv,
                           const int eflag, const int vflag, const int inum,
                           const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 mor1[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp2 mor2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    mor1[tid]=mor1_in[tid];
    if (eflag>0)
      mor2[tid]=mor2_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp r = delx*delx+dely*dely+delz*delz;

      if (r<mor1[mtype].x) {
        r=ucl_sqrt(r);
        numtyp dexp=r-mor1[mtype].z;
        dexp=ucl_exp(-mor1[mtype].w*dexp);
        numtyp dm=dexp*dexp-dexp;
        numtyp force = mor1[mtype].y*dm/r*factor_lj;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=mor2[mtype].x*(dm-dexp)-mor2[mtype].y;
          energy+=e*factor_lj;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

